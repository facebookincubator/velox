#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "velox/experimental/wave/exec/ExprKernel.h"

#include <gflags/gflags.h>
#include "velox/experimental/wave/common/Block.cuh"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/exec/Aggregate.cuh"
#include "velox/experimental/wave/exec/WaveCore.cuh"

DECLARE_bool(kernel_gdb);

namespace facebook::velox::wave {

__global__ void oneFilter(KernelParams params, int32_t pc, int32_t base) {
  PROGRAM_PREAMBLE(base);
  filterKernel(
      instruction[pc]._.filter, operands, blockBase, shared, laneStatus);
  wrapKernel(
      instruction[pc + 1]._.wrap,
      operands,
      blockBase,
      shared->numRows,
      &shared->data);
  PROGRAM_EPILOGUE();
}

} // namespace facebook::velox::wave
