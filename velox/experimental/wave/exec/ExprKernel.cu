#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "velox/experimental/wave/exec/ExprKernel.h"

#include <assert.h>
#include <gflags/gflags.h>
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/exec/ExprKernelStream.h"
#include "velox/experimental/wave/exec/WaveCore.cuh"

DEFINE_bool(kernel_gdb, false, "Run kernels sequentially for debugging");

namespace facebook::velox::wave {

void __global__ setupAggregationKernel(AggregationControl op) {
  assert(!op.oldBuckets);
  auto* data = new (op.head) DeviceAggregation();
  data->rowSize = op.rowSize;
  data->singleRow = reinterpret_cast<char*>(data + 1);
  memset(data->singleRow, 0, op.rowSize);
}

void WaveKernelStream::setupAggregation(
    AggregationControl& op,
    int32_t entryPoint,
    CompiledKernel* kernel) {
  int32_t numBlocks = 1;
  int32_t numThreads = 1;
  if (op.oldBuckets) {
    // One thread per bucket. Enough TBs for full device.
    numThreads = kBlockSize;
    numBlocks = std::min<int64_t>(
        roundUp(op.numOldBuckets, kBlockSize) / kBlockSize, 640);
  }
  if (kernel) {
    void* args = &op;
    kernel->launch(entryPoint, numBlocks, numThreads, 0, this, &args);
  } else {
    setupAggregationKernel<<<numBlocks, numThreads, 0, stream_->stream>>>(op);
  }
  wait();
}

} // namespace facebook::velox::wave
