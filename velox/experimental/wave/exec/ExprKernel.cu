#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "velox/experimental/wave/exec/WaveCore.cuh"

namespace facebook::velox::wave {

template <typename T>
__device__ inline T opFunc_kPlus(T left, T right) {
  return left + right;
}

template <typename T, typename OpFunc>
__device__ inline void binaryOpKernel(
    OpFunc func,
    IBinary& op,
    int32_t blockBase,
    char* shared,
    BlockStatus* status) {}

__device__ void filterKernel(
    const IFilter& filter,
    int32_t blockBase,
    char* shared,
    int32_t& numRows) {
  auto* flags = filter.flags;
  if (flags->nulls) {
    boolBlockToIndices<kBlockSize>(
        [&]() -> uint8_t {
          return threadIdx.x >= numRows
              ? 0
              : flatValue<uint8_t>(flags->base, blockBase) &
                  flatValue<uint8_t>(flags->nulls, blockBase);
        },
        blockBase,
        filter.indices + blockBase,
        shared,
        numRows);
  } else {
    boolBlockToIndices<kBlockSize>(
        [&]() -> uint8_t {
          return threadIdx.x >= numRows
              ? 0
              : flatValue<uint8_t>(flags->base, blockBase);
        },
        blockBase,
        filter.indices + blockBase,
        shared,
        numRows);
  }
}

__device__ void wrapKernel(IWrap& wrap, int32_t blockBase, int32_t& numRows) {}

#define OP_MIX(op, t) \
  static_cast<OpCode>(static_cast<int32_t>(t) + 8 * static_cast<int32_t>(op))

#define BINARY_TYPES(opCode, OP)                             \
  case OP_MIX(opCode, ScalarType::kInt32):                   \
    binaryOpKernel<int32_t>(                                 \
        [](auto left, auto right) { return left OP right; }, \
        instruction->_.binary,                               \
        blockBase,                                           \
        shared,                                              \
        status);                                             \
    break;

__global__ void waveBaseKernel(
    ThreadBlockProgram** programs,
    int32_t* baseIndices,
    BlockStatus* blockStatusArray) {
  using ScanAlgorithm = hipcub::BlockScan<int, 256, hipcub::BLOCK_SCAN_RAKING>;
  extern __shared__ __align__(
      alignof(typename ScanAlgorithm::TempStorage)) char shared[];
  auto* program = programs[blockIdx.x];
  auto* status = &blockStatusArray[blockIdx.x];
  int32_t blockBase = blockIdx.x - baseIndices[blockIdx.x];
  for (auto i = 0; i < program->numInstructions; ++i) {
    auto instruction = program->instructions[i];
    switch (instruction->opCode) {
      case OpCode::kFilter:
        filterKernel(instruction->_.filter, blockBase, shared, status->numRows);
        break;

      case OpCode::kWrap:
        wrapKernel(instruction->_.wrap, blockBase, status->numRows);
        break;

        BINARY_TYPES(OpCode::kPlus, +);
    }
  }
}

} // namespace facebook::velox::wave
