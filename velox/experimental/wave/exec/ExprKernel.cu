#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "velox/experimental/wave/exec/ExprKernel.h"

#include <gflags/gflags.h>
#include "velox/experimental/wave/common/Block.cuh"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/exec/Aggregate.cuh"
#include "velox/experimental/wave/exec/WaveCore.cuh"

DEFINE_bool(kernel_gdb, false, "Run kernels sequentially for debugging");

namespace facebook::velox::wave {

#define BINARY_TYPES(opCode, TP, OP)                         \
  case opCode:                                               \
    binaryOpKernel<TP>(                                      \
        [](auto left, auto right) { return left OP right; }, \
        instruction->_.binary,                               \
        operands,                                            \
        blockBase,                                           \
        &shared->data,                                       \
        laneStatus);                                         \
    break;

__global__ void oneAggregate(KernelParams params, int32_t pc, int32_t base) {
  PROGRAM_PREAMBLE(base);
  aggregateKernel(instruction[pc]._.aggregate, shared, laneStatus);
  PROGRAM_EPILOGUE();
}

__global__ void oneReadAggregate(KernelParams params, int32_t pc, int32_t base);

__global__ void onePlusBigint(KernelParams params, int32_t pc, int32_t base);
template <typename T>
__global__ void oneLt(KernelParams params, int32_t pc, int32_t base) {
  PROGRAM_PREAMBLE(base);
  binaryOpKernel<T>(
      [](auto left, auto right) { return left < right; },
      instruction[pc]._.binary,
      operands,
      blockBase,
      &shared->data,
      laneStatus);
  PROGRAM_EPILOGUE();
}

__global__ void oneFilter(KernelParams params, int32_t pc, int32_t base);

__global__ void waveBaseKernel(KernelParams params) {
  PROGRAM_PREAMBLE(0);
  for (;;) {
    switch (instruction->opCode) {
      case OpCode::kReturn:
        PROGRAM_EPILOGUE();
        return;

      case OpCode::kFilter:
        filterKernel(
            instruction->_.filter, operands, blockBase, shared, laneStatus);
        break;

      case OpCode::kWrap:
        wrapKernel(
            instruction->_.wrap,
            operands,
            blockBase,
            shared->numRows,
            &shared->data);
        break;
      case OpCode::kAggregate:
        aggregateKernel(instruction->_.aggregate, shared, laneStatus);
        break;
      case OpCode::kReadAggregate:
        readAggregateKernel(instruction->_.aggregate, shared);
        break;
        BINARY_TYPES(OpCode::kPlus_BIGINT, int64_t, +);
        BINARY_TYPES(OpCode::kLT_BIGINT, int64_t, <);
    }
    ++instruction;
  }
}

int32_t instructionSharedMemory(const Instruction& instruction) {
  switch (instruction.opCode) {
    case OpCode::kFilter:
      return sizeof(WaveShared) +
          (2 + (kBlockSize / kWarpThreads)) * sizeof(int32_t);
    default:
      return sizeof(WaveShared);
  }
}

#define CALL_ONE(k, params, pc, base) \
  k<<<blocksPerExe,                   \
      kBlockSize,                     \
      sharedSize,                     \
      alias ? alias->stream()->stream : stream()->stream>>>(params, pc, base);

void WaveKernelStream::callOne(
    Stream* alias,
    int32_t numBlocks,
    int32_t sharedSize,
    KernelParams& params) {
  int32_t blocksPerExe = 0;
  auto first = params.programIdx[0];
  for (; blocksPerExe < numBlocks; ++blocksPerExe) {
    if (params.programIdx[blocksPerExe] != first) {
      break;
    }
  }
  std::vector<std::vector<OpCode>> programs;
  for (auto i = 0; i < numBlocks; i += blocksPerExe) {
    auto programIdx = programs.size();
    programs.emplace_back();
    auto* instructions = params.programs[programIdx]->instructions;
    for (auto pc = 0; instructions[pc].opCode != OpCode::kReturn; ++pc) {
      programs.back().push_back(instructions[pc].opCode);
    }
  }
  auto initialStartPC = params.startPC;
  for (auto programIdx = 0; programIdx < programs.size(); ++programIdx) {
    auto& program = programs[programIdx];
    int32_t base = programIdx * blocksPerExe;
    params.startPC = initialStartPC;
    int32_t start = 0;
    if (params.startPC) {
      start = params.startPC[programIdx];
    }
    for (auto pc = start; pc < program.size(); ++pc) {
      assert(params.programs[0]->instructions != nullptr);
      switch (program[pc]) {
        case OpCode::kFilter:
          CALL_ONE(oneFilter, params, pc, base)
          ++pc;
          break;
        case OpCode::kAggregate:
          CALL_ONE(oneAggregate, params, pc, base)
          break;
        case OpCode::kReadAggregate:
          CALL_ONE(oneReadAggregate, params, pc, base)
          break;
        case OpCode::kPlus_BIGINT:
          CALL_ONE(onePlusBigint, params, pc, base);
          break;
        case OpCode::kLT_BIGINT:
          CALL_ONE(oneLt<int64_t>, params, pc, base);
          break;
        default:
          assert(false);
      }
    }
    params.startPC = nullptr;
  }
}

void WaveKernelStream::call(
    Stream* alias,
    int32_t numBlocks,
    int32_t sharedSize,
    KernelParams& params) {
  if (FLAGS_kernel_gdb) {
    callOne(alias, numBlocks, sharedSize, params);
    (alias ? alias : this)->wait();
    return;
  }

  waveBaseKernel<<<
      numBlocks,
      kBlockSize,
      sharedSize,
      alias ? alias->stream()->stream : stream()->stream>>>(params);
}

REGISTER_KERNEL("expr", waveBaseKernel);

void __global__ setupAggregationKernel(AggregationControl op) {
  //    assert(op.maxTableEntries == 0);
  auto* data = new (op.head) DeviceAggregation();
  data->rowSize = op.rowSize;
  data->singleRow = reinterpret_cast<char*>(data + 1);
  memset(data->singleRow, 0, op.rowSize);
}

void WaveKernelStream::setupAggregation(AggregationControl& op) {
  setupAggregationKernel<<<1, 1, 0, stream_->stream>>>(op);
  wait();
}

} // namespace facebook::velox::wave
