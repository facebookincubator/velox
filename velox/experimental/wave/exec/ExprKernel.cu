#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "velox/experimental/wave/exec/ExprKernel.h"

#include <gflags/gflags.h>
#include "velox/experimental/wave/common/Block.cuh"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/exec/WaveCore.cuh"

DEFINE_bool(kernel_gdb, false, "Run kernels sequentially for debugging");

namespace facebook::velox::wave {

template <typename T>
__device__ inline T opFunc_kPlus(T left, T right) {
  return left + right;
}

template <typename T, typename OpFunc>
__device__ inline void binaryOpKernel(
    OpFunc func,
    IBinary& instr,
    Operand** operands,
    int32_t blockBase,
    char* shared,
    BlockStatus* status) {
  if (threadIdx.x >= status->numRows) {
    return;
  }
  T left;
  T right;
  if (operandOrNull(operands, instr.left, blockBase, shared, left) &&
      operandOrNull(operands, instr.right, blockBase, shared, right)) {
    flatResult<decltype(func(left, right))>(
        operands, instr.result, blockBase, shared) = func(left, right);
  } else {
    resultNull(operands, instr.result, blockBase, shared);
  }
}

__device__ void filterKernel(
    const IFilter& filter,
    Operand** operands,
    int32_t blockBase,
    char* shared,
    int32_t& numRows) {
  auto* flags = operands[filter.flags];
  auto* indices = operands[filter.indices];
  if (flags->nulls) {
    boolBlockToIndices<kBlockSize>(
        [&]() -> uint8_t {
          return threadIdx.x >= numRows
              ? 0
              : flatValue<uint8_t>(flags->base, blockBase) &
                  flatValue<uint8_t>(flags->nulls, blockBase);
        },
        blockBase,
        reinterpret_cast<int32_t*>(indices->base) + blockBase,
        shared,
        numRows);
  } else {
    boolBlockToIndices<kBlockSize>(
        [&]() -> uint8_t {
          return threadIdx.x >= numRows
              ? 0
              : flatValue<uint8_t>(flags->base, blockBase);
        },
        blockBase,
        reinterpret_cast<int32_t*>(indices->base) + blockBase,
        shared,
        numRows);
  }
}

__device__ void wrapKernel(
    const IWrap& wrap,
    Operand** operands,
    int32_t blockBase,
    int32_t numRows) {
  Operand* op = operands[wrap.indices];
  auto* filterIndices = reinterpret_cast<int32_t*>(op->base);
  if (filterIndices[blockBase + numRows - 1] == numRows + blockBase - 1) {
    // There is no cardinality change.
    return;
  }
  bool rowActive = threadIdx.x < numRows;
  for (auto column = 0; column < wrap.numColumns; ++column) {
    int32_t newIndex;
    int32_t** opIndices;
    bool remap = false;
    if (rowActive) {
      auto opIndex = wrap.columns[column];
      auto* op = operands[opIndex];
      opIndices = &op->indices[blockBase / kBlockSize];
      remap = *opIndices != nullptr;
      if (remap) {
        newIndex =
            (*opIndices)[filterIndices[blockBase + threadIdx.x] - blockBase];
      } else if (threadIdx.x == 0) {
        *opIndices = filterIndices + blockBase;
      }
    }
    // All threads hit this.
    __syncthreads();
    if (remap) {
      // remap can b true only on activ rows.
      (*opIndices)[threadIdx.x] = newIndex;
    }
  }
  __syncthreads();
}

#define BINARY_TYPES(opCode, OP)                             \
  case OP_MIX(opCode, WaveTypeKind::BIGINT):                 \
    binaryOpKernel<int64_t>(                                 \
        [](auto left, auto right) { return left OP right; }, \
        instruction->_.binary,                               \
        operands,                                            \
        blockBase,                                           \
        shared,                                              \
        status);                                             \
    break;

__global__ void waveBaseKernel(
    int32_t* baseIndices,
    int32_t* programIndices,
    ThreadBlockProgram** programs,
    Operand*** programOperands,
    BlockStatus* blockStatusArray) {
  using ScanAlgorithm = hipcub::BlockScan<int, 256, hipcub::BLOCK_SCAN_RAKING>;
  extern __shared__ __align__(
      alignof(typename ScanAlgorithm::TempStorage)) char shared[];
  int programIndex = programIndices[blockIdx.x];
  auto* program = programs[programIndex];
  auto* operands = programOperands[programIndex];
  auto* status = &blockStatusArray[blockIdx.x - baseIndices[blockIdx.x]];
  int32_t blockBase = (blockIdx.x - baseIndices[blockIdx.x]) * blockDim.x;
  auto instruction = program->instructions;
  for (;;) {
    switch (instruction->opCode) {
      case OpCode::kReturn:
        return;
      case OpCode::kFilter:
        filterKernel(
            instruction->_.filter,
            operands,
            blockBase,
            shared,
            status->numRows);
        break;

      case OpCode::kWrap:
        wrapKernel(instruction->_.wrap, operands, blockBase, status->numRows);
        break;

        BINARY_TYPES(OpCode::kPlus, +);
        BINARY_TYPES(OpCode::kLT, <);
    }
    ++instruction;
  }
}

int32_t instructionSharedMemory(const Instruction& instruction) {
  using ScanAlgorithm = hipcub::BlockScan<int, 256, hipcub::BLOCK_SCAN_RAKING>;

  switch (instruction.opCode) {
    case OpCode::kFilter:
      return sizeof(ScanAlgorithm::TempStorage);
    default:
      return 0;
  }
}

void WaveKernelStream::call(
    Stream* alias,
    int32_t numBlocks,
    int32_t* bases,
    int32_t* programIdx,
    ThreadBlockProgram** programs,
    Operand*** operands,
    BlockStatus* status,
    int32_t sharedSize) {
  waveBaseKernel<<<
      numBlocks,
      kBlockSize,
      sharedSize,
      alias ? alias->stream()->stream : stream()->stream>>>(
      bases, programIdx, programs, operands, status);
  if (FLAGS_kernel_gdb) {
    (alias ? alias : this)->wait();
  }
}

} // namespace facebook::velox::wave
