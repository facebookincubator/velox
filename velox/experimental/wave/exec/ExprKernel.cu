#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "velox/experimental/wave/exec/ExprKernel.h"

#include "velox/experimental/wave/common/Block.cuh"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/exec/WaveCore.cuh"

namespace facebook::velox::wave {

template <typename T>
__device__ inline T opFunc_kPlus(T left, T right) {
  return left + right;
}

template <typename T, typename OpFunc>
__device__ inline void binaryOpKernel(
    OpFunc func,
    IBinary& op,
    Operand** operands,
    int32_t blockBase,
    char* shared,
    BlockStatus* status) {
  if (threadIdx.x >= status->numRows) {
    return;
  }
  flatResult<T>(operands, op.result, blockBase, shared) = func(
      getOperand<T>(operands, op.left, blockBase, shared),
      getOperand<T>(operands, op.right, blockBase, shared));
}

__device__ void filterKernel(
    const IFilter& filter,
    Operand** operands,
    int32_t blockBase,
    char* shared,
    int32_t& numRows) {
  auto* flags = operands[filter.flags];
  auto* indices = operands[filter.indices];
  if (flags->nulls) {
    boolBlockToIndices<kBlockSize>(
        [&]() -> uint8_t {
          return threadIdx.x >= numRows
              ? 0
              : flatValue<uint8_t>(flags->base, blockBase) &
                  flatValue<uint8_t>(flags->nulls, blockBase);
        },
        blockBase,
        reinterpret_cast<int32_t*>(indices->base) + blockBase,
        shared,
        numRows);
  } else {
    boolBlockToIndices<kBlockSize>(
        [&]() -> uint8_t {
          return threadIdx.x >= numRows
              ? 0
              : flatValue<uint8_t>(flags->base, blockBase);
        },
        blockBase,
        reinterpret_cast<int32_t*>(indices->base) + blockBase,
        shared,
        numRows);
  }
}

__device__ void wrapKernel(
    IWrap& wrap,
    Operand** operands,
    int32_t blockBase,
    int32_t& numRows) {}

#define BINARY_TYPES(opCode, OP)                             \
  case OP_MIX(opCode, ScalarType::kInt64):                   \
    binaryOpKernel<int64_t>(                                 \
        [](auto left, auto right) { return left OP right; }, \
        instruction->_.binary,                               \
        operands,                                            \
        blockBase,                                           \
        shared,                                              \
        status);                                             \
    break;

__global__ void waveBaseKernel(
    int32_t* baseIndices,
    int32_t* programIndices,
    ThreadBlockProgram** programs,
    Operand*** programOperands,
    BlockStatus* blockStatusArray) {
  using ScanAlgorithm = hipcub::BlockScan<int, 256, hipcub::BLOCK_SCAN_RAKING>;
  extern __shared__ __align__(
      alignof(typename ScanAlgorithm::TempStorage)) char shared[];
  int programIndex = programIndices[blockIdx.x];
  auto* program = programs[programIndex];
  auto* operands = programOperands[programIndex];
  auto* status = &blockStatusArray[blockIdx.x - baseIndices[blockIdx.x]];
  int32_t blockBase = (blockIdx.x - baseIndices[blockIdx.x]) * blockDim.x;
  for (auto i = 0; i < program->numInstructions; ++i) {
    auto instruction = program->instructions[i];
    switch (instruction->opCode) {
      case OpCode::kFilter:
        filterKernel(
            instruction->_.filter,
            operands,
            blockBase,
            shared,
            status->numRows);
        break;

      case OpCode::kWrap:
        wrapKernel(instruction->_.wrap, operands, blockBase, status->numRows);
        break;

        BINARY_TYPES(OpCode::kPlus, +);
    }
  }
}

void WaveKernelStream::call(
    Stream* alias,
    int32_t numBlocks,
    int32_t* bases,
    int32_t* programIdx,
    ThreadBlockProgram** programs,
    Operand*** operands,
    BlockStatus* status,
    int32_t sharedSize) {
  waveBaseKernel<<<
      numBlocks,
      kBlockSize,
      sharedSize,
      alias ? alias->stream()->stream : stream()->stream>>>(
      bases, programIdx, programs, operands, status);
}

} // namespace facebook::velox::wave
