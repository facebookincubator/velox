#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "velox/experimental/wave/exec/ExprKernel.h"

#include <gflags/gflags.h>
#include "velox/experimental/wave/common/Block.cuh"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/exec/WaveCore.cuh"

DEFINE_bool(kernel_gdb, false, "Run kernels sequentially for debugging");

namespace facebook::velox::wave {

template <typename T>
__device__ inline T opFunc_kPlus(T left, T right) {
  return left + right;
}

template <typename T, typename OpFunc>
__device__ inline void binaryOpKernel(
    OpFunc func,
    IBinary& instr,
    Operand** operands,
    int32_t blockBase,
    char* shared,
    BlockStatus* status) {
  if (threadIdx.x >= status->numRows) {
    return;
  }
  T left;
  T right;
  if (operandOrNull(operands, instr.left, blockBase, shared, left) &&
      operandOrNull(operands, instr.right, blockBase, shared, right)) {
    flatResult<decltype(func(left, right))>(
        operands, instr.result, blockBase, shared) = func(left, right);
  } else {
    resultNull(operands, instr.result, blockBase, shared);
  }
}

__device__ void filterKernel(
    const IFilter& filter,
    Operand** operands,
    int32_t blockBase,
    char* shared,
    int32_t& numRows) {
  auto* flags = operands[filter.flags];
  auto* indices = operands[filter.indices];
  __syncthreads();
  if (flags->nulls) {
    bool256ToIndices<int32_t>(
        [&](int32_t group) -> uint64_t {
          int32_t offset = group * 8;
          int32_t base = blockBase + offset;
          if (offset + 8 <= numRows) {
            return *addCast<uint64_t>(flags->base, base) &
                *addCast<uint64_t>(flags->nulls, base);
          }
          if (offset >= numRows) {
            return 0;
          }
          return lowMask<uint64_t>((offset + 8 - numRows) * 8) &
              *addCast<uint64_t>(flags->base, base) &
              *addCast<uint64_t>(flags->nulls, base);
        },
        blockBase,
        reinterpret_cast<int32_t*>(indices->base) + blockBase,
        numRows,
        shared);
  } else {
    bool256ToIndices<int32_t>(
        [&](int32_t group) -> uint64_t {
          int32_t offset = group * 8;
          int32_t base = blockBase + offset;
          if (offset + 8 <= numRows) {
            return *addCast<uint64_t>(flags->base, base);
          }
          if (offset >= numRows) {
            return 0;
          }
          return lowMask<uint64_t>((numRows - offset) * 8) &
              *addCast<uint64_t>(flags->base, base);
        },
        blockBase,
        reinterpret_cast<int32_t*>(indices->base) + blockBase,
        numRows,
        shared);
  }
}

__device__ void wrapKernel(
    const IWrap& wrap,
    Operand** operands,
    int32_t blockBase,
    int32_t numRows,
    void* shared) {
  Operand* op = operands[wrap.indices];
  auto* filterIndices = reinterpret_cast<int32_t*>(op->base);
  if (filterIndices[blockBase + numRows - 1] == numRows + blockBase - 1) {
    // There is no cardinality change.
    return;
  }

  struct WrapState {
    int32_t* indices;
  };

  auto* state = reinterpret_cast<WrapState*>(shared);
  bool rowActive = threadIdx.x < numRows;
  for (auto column = 0; column < wrap.numColumns; ++column) {
    if (threadIdx.x == 0) {
      auto opIndex = wrap.columns[column];
      auto* op = operands[opIndex];
      int32_t** opIndices = &op->indices[blockBase / kBlockSize];
      if (!*opIndices) {
        *opIndices = filterIndices + blockBase;
        state->indices = nullptr;
      } else {
        state->indices = *opIndices;
      }
    }
    __syncthreads();
    // Every thread sees the decision on thred 0 above.
    if (!state->indices) {
      continue;
    }
    int32_t newIndex;
    if (rowActive) {
      newIndex =
          state->indices[filterIndices[blockBase + threadIdx.x] - blockBase];
    }
    // All threads hit this.
    __syncthreads();
    if (rowActive) {
      state->indices[threadIdx.x] = newIndex;
    }
  }
  __syncthreads();
}

#define BINARY_TYPES(opCode, OP)                             \
  case OP_MIX(opCode, WaveTypeKind::BIGINT):                 \
    binaryOpKernel<int64_t>(                                 \
        [](auto left, auto right) { return left OP right; }, \
        instruction->_.binary,                               \
        operands,                                            \
        blockBase,                                           \
        shared,                                              \
        status);                                             \
    break;

__global__ void waveBaseKernel(
    int32_t* baseIndices,
    int32_t* programIndices,
    ThreadBlockProgram** programs,
    Operand*** programOperands,
    BlockStatus* blockStatusArray) {
  extern __shared__ __align__(16) char shared[];
  int programIndex = programIndices[blockIdx.x];
  auto* program = programs[programIndex];
  auto* operands = programOperands[programIndex];
  auto* status = &blockStatusArray[blockIdx.x - baseIndices[blockIdx.x]];
  int32_t blockBase = (blockIdx.x - baseIndices[blockIdx.x]) * blockDim.x;
  auto instruction = program->instructions;
  for (;;) {
    switch (instruction->opCode) {
      case OpCode::kReturn:
        __syncthreads();
        return;
      case OpCode::kFilter:
        filterKernel(
            instruction->_.filter,
            operands,
            blockBase,
            shared,
            status->numRows);
        break;

      case OpCode::kWrap:
        wrapKernel(
            instruction->_.wrap, operands, blockBase, status->numRows, shared);
        break;

        BINARY_TYPES(OpCode::kPlus, +);
        BINARY_TYPES(OpCode::kLT, <);
    }
    ++instruction;
  }
}

int32_t instructionSharedMemory(const Instruction& instruction) {
  switch (instruction.opCode) {
    case OpCode::kFilter:
      return (2 + (kBlockSize / kWarpThreads)) * sizeof(int32_t);
    default:
      return 0;
  }
}

void WaveKernelStream::call(
    Stream* alias,
    int32_t numBlocks,
    int32_t* bases,
    int32_t* programIdx,
    ThreadBlockProgram** programs,
    Operand*** operands,
    BlockStatus* status,
    int32_t sharedSize) {
  waveBaseKernel<<<
      numBlocks,
      kBlockSize,
      sharedSize,
      alias ? alias->stream()->stream : stream()->stream>>>(
      bases, programIdx, programs, operands, status);
  if (FLAGS_kernel_gdb) {
    (alias ? alias : this)->wait();
  }
}
REGISTER_KERNEL("expr", waveBaseKernel);

} // namespace facebook::velox::wave
