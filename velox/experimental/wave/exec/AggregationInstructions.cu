#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "velox/experimental/wave/exec/AggregationInstructions.h"

#include <hipcub/hipcub.hpp> // @manual
#include "velox/experimental/wave/common/IdMap.cuh"
#include "velox/experimental/wave/exec/AggregateFunctionRegistry.h"
#include "velox/experimental/wave/exec/BuiltInAggregateFunctions.cuh"
#include "velox/experimental/wave/exec/WaveCore.cuh"

#define VELOX_WAVE_RETURN_NOT_OK(_expr)            \
  if (auto _ec = (_expr); _ec != ErrorCode::kOk) { \
    return _ec;                                    \
  }

#ifdef NDEBUG
#define LOG_TYPE_DISPATCH_ERROR(_kind)
#else
#define LOG_TYPE_DISPATCH_ERROR(_kind) \
  printf("%s:%d: Unsupported type %d\n", __FILE__, __LINE__, _kind)
#endif

#define KEY_TYPE_DISPATCH(_func, _kindExpr, ...) \
  [&]() {                                        \
    auto _kind = (_kindExpr);                    \
    switch (_kind) {                             \
      case PhysicalType::kInt32:                 \
        return _func<int32_t>(__VA_ARGS__);      \
      case PhysicalType::kInt64:                 \
        return _func<int64_t>(__VA_ARGS__);      \
      case PhysicalType::kString:                \
        return _func<StringView>(__VA_ARGS__);   \
      default:                                   \
        LOG_TYPE_DISPATCH_ERROR(_kind);          \
        return ErrorCode::kError;                \
    };                                           \
  }()

#define VALUE_TYPE_DISPATCH(_func, _kindExpr, ...) \
  [&]() {                                          \
    auto _kind = (_kindExpr);                      \
    switch (_kind) {                               \
      case PhysicalType::kInt8:                    \
        return _func<int8_t>(__VA_ARGS__);         \
      case PhysicalType::kInt16:                   \
        return _func<int16_t>(__VA_ARGS__);        \
      case PhysicalType::kInt32:                   \
        return _func<int32_t>(__VA_ARGS__);        \
      case PhysicalType::kInt64:                   \
        return _func<int64_t>(__VA_ARGS__);        \
      case PhysicalType::kFloat32:                 \
        return _func<float>(__VA_ARGS__);          \
      case PhysicalType::kFloat64:                 \
        return _func<double>(__VA_ARGS__);         \
      case PhysicalType::kString:                  \
        return _func<StringView>(__VA_ARGS__);     \
      default:                                     \
        LOG_TYPE_DISPATCH_ERROR(_kind);            \
        return ErrorCode::kError;                  \
    };                                             \
  }()

namespace facebook::velox::wave::aggregation {

namespace {

constexpr int kNormalizationRadix = 4;

struct BlockInfo {
  int base;
  char* shared;
};

template <typename T>
__device__ ErrorCode
normalize(BlockInfo* block, void* idMap, Operand* key, int32_t& result) {
  auto* typedIdMap = reinterpret_cast<IdMap<T>*>(idMap);
  auto id = typedIdMap->makeId(value<T>(key, block->base, block->shared));
  if (id == -1) {
    return ErrorCode::kInsuffcientMemory;
  }
  assert(typedIdMap->cardinality() <= kNormalizationRadix);
  result = kNormalizationRadix * result + id - 1;
  return ErrorCode::kOk;
}

template <typename T>
__device__ ErrorCode setGroupKey(
    BlockInfo* block,
    NormalizeKeys* normalizeKeys,
    int keyIndex,
    int groupIndex) {
  auto* container = normalizeKeys->container;
  *reinterpret_cast<T*>(container->groups[groupIndex].keys[keyIndex]) =
      value<T>(&normalizeKeys->inputs[keyIndex], block->base, block->shared);
  return ErrorCode::kOk;
}

__device__ ErrorCode run(BlockInfo* block, NormalizeKeys* normalizeKeys) {
  auto size = normalizeKeys->inputs[0].size;
  assert(normalizeKeys->result->size == size);
  if (threadIdx.x + block->base >= size) {
    return ErrorCode::kOk;
  }
  int32_t result = 0;
  auto* container = normalizeKeys->container;
  for (int i = 0; i < container->numKeys; ++i) {
    VELOX_WAVE_RETURN_NOT_OK(KEY_TYPE_DISPATCH(
        normalize,
        container->keyTypes[i].kind,
        block,
        container->idMaps[i],
        &normalizeKeys->inputs[i],
        result));
  }
  assert(result < container->numGroups);
  if (!atomicExch(&container->groups[result].initialized, 1)) {
    atomicAdd(&container->actualNumGroups, 1);
    for (int i = 0; i < container->numKeys; ++i) {
      VELOX_WAVE_RETURN_NOT_OK(KEY_TYPE_DISPATCH(
          setGroupKey,
          container->keyTypes[i].kind,
          block,
          normalizeKeys,
          i,
          result));
    }
  }
  flatResult<int32_t>(normalizeKeys->result, block->base) = result;
  return ErrorCode::kOk;
}

// Only one block should be writing to the same accumulator.
__device__ ErrorCode run(BlockInfo*, Aggregate* aggregate) {
  assert(aggregate->normalizedKey);
  assert(aggregate->container->useThreadLocalAccumulator);
  auto* function = aggregate->function;
  for (int i = threadIdx.x; i < aggregate->normalizedKey->size;
       i += blockDim.x) {
    auto key = value<int32_t>(aggregate->normalizedKey, i);
    auto& group = aggregate->container->groups[key];
    void* accumulator = (char*)group.accumulators[aggregate->accumulatorIndex] +
        threadIdx.x * function->accumulatorSize;
    // TODO: Try inline small common aggregate functions.
    VELOX_WAVE_RETURN_NOT_OK(function->addRawInput(
        aggregate->numInputs, aggregate->inputs, i, accumulator));
  }
  return ErrorCode::kOk;
}

template <typename T>
__device__ ErrorCode extractKey(Operand* result, int i, void* key) {
  reinterpret_cast<T*>(result->base)[i] = *reinterpret_cast<const T*>(key);
  return ErrorCode::kOk;
}

__device__ ErrorCode run(BlockInfo* block, ExtractKeys* extractKeys) {
  using Scan = hipcub::BlockScan<int, kBlockSize>;
  auto* tmp = reinterpret_cast<Scan::TempStorage*>(block->shared);
  auto* container = extractKeys->container;
  for (int i = threadIdx.x; i / kBlockSize * blockDim.x < container->numGroups;
       i += blockDim.x) {
    int outIndex =
        i < container->numGroups ? container->groups[i].initialized : 0;
    Scan(*tmp).ExclusiveSum(outIndex, outIndex);
    __syncthreads();
    if (i >= container->numGroups) {
      break;
    }
    if (container->groups[i].initialized) {
      KEY_TYPE_DISPATCH(
          extractKey,
          container->keyTypes[extractKeys->keyIndex].kind,
          extractKeys->result,
          outIndex,
          container->groups[i].keys[extractKeys->keyIndex]);
    }
  }
  return ErrorCode::kOk;
}

__device__ ErrorCode run(BlockInfo* block, ExtractValues* extractValues) {
  using Reduce = hipcub::BlockReduce<void*, kBlockSize>;
  auto* tmp = reinterpret_cast<Reduce::TempStorage*>(block->shared);
  auto* container = extractValues->container;
  assert(container->useThreadLocalAccumulator);
  auto* function = extractValues->function;
  for (int i = 0, outIndex = 0; i < container->numGroups; ++i) {
    auto& group = container->groups[i];
    if (!group.initialized) {
      continue;
    }
    void* accumulator =
        (char*)group.accumulators[extractValues->accumulatorIndex] +
        threadIdx.x * function->accumulatorSize;
    __syncthreads();
    accumulator =
        Reduce(*tmp).Reduce(accumulator, [function](void* a, void* b) {
          return function->mergeAccumulators(a, b);
        });
    __syncthreads();
    auto* ec = reinterpret_cast<ErrorCode*>(block->shared);
    if (threadIdx.x == 0) {
      *ec =
          function->extractValues(accumulator, extractValues->result, outIndex);
    }
    __syncthreads();
    VELOX_WAVE_RETURN_NOT_OK(*ec);
    ++outIndex;
  }
  return ErrorCode::kOk;
}

__global__ void runPrograms(
    ThreadBlockProgram* programs,
    int32_t* baseIndices,
    BlockStatus* blockStatusArray) {
  extern __shared__ __align__(64) char shared[];
  int baseIndex = baseIndices ? baseIndices[blockIdx.x] : 0;
  BlockInfo block = {
      .base = (int)(blockDim.x * (blockIdx.x - baseIndex)),
      .shared = shared,
  };
  auto& status = blockStatusArray[blockIdx.x];
  auto& program = programs[blockIdx.x];
  assert(status.errors[threadIdx.x] == ErrorCode::kOk);
  for (auto i = 0; i < program.numInstructions; ++i) {
    if (status.errors[threadIdx.x] != ErrorCode::kOk) {
      break;
    }
    auto& instruction = program.instructions[i];
    switch (instruction.opCode) {
      case OpCode::kNormalizeKeys:
        status.errors[threadIdx.x] = run(&block, &instruction._.normalizeKeys);
        break;
      case OpCode::kAggregate:
        status.errors[threadIdx.x] = run(&block, &instruction._.aggregate);
        break;
      case OpCode::kExtractKeys:
        status.errors[threadIdx.x] = run(&block, &instruction._.extractKeys);
        break;
      case OpCode::kExtractValues:
        status.errors[threadIdx.x] = run(&block, &instruction._.extractValues);
        break;
      default:
#ifndef NDEBUG
        printf(
            "%s:%d: Unsupported OpCode %d\n",
            __FILE__,
            __LINE__,
            instruction.opCode);
#endif
        status.errors[threadIdx.x] = ErrorCode::kError;
    }
  }
  assert(status.errors[threadIdx.x] == ErrorCode::kOk);
}

} // namespace

AggregateFunctionRegistry::AggregateFunctionRegistry(GpuAllocator* allocator)
    : allocator_(allocator) {}

// The definitions of concrete functions must be in one compilation unit, same
// as where they are used in kernel.
void AggregateFunctionRegistry::addAllBuiltInFunctions(Stream& stream) {
  // TODO: Parallelize the kernel calls.
  Entry entry;

  entry.accept = [](auto&) { return true; };
  entry.function = allocator_->allocate<AggregateFunction>();
  createFunction<Count>
      <<<1, 1, 0, stream.stream()->stream>>>(entry.function.get());
  CUDA_CHECK(hipGetLastError());
  stream.wait();
  entries_["count"].push_back(std::move(entry));

  entry.accept = [](auto& argTypes) {
    return argTypes.size() == 1 && argTypes[0].kind == PhysicalType::kInt32;
  };
  entry.function = allocator_->allocate<AggregateFunction>();
  createFunction<Sum<int32_t, int64_t>>
      <<<1, 1, 0, stream.stream()->stream>>>(entry.function.get());
  CUDA_CHECK(hipGetLastError());
  stream.wait();
  entries_["sum"].push_back(std::move(entry));

  entry.accept = [](auto& argTypes) {
    return argTypes.size() == 1 && argTypes[0].kind == PhysicalType::kInt64;
  };
  entry.function = allocator_->allocate<AggregateFunction>();
  createFunction<Sum<int64_t, int64_t>>
      <<<1, 1, 0, stream.stream()->stream>>>(entry.function.get());
  CUDA_CHECK(hipGetLastError());
  stream.wait();
  entries_["sum"].push_back(std::move(entry));

  entry.accept = [](auto& argTypes) {
    return argTypes.size() == 1 && argTypes[0].kind == PhysicalType::kFloat64;
  };
  entry.function = allocator_->allocate<AggregateFunction>();
  createFunction<Sum<double, double>>
      <<<1, 1, 0, stream.stream()->stream>>>(entry.function.get());
  CUDA_CHECK(hipGetLastError());
  stream.wait();
  entries_["sum"].push_back(std::move(entry));

  entry.accept = [](auto& argTypes) {
    return argTypes.size() == 1 && argTypes[0].kind == PhysicalType::kInt64;
  };
  entry.function = allocator_->allocate<AggregateFunction>();
  createFunction<Avg<int64_t>>
      <<<1, 1, 0, stream.stream()->stream>>>(entry.function.get());
  CUDA_CHECK(hipGetLastError());
  stream.wait();
  entries_["avg"].push_back(std::move(entry));

  entry.accept = [](auto& argTypes) {
    return argTypes.size() == 1 && argTypes[0].kind == PhysicalType::kFloat64;
  };
  entry.function = allocator_->allocate<AggregateFunction>();
  createFunction<Avg<double>>
      <<<1, 1, 0, stream.stream()->stream>>>(entry.function.get());
  CUDA_CHECK(hipGetLastError());
  stream.wait();
  entries_["avg"].push_back(std::move(entry));
}

AggregateFunction* AggregateFunctionRegistry::getFunction(
    const std::string& name,
    const Types& argTypes) const {
  auto it = entries_.find(name);
  if (it == entries_.end()) {
    return nullptr;
  }
  for (int i = it->second.size() - 1; i >= 0; --i) {
    if (it->second[i].accept(argTypes)) {
      return it->second[i].function.get();
    }
  }
  return nullptr;
}

int ExtractKeys::sharedSize() {
  return sizeof(hipcub::BlockScan<int, kBlockSize>::TempStorage);
}

int ExtractValues::sharedSize() {
  return sizeof(hipcub::BlockReduce<void*, kBlockSize>::TempStorage);
}

void call(
    Stream& stream,
    int numBlocks,
    ThreadBlockProgram* programs,
    int32_t* baseIndices,
    BlockStatus* status,
    int sharedSize) {
  runPrograms<<<numBlocks, kBlockSize, sharedSize, stream.stream()->stream>>>(
      programs, baseIndices, status);
  CUDA_CHECK(hipGetLastError());
}

} // namespace facebook::velox::wave::aggregation
