#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "velox/experimental/wave/common/Buffer.h"
#include "velox/experimental/wave/common/Cuda.h"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/GpuArena.h"
#include "velox/experimental/wave/dwio/decode/GpuDecoder.cuh"

namespace facebook::velox::wave {

int32_t GpuDecode::sharedMemorySize() const {
  return detail::sharedMemorySizeForDecode<kBlockSize>(step);
}

/// Describes multiple sequences of decode ops. Each TB executes a sequence of
/// decode steps. The data area starts with a range of instruction numbers for
/// each thread block. The first TB runs from 0 to ends[0]. The nth runs from
/// ends[nth-1] to ends[nth]. After gridDim.x ends, we round to an 8 aligned
/// offset and have an array of GpuDecodes.]
struct GpuDecodeParams {
  // If need to represent more than this many ops, use a dynamically allocated
  // external array in 'external'.
  static constexpr int32_t kMaxInlineOps = 50;

  // Pointer to standalone description of work. If nullptr, the description of
  // work fits inline in 'this'.
  GpuDecodeParams* external{nullptr};
  // The end of each decode program. The first starts at 0. The end is
  // ends[blockIdx.x].
  int32_t ends
      [kMaxInlineOps * (sizeof(GpuDecode) + sizeof(int32_t)) /
       sizeof(int32_t)] = {};
};

__global__ void decodeKernel(GpuDecodeParams inlineParams) {
  GpuDecodeParams* params =
      inlineParams.external ? inlineParams.external : &inlineParams;
  int32_t programStart = blockIdx.x == 0 ? 0 : params->ends[blockIdx.x - 1];
  int32_t programEnd = params->ends[blockIdx.x];
  GpuDecode* ops =
      reinterpret_cast<GpuDecode*>(&params->ends[0] + roundUp(gridDim.x, 2));
  for (auto i = programStart; i < programEnd; ++i) {
    detail::decodeSwitch<kBlockSize>(ops[i]);
  }
}

void launchDecode(
    const DecodePrograms& programs,
    GpuArena* arena,
    WaveBufferPtr& extra,
    Stream* stream) {
  int32_t numBlocks = programs.programs.size();
  int32_t numOps = 0;
  int32_t shared = 0;
  for (auto& program : programs.programs) {
    numOps += program.size();
    for (auto& step : program) {
      shared = std::max(
          shared, detail::sharedMemorySizeForDecode<kBlockSize>(step->step));
    }
  }
  if (shared > 0) {
    shared += 15; // allow align at 16.
  }
  GpuDecodeParams localParams;
  GpuDecodeParams* params = &localParams;
  if (numOps > GpuDecodeParams::kMaxInlineOps) {
    extra = arena->allocate<char>(
        (numBlocks + 1) * (sizeof(GpuDecode) + sizeof(int32_t)));
    params = extra->as<GpuDecodeParams>();
  }
  int32_t end = programs.programs[0].size();
  GpuDecode* decodes =
      reinterpret_cast<GpuDecode*>(&params->ends[0] + roundUp(numBlocks, 2));
  int32_t fill = 0;
  for (auto i = 0; i < programs.programs.size(); ++i) {
    params->ends[i] =
        (i == 0 ? 0 : params->ends[i - 1]) + programs.programs[i].size();
    for (auto& op : programs.programs[i]) {
      decodes[fill++] = *op;
    }
  }
  if (extra) {
    localParams.external = params;
  }

  decodeKernel<<<numBlocks, kBlockSize, shared, stream->stream()->stream>>>(
      localParams);
  CUDA_CHECK(hipGetLastError());
  if (programs.result) {
    if (!programs.hostResult) {
      stream->prefetch(
          nullptr, programs.result->as<char>(), programs.result->size());
    } else {
      stream->deviceToHostAsync(
          programs.hostResult->as<char>(),
          programs.result->as<char>(),
          programs.hostResult->size());
    }
  }
}

} // namespace facebook::velox::wave
