#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "velox/experimental/wave/common/Buffer.h"
#include "velox/experimental/wave/common/Cuda.h"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/GpuArena.h"
#include "velox/experimental/wave/dwio/decode/GpuDecoder.cuh"

namespace facebook::velox::wave {

int32_t GpuDecode::tempSize() const {
  // 1 int32 per lane as an upper limit.
  return std::max<int32_t>(kBlockSize * sizeof(int32_t), sizeof(NonNullState));
}

int32_t GpuDecode::sharedMemorySize() const {
  return detail::sharedMemorySizeForDecode<kBlockSize>(step);
}

/// Describes multiple sequences of decode ops. Each TB executes a sequence of
/// decode steps. The data area starts with a range of instruction numbers for
/// each thread block. The first TB runs from 0 to ends[0]. The nth runs from
/// ends[nth-1] to ends[nth]. After gridDim.x ends, we round to an 8 aligned
/// offset and have an array of GpuDecodes.]
struct alignas(16) GpuDecodeParams {
  // If need to represent more than this many ops, use a dynamically allocated
  // external array in 'external'.
  static constexpr int32_t kMaxInlineOps = 19;

  // Pointer to standalone description of work. If nullptr, the description of
  // work fits inline in 'this'.
  GpuDecodeParams* external{nullptr};
  void* padding;
  // The end of each decode program. The first starts at 0. The end is
  // ends[blockIdx.x].
  int32_t ends
      [kMaxInlineOps * (sizeof(GpuDecode) + sizeof(int32_t)) /
       sizeof(int32_t)] = {};
};

void __global__ __launch_bounds__(1024)
    decodeKernel(GpuDecodeParams inlineParams) {
  __shared__ GpuDecodeParams* params;
  __shared__ int32_t programStart;
  __shared__ int32_t programEnd;
  __shared__ GpuDecode* ops;
  if (threadIdx.x == 0) {
    params = inlineParams.external ? inlineParams.external : &inlineParams;
    programStart = blockIdx.x == 0 ? 0 : params->ends[blockIdx.x - 1];
    programEnd = params->ends[blockIdx.x];
    ops =
        reinterpret_cast<GpuDecode*>(&params->ends[0] + roundUp(gridDim.x, 4));
  }
  __syncthreads();
  for (auto i = programStart; i < programEnd; ++i) {
    detail::decodeSwitch<kBlockSize>(ops[i]);
  }
  __syncthreads();
}

void launchDecode(
    const DecodePrograms& programs,
    LaunchParams& launchParams,
    Stream* stream) {
  int32_t numBlocks = programs.programs.size();
  int32_t numOps = 0;
  bool allSingle = true;
  int32_t shared = 0;
  for (auto& program : programs.programs) {
    int numSteps = program.size();
    ;
    if (numSteps != 1) {
      allSingle = false;
    }
    numOps += numSteps;
    for (auto& step : program) {
      shared = std::max(
          shared, detail::sharedMemorySizeForDecode<kBlockSize>(step->step));
    }
  }
  if (shared > 0) {
    shared += 15; // allow align at 16.
  }
  GpuDecodeParams localParams;
  GpuDecodeParams* params = &localParams;
  char* host = nullptr;
  char* device = nullptr;
  if (numOps > GpuDecodeParams::kMaxInlineOps || allSingle) {
    auto pair = launchParams.setup(
        (numOps + 1) * (sizeof(GpuDecode) + sizeof(int32_t)) + 16);
    host = pair.first;
    device = pair.second;
    uintptr_t aligned = roundUp(reinterpret_cast<uintptr_t>(host), 16);
    params = reinterpret_cast<GpuDecodeParams*>(aligned);
  }
  int32_t end = programs.programs[0].size();
  GpuDecode* decodes =
      reinterpret_cast<GpuDecode*>(&params->ends[0] + roundUp(numBlocks, 4));
  uintptr_t decodeOffset = reinterpret_cast<char*>(decodes) - host;
  int32_t fill = 0;
  for (auto i = 0; i < programs.programs.size(); ++i) {
    params->ends[i] =
        (i == 0 ? 0 : params->ends[i - 1]) + programs.programs[i].size();
    for (auto& op : programs.programs[i]) {
      decodes[fill++] = *op;
    }
  }
  if (allSingle) {
    launchParams.transfer(*stream);
    detail::decodeGlobal<kBlockSize>
        <<<numBlocks, kBlockSize, shared, stream->stream()->stream>>>(
            reinterpret_cast<GpuDecode*>(device + decodeOffset));
    CUDA_CHECK(hipGetLastError());
    programs.result.transfer(*stream);
    return;
  }
  if (launchParams.device) {
    localParams.external = reinterpret_cast<GpuDecodeParams*>(device);
    launchParams.transfer(*stream);
  }

  decodeKernel<<<numBlocks, kBlockSize, shared, stream->stream()->stream>>>(
      localParams);
  CUDA_CHECK(hipGetLastError());
  programs.result.transfer(*stream);
}

REGISTER_KERNEL("decode", decodeKernel);
namespace {
static bool decSingles_reg = registerKernel(
    "decodeSingle",
    reinterpret_cast<const void*>(detail::decodeGlobal<kBlockSize>));
}

} // namespace facebook::velox::wave
