#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <fmt/format.h>
#include <folly/init/Init.h>
#include <gflags/gflags.h>
#include <gtest/gtest.h>
#include "velox/experimental/gpu/Common.h"
#include "velox/experimental/wave/dwio/decode/GpuDecoder.cuh"

DEFINE_int32(device_id, 0, "");
DEFINE_bool(benchmark, false, "");
DEFINE_bool(print_kernels, false, "Print register and smem usage");
DEFINE_bool(use_selective, false, "Use selective path for test");

namespace facebook::velox::wave {
namespace {

using namespace facebook::velox;

// define to use the flexible call path wiht multiple ops per TB
#define USE_PROGRAM_API
#define USE_SEL_BITPACK true

// Returns the number of bytes the "values" will occupy after varint encoding.
uint64_t bulkVarintSize(const uint64_t* values, int count) {
  constexpr uint8_t kLookupSizeTable64[64] = {
      10, 9, 9, 9, 9, 9, 9, 9, 8, 8, 8, 8, 8, 8, 8, 7, 7, 7, 7, 7, 7, 7,
      6,  6, 6, 6, 6, 6, 6, 5, 5, 5, 5, 5, 5, 5, 4, 4, 4, 4, 4, 4, 4, 3,
      3,  3, 3, 3, 3, 3, 2, 2, 2, 2, 2, 2, 2, 1, 1, 1, 1, 1, 1, 1};
  uint64_t size = 0;
  for (int i = 0; i < count; ++i) {
    size += kLookupSizeTable64[__builtin_clzll(values[i] | 1ULL)];
  }
  return size;
}

template <typename T>
void writeVarint(T val, char** pos) noexcept {
  while (val >= 128) {
    *((*pos)++) = 0x80 | (val & 0x7f);
    val >>= 7;
  }
  *((*pos)++) = val;
}

template <typename T>
gpu::CudaPtr<T[]> allocate(int n) {
  T* ptr;
  CUDA_CHECK_FATAL(hipMallocManaged(&ptr, n * sizeof(T)));
  return gpu::CudaPtr<T[]>(ptr);
}

template <typename T>
void fillRandom(T* values, int32_t numValues) {
  uint64_t seed = 0xafbe1647deba879LU;
  for (auto i = 0; i < numValues; ++i) {
    values[i] = seed;
    seed = (seed * 0x5def1) ^ (seed >> 21);
  }
}

// Generate random bits with probability "p" being true and "1 - p" being false.
void fillRandomBits(uint8_t* bits, double p, int numValues) {
  for (int i = 0; i < numValues; ++i) {
    setBit(bits, i, (double)rand() / RAND_MAX < p);
  }
}

template <typename T>
inline T* addBytes(T* ptr, int bytes) {
  return reinterpret_cast<T*>(reinterpret_cast<char*>(ptr) + bytes);
}

template <typename T>
inline const T* addBytes(const T* ptr, int bytes) {
  return reinterpret_cast<const T*>(reinterpret_cast<const char*>(ptr) + bytes);
}

void prefetchToDevice(void* ptr, size_t size) {
  CUDA_CHECK_FATAL(hipMemPrefetchAsync(ptr, size, FLAGS_device_id, nullptr));
}

template <typename T>
void makeBitpackDict(
    int32_t bitWidth,
    int32_t numValues,
    gpu::CudaPtr<char[]>& cudaPtr,
    T*& dict,
    uint64_t*& bits,
    T*& result,
    int32_t** scatter,
    bool bitsOnly,
    BlockStatus*& blockStatus,
    int32_t numBlocks,
    int32_t blockSize) {
  int64_t dictBytes = bitsOnly ? 0 : sizeof(T) << bitWidth;
  int64_t bitBytes = (roundUp(numValues * bitWidth, 128) / 8) + 16;
  int64_t resultBytes = numValues * sizeof(T);
  int scatterBytes =
      scatter ? roundUp(numValues * sizeof(int32_t), sizeof(T)) : 0;
  int32_t statusBytes = sizeof(BlockStatus) * numBlocks;
  if (scatterBytes) {
    resultBytes += resultBytes / 2;
  }
  cudaPtr = allocate<char>(
      dictBytes + bitBytes + scatterBytes + resultBytes + statusBytes);
  T* memory = (T*)cudaPtr.get();

  dict = bitsOnly ? nullptr : memory;

  static int sequence = 1;
  ++sequence;
  for (auto i = 0; i < dictBytes / sizeof(T); ++i) {
    dict[i] = (10 + sequence) * i;
  }

  // The bit packed data does not start at a word boundary.
  bits = addBytes(reinterpret_cast<uint64_t*>(memory), dictBytes + 1);
  fillRandom(bits, bitBytes / 8);

  if (scatterBytes) {
    // Make a scatter vector that makes gaps in the result sequence.
    *scatter =
        addBytes(reinterpret_cast<int32_t*>(memory), dictBytes + bitBytes);
    for (auto i = 0; i < numValues; ++i) {
      (*scatter)[i] = i + i / 4;
    }
  }
  result = addBytes(
      reinterpret_cast<T*>(memory), dictBytes + bitBytes + scatterBytes);
  blockStatus =
      reinterpret_cast<BlockStatus*>(addBytes(result, numValues * sizeof(T)));
  for (auto i = 0; i < numBlocks; ++i) {
    blockStatus[i].numRows =
        i < numBlocks - 1 ? blockSize : numValues - (i * blockSize);
  }
  prefetchToDevice(
      memory, dictBytes + bitBytes + scatterBytes + resultBytes + statusBytes);
}

class GpuDecoderTest : public ::testing::Test {
 protected:
  void SetUp() override {
    if (int device; hipGetDevice(&device) != hipSuccess) {
      GTEST_SKIP() << "No CUDA detected, skipping all tests";
    }
    arena_ = std::make_unique<GpuArena>(100000000, getAllocator(getDevice()));
    CUDA_CHECK_FATAL(hipEventCreate(&startEvent_));
    CUDA_CHECK_FATAL(hipEventCreate(&stopEvent_));
  }

  void TearDown() override {
    CUDA_CHECK_FATAL(hipEventDestroy(startEvent_));
    CUDA_CHECK_FATAL(hipEventDestroy(stopEvent_));
  }

  void testCase(
      const std::string& label,
      std::function<void()> func,
      int64_t bytes,
      int32_t numReps) {
    func();
    CUDA_CHECK_FATAL(hipGetLastError());
    if (!FLAGS_benchmark) {
      CUDA_CHECK_FATAL(hipDeviceSynchronize());
      return;
    }
    CUDA_CHECK_FATAL(hipEventRecord(startEvent_, 0));
    for (auto count = 0; count < numReps; ++count) {
      func();
    }
    CUDA_CHECK_FATAL(hipEventRecord(stopEvent_, 0));
    CUDA_CHECK_FATAL(hipEventSynchronize(stopEvent_));
    float ms;
    CUDA_CHECK_FATAL(hipEventElapsedTime(&ms, startEvent_, stopEvent_));
    printf(
        "%s %.2f (%d at %.2f us each)\n",
        label.c_str(),
        bytes * numReps * 1e-6 / ms,
        numReps,
        ms * 1000 / numReps);
  }

  template <typename T, int32_t kBlockSize>
  void testCopyPlan(int64_t numValues, int numBlocks, bool useScatter) {
    auto source = allocate<T>(numValues);
    auto result = allocate<T>(numValues * 4 / 3);
    gpu::CudaPtr<int32_t[]> scatter;
    if (useScatter) {
      scatter = allocate<int32_t>(numValues);
      for (auto i = 0; i < numValues; ++i) {
        scatter[i] = i * 5 / 4;
      }
    }
    fillRandom(source.get(), numValues);
    result[numValues] = 0xdeadbeef;
    int valuesPerOp = roundUp(numValues / numBlocks, kBlockSize);
    int numOps = roundUp(numValues, valuesPerOp) / valuesPerOp;
    auto ops = allocate<GpuDecode>(numOps);
    for (auto i = 0; i < numOps; ++i) {
      int32_t begin = i * valuesPerOp;
      ops[i].step = DecodeStep::kTrivial;
      auto& op = ops[i].data.trivial;
      op.dataType = WaveTypeTrait<T>::typeKind;
      op.begin = begin;
      op.end = std::min<int32_t>(numValues, (i + 1) * valuesPerOp);
      op.result = result.get();
      op.input = source.get();
      op.scatter = scatter.get();
    }
    testCase(
        fmt::format(
            "copy plan {} numValues={} useScatter={}",
            sizeof(T) * 8,
            numValues,
            useScatter),
        [&] { decodeGlobal<kBlockSize>(ops.get(), numOps); },
        numValues * sizeof(T),
        10);
    if (!scatter) {
      EXPECT_EQ(0xdeadbeef, result[numValues]);
    }
    for (auto i = 0; i < numValues; ++i) {
      ASSERT_EQ(source[i], result[scatter ? scatter[i] : i]);
    }
  }

  template <typename T, int kBlockSize>
  void dictTestPlan(
      int32_t bitWidth,
      int64_t numValues,
      int numBlocks,
      bool useScatter,
      bool bitsOnly = false,
      bool useSelective = false) {
    gpu::CudaPtr<char[]> ptr;
    T* dict;
    uint64_t* bits;
    T* result;
    int32_t* scatter = nullptr;
    BlockStatus* blockStatus;
    makeBitpackDict(
        bitWidth,
        numValues,
        ptr,
        dict,
        bits,
        result,
        useScatter ? &scatter : nullptr,
        bitsOnly,
        blockStatus,
        roundUp(numValues, kBlockSize) / kBlockSize,
        kBlockSize);
    result[numValues] = 0xdeadbeef;
    int valuesPerOp = roundUp(numValues / numBlocks, kBlockSize);
    int numOps = roundUp(numValues, valuesPerOp) / valuesPerOp;
    auto valuesPerThread = valuesPerOp / kBlockSize;
    auto ops = allocate<GpuDecode>(numOps);
    for (auto i = 0; i < numOps; ++i) {
      int32_t begin = i * valuesPerOp;
      ops[i].step = useSelective ? (sizeof(T) == 8 ? DecodeStep::kSelective64
                                                   : DecodeStep::kSelective32)
                                 : DecodeStep::kDictionaryOnBitpack;
      ops[i].encoding = DecodeStep::kDictionaryOnBitpack;
      ops[i].dataType = WaveTypeTrait<T>::typeKind;
      ops[i].nullMode = NullMode::kDenseNonNull;
      ops[i].nthBlock = i;
      ops[i].numRowsPerThread = i == numOps - 1
          ? roundUp(numValues - (valuesPerOp * i), kBlockSize) / kBlockSize
          : valuesPerThread;
      ops[i].baseRow = i * valuesPerOp;
      ops[i].maxRow = std::min<int32_t>((i + 1) * valuesPerOp, numValues);
      ops[i].result = reinterpret_cast<T*>(result) + i * valuesPerOp;

      ops[i].blockStatus = blockStatus + (i * valuesPerThread);
      auto& op = ops[i].data.dictionaryOnBitpack;
      op.begin = begin;
      op.end = std::min<int>(numValues, (i + 1) * valuesPerOp);
      op.result = result;
      op.bitWidth = bitWidth;
      op.indices = bits;
      op.alphabet = dict;
      op.scatter = scatter;
      op.baseline = 0;
      op.dataType = WaveTypeTrait<T>::typeKind;
    }
    testCase(
        fmt::format(
            "bitpack dictplan {} -> {} numValues={} useScatter={}",
            bitWidth,
            sizeof(T) * 8,
            numValues,
            useScatter),
        [&] {
#ifdef USE_PROGRAM_API
          callViaPrograms(ops.get(), numOps);
#else
          decodeGlobal<kBlockSize>(ops.get(), numOps);
#endif
        },
        numValues * sizeof(T),
        10);
    if (!scatter) {
      EXPECT_EQ(0xdeadbeef, result[numValues]);
    }
    auto mask = (1uL << bitWidth) - 1;
    for (auto i = 0; i < numValues; ++i) {
      int32_t bit = i * bitWidth;
      uint64_t word = *addBytes(bits, bit / 8);
      uint64_t index = (word >> (bit & 7)) & mask;
      T expected = bitsOnly ? index : dict[index];
      ASSERT_EQ(result[scatter ? scatter[i] : i], expected) << i;
    }
  }

  template <int kBlockSize>
  void testSparseBool(int numValues, int numBlocks) {
    auto expected = allocate<uint8_t>((numValues + 7) / 8);
    fillRandomBits(expected.get(), 0.99, numValues);
    auto indices = allocate<int32_t>(numValues);
    int indicesCount = 0;
    for (int i = 0; i < numValues; ++i) {
      if (!isSet(expected.get(), i)) {
        indices[indicesCount++] = i;
      }
    }
    auto bools = allocate<bool>(numValues * numBlocks);
    auto resultSize = (numValues + 7) / 8;
    auto result = allocate<uint8_t>(resultSize * numBlocks);
    auto ops = allocate<GpuDecode>(numBlocks);
    for (int i = 0; i < numBlocks; ++i) {
      ops[i].step = DecodeStep::kSparseBool;
      auto& op = ops[i].data.sparseBool;
      op.totalCount = numValues;
      op.sparseValue = false;
      op.sparseIndices = indices.get();
      op.sparseCount = indicesCount;
      op.bools = bools.get() + i * numValues;
      op.result = result.get() + i * resultSize;
    }
    testCase(
        "",
        [&] { decodeGlobal<kBlockSize>(ops.get(), numBlocks); },
        resultSize * numBlocks,
        3);
    for (int j = 0; j < numBlocks; ++j) {
      auto* actual = ops[j].data.sparseBool.result;
      for (int i = 0; i < numValues; ++i) {
        ASSERT_EQ(isSet(actual, i), isSet(expected.get(), i)) << i;
      }
    }
  }

  template <int kBlockSize>
  void testVarint(int numValues, int numBlocks) {
    std::vector<uint64_t> expected(numValues);
    fillRandom(expected.data(), numValues);
    for (int i = 0; i < numValues; ++i) {
      if (i % 100 != 0) {
        expected[i] %= 128;
      }
    }
    auto inputSize = bulkVarintSize(expected.data(), numValues);
    auto input = allocate<char>(inputSize);
    auto* rawInput = input.get();
    for (int i = 0; i < numValues; ++i) {
      writeVarint(expected[i], &rawInput);
    }
    auto ends = allocate<bool>(inputSize * numBlocks);
    auto endPos = allocate<int32_t>(inputSize * numBlocks);
    auto result = allocate<uint64_t>(inputSize * numBlocks);
    auto ops = allocate<GpuDecode>(numBlocks);
    for (int i = 0; i < numBlocks; ++i) {
      ops[i].step = DecodeStep::kVarint;
      auto& op = ops[i].data.varint;
      op.input = input.get();
      op.size = inputSize;
      op.ends = ends.get() + i * inputSize;
      op.endPos = endPos.get() + i * inputSize;
      op.resultType = WaveTypeKind::BIGINT;
      op.result = result.get() + i * inputSize;
    }
    testCase(
        "",
        [&] { decodeGlobal<kBlockSize>(ops.get(), numBlocks); },
        numValues * sizeof(uint64_t) * numBlocks,
        3);
    for (int j = 0; j < numBlocks; ++j) {
      auto& op = ops[j].data.varint;
      ASSERT_EQ(op.resultSize, numValues);
      for (int i = 0; i < numValues; ++i) {
        ASSERT_EQ(reinterpret_cast<const uint64_t*>(op.result)[i], expected[i]);
      }
    }
  }

  template <typename T, int kBlockSize>
  void testMainlyConstant(int numValues, int numBlocks) {
    auto isCommon = allocate<uint8_t>((numValues + 7) / 8);
    fillRandomBits(isCommon.get(), 0.99, numValues);
    auto values = allocate<T>(numValues + 1);
    fillRandom(values.get(), numValues + 1);
    auto otherIndices = allocate<int32_t>(numValues * numBlocks);
    auto result = allocate<T>(numValues * numBlocks);
    auto otherCounts = allocate<int32_t>(numBlocks);
    auto ops = allocate<GpuDecode>(numBlocks);
    for (int i = 0; i < numBlocks; ++i) {
      ops[i].step = DecodeStep::kMainlyConstant;
      auto& op = ops[i].data.mainlyConstant;
      op.dataType = WaveTypeTrait<T>::typeKind;
      op.count = numValues;
      op.commonValue = &values[numValues];
      op.otherValues = values.get();
      op.isCommon = isCommon.get();
      op.otherIndices = otherIndices.get() + i * numValues;
      op.result = result.get() + i * numValues;
      op.otherCount = otherCounts.get() + i;
    }
    testCase(
        "",
        [&] { decodeGlobal<kBlockSize>(ops.get(), numBlocks); },
        numValues * numBlocks * sizeof(T),
        3);
    for (int k = 0; k < numBlocks; ++k) {
      auto& op = ops[k].data.mainlyConstant;
      auto* result = (const T*)op.result;
      int j = 0;
      for (int i = 0; i < numValues; ++i) {
        if (isSet(isCommon.get(), i)) {
          ASSERT_EQ(result[i], values[numValues]);
        } else {
          ASSERT_EQ(result[i], values[j++]);
        }
      }
      ASSERT_EQ(*op.otherCount, j);
    }
  }

  template <int kBlockSize>
  void testRleTotalLength(int numValues, int numBlocks) {
    auto values = allocate<int32_t>(numValues);
    fillRandom(values.get(), numValues);
    int valuesPerOp = (numValues + numBlocks - 1) / numBlocks;
    auto ops = allocate<GpuDecode>(numBlocks);
    auto lengths = allocate<int64_t>(numBlocks);
    for (auto i = 0; i < numBlocks; ++i) {
      ops[i].step = DecodeStep::kRleTotalLength;
      auto& op = ops[i].data.rleTotalLength;
      op.input = values.get() + i * valuesPerOp;
      op.count = std::min(valuesPerOp, numValues - i * valuesPerOp);
      op.result = &lengths[i];
    }
    testCase(
        "",
        [&] { decodeGlobal<kBlockSize>(ops.get(), numBlocks); },
        numValues * sizeof(int32_t),
        5);
    for (int i = 0; i < numBlocks; ++i) {
      auto& op = ops[i].data.rleTotalLength;
      int64_t expected = 0;
      for (int j = 0; j < op.count; ++j) {
        expected += op.input[j];
      }
      ASSERT_EQ(*op.result, expected);
    }
  }

  template <typename T, int kBlockSize>
  void testRle(int numValues, int numBlocks) {
    auto values = allocate<T>(numValues);
    auto lengths = allocate<int32_t>(numValues);
    int totalLength = 0;
    fillRandom(values.get(), numValues);
    fillRandom(lengths.get(), numValues);
    for (int i = 0; i < numValues; ++i) {
      int limit = i % 1000 == 0 ? 1000 : 10;
      lengths[i] = (uint32_t)lengths[i] % limit;
      totalLength += lengths[i];
    }
    auto ops = allocate<GpuDecode>(numBlocks);
    auto results = allocate<int64_t>(numBlocks);
    int valuesPerOp = (numValues + numBlocks - 1) / numBlocks;
    for (int i = 0; i < numBlocks; ++i) {
      ops[i].step = DecodeStep::kRleTotalLength;
      auto& op = ops[i].data.rleTotalLength;
      op.input = lengths.get() + i * valuesPerOp;
      op.count = std::min(valuesPerOp, numValues - i * valuesPerOp);
      op.result = &results[i];
    }
    decodeGlobal<kBlockSize>(ops.get(), numBlocks);
    CUDA_CHECK_FATAL(hipGetLastError());
    CUDA_CHECK_FATAL(hipDeviceSynchronize());
    auto result = allocate<T>(totalLength);
    int lengthSofar = 0;
    for (int i = 0; i < numBlocks; ++i) {
      int subtotal = *ops[i].data.rleTotalLength.result;
      ops[i].step = DecodeStep::kRle;
      auto& op = ops[i].data.rle;
      op.valueType = WaveTypeTrait<T>::typeKind;
      op.values = values.get() + i * valuesPerOp;
      op.lengths = lengths.get() + i * valuesPerOp;
      op.count = std::min(valuesPerOp, numValues - i * valuesPerOp);
      op.result = result.get() + lengthSofar;
      lengthSofar += subtotal;
    }
    testCase(
        "",
        [&] { decodeGlobal<kBlockSize>(ops.get(), numBlocks); },
        totalLength * sizeof(T),
        3);
    for (int i = 0; i < numBlocks; ++i) {
    }
    for (int i = 0, j = 0; i < numValues; ++i) {
      for (int k = 0; k < lengths[i]; ++k) {
        ASSERT_EQ(result[j++], values[i]);
      }
    }
  }

  template <int kBlockSize>
  void testMakeScatterIndices(int numValues, int numBlocks) {
    auto bits = allocate<uint8_t>((numValues * numBlocks + 7) / 8);
    fillRandomBits(bits.get(), 0.5, numValues * numBlocks);
    auto indices = allocate<int32_t>(numValues * numBlocks);
    auto indicesCounts = allocate<int32_t>(numBlocks);
    auto ops = allocate<GpuDecode>(numBlocks);
    for (int i = 0; i < numBlocks; ++i) {
      ops[i].step = DecodeStep::kMakeScatterIndices;
      auto& op = ops[i].data.makeScatterIndices;
      op.bits = bits.get();
      op.findSetBits = true;
      op.begin = i * numValues;
      op.end = op.begin + numValues;
      op.indices = indices.get() + i * numValues;
      op.indicesCount = indicesCounts.get() + i;
    }
    testCase(
        "",
        [&] { decodeGlobal<kBlockSize>(ops.get(), numBlocks); },
        numValues * numBlocks * sizeof(int32_t),
        3);
    for (int i = 0; i < numBlocks; ++i) {
      auto& op = ops[i].data.makeScatterIndices;
      int k = 0;
      for (int j = 0; j < numValues; ++j) {
        if (isSet(bits.get(), j + i * numValues)) {
          ASSERT_LT(k, *op.indicesCount);
          ASSERT_EQ(op.indices[k++], j);
        }
      }
      ASSERT_EQ(k, *op.indicesCount);
    }
  }

  void callViaPrograms(GpuDecode* ops, int32_t numOps) {
    auto stream = std::make_unique<Stream>();
    LaunchParams params(*arena_);
    DecodePrograms programs;
    for (int i = 0; i < numOps; ++i) {
      programs.programs.emplace_back();
      programs.programs.back().push_back(std::make_unique<GpuDecode>());
      auto opPtr = programs.programs.back().front().get();
      *opPtr = ops[i];
    }
    launchDecode(programs, params, stream.get());
    stream->wait();
  }

  void testMakeScatterIndicesStream(int numValues, int numBlocks) {
    auto bits = allocate<uint8_t>((numValues * numBlocks + 7) / 8);
    fillRandomBits(bits.get(), 0.5, numValues * numBlocks);
    auto indices = allocate<int32_t>(numValues * numBlocks);
    auto indicesCounts = allocate<int32_t>(numBlocks);
    DecodePrograms programs;
    for (int i = 0; i < numBlocks; ++i) {
      programs.programs.emplace_back();
      programs.programs.back().push_back(std::make_unique<GpuDecode>());
      auto opPtr = programs.programs.back().front().get();
      opPtr->step = DecodeStep::kMakeScatterIndices;
      auto& op = opPtr->data.makeScatterIndices;
      op.bits = bits.get();
      op.findSetBits = true;
      op.begin = i * numValues;
      op.end = op.begin + numValues;
      op.indices = indices.get() + i * numValues;
      op.indicesCount = indicesCounts.get() + i;
    }
    auto stream = std::make_unique<Stream>();
    LaunchParams params(*arena_);
    launchDecode(programs, params, stream.get());
    stream->wait();
    for (int i = 0; i < numBlocks; ++i) {
      auto& op = programs.programs[i].front()->data.makeScatterIndices;
      int k = 0;
      for (int j = 0; j < numValues; ++j) {
        if (isSet(bits.get(), j + i * numValues)) {
          ASSERT_LT(k, *op.indicesCount);
          ASSERT_EQ(op.indices[k++], j);
        }
      }
      ASSERT_EQ(k, *op.indicesCount);
    }
  }

  void testCountBits(int32_t numWords, int32_t stride) {
    auto bits = allocate<uint8_t>(numWords * 8);
    fillRandomBits(bits.get(), 0.5, numWords * 64);
    auto result = allocate<int32_t>(numWords * 64 / stride);
    // One int per warp.
    auto temp = allocate<int32_t>(8);
    DecodePrograms programs;
    programs.programs.emplace_back();
    programs.programs.back().push_back(std::make_unique<GpuDecode>());
    auto opPtr = programs.programs.back().front().get();
    opPtr->step = DecodeStep::kCountBits;
    auto& op = opPtr->data.countBits;
    opPtr->temp = temp.get();
    op.bits = bits.get();
    op.numBits = numWords * 64;
    op.resultStride = stride;
    opPtr->result = result.get();
    auto stream = std::make_unique<Stream>();
    LaunchParams params(*arena_);
    launchDecode(programs, params, stream.get());
    stream->wait();
    auto numResults = ((numWords * 64) - 1) / stride;
    auto* rawResult = result.get();
    int32_t count = 0;
    for (auto i = 0; i < numResults; ++i) {
      for (auto j = 0; j < stride / 64; j++) {
        count += __builtin_popcountl(
            reinterpret_cast<const uint64_t*>(op.bits)[i * (stride / 64) + j]);
      }
      EXPECT_EQ(count, rawResult[i]);
    }
  }

 private:
  std::unique_ptr<GpuArena> arena_;

  hipEvent_t startEvent_;
  hipEvent_t stopEvent_;
};

TEST_F(GpuDecoderTest, trivial) {
  testCopyPlan<uint64_t, 128>(40'000'003, 1024, false);
  testCopyPlan<uint64_t, 128>(40'000'003, 1024, true);
}

TEST_F(GpuDecoderTest, dictionaryOnBitpack) {
  dictTestPlan<int32_t, 256>(11, 4'000'037, 1024, false);
  dictTestPlan<int64_t, 256>(11, 4'000'037, 1024, false);
  dictTestPlan<int32_t, 256>(11, 40'000'003, 1024, false);
  dictTestPlan<int64_t, 256>(11, 40'000'003, 1024, false);
  dictTestPlan<int64_t, 256>(11, 40'000'003, 1024, true);
}

TEST_F(GpuDecoderTest, bitpack) {
  bool useSelective = FLAGS_use_selective;
  dictTestPlan<int32_t, 256>(27, 4000001, 1024, false, true, useSelective);
  dictTestPlan<int64_t, 256>(28, 4'000'037, 1024, false, true, useSelective);
  dictTestPlan<int32_t, 256>(26, 40'000'003, 1024, false, true, useSelective);
  dictTestPlan<int64_t, 256>(30, 40'000'003, 1024, false, true, useSelective);
  dictTestPlan<int64_t, 256>(47, 40'000'003, 1024, false, true, useSelective);
  dictTestPlan<int64_t, 256>(22, 40'000'003, 1024, true, true, false);
}

TEST_F(GpuDecoderTest, sparseBool) {
  testSparseBool<256>(40013, 1024);
}

TEST_F(GpuDecoderTest, varint) {
  testVarint<256>(4001, 1024);
}

TEST_F(GpuDecoderTest, mainlyConstant) {
  testMainlyConstant<int64_t, 256>(40013, 1024);
}

TEST_F(GpuDecoderTest, rleTotalLength) {
  testRleTotalLength<256>(40'000'003, 1024);
}

TEST_F(GpuDecoderTest, rle) {
  testRle<int64_t, 256>(40'000'003, 1024);
}

TEST_F(GpuDecoderTest, makeScatterIndices) {
  testMakeScatterIndices<256>(40013, 1024);
}

TEST_F(GpuDecoderTest, countBits) {
  testCountBits(10000, 256);
  testCountBits(20000, 512);
  testCountBits(30000, 1024);
  testCountBits(100000, 2048);
}

TEST_F(GpuDecoderTest, streamApi) {
  //  One call with few blocks, another with many, to cover inlined and out of
  //  line params.
  testMakeScatterIndicesStream(100, 20);
  testMakeScatterIndicesStream(999, 999);
}
} // namespace
} // namespace facebook::velox::wave

void printFuncAttrs(
    const std::string& heading,
    const hipFuncAttributes& attrs) {
  std::cout << heading << " sharedSizeBytes=" << attrs.sharedSizeBytes
            << " constSizeBytes" << attrs.constSizeBytes
            << " localSizeBytes =" << attrs.localSizeBytes
            << "maxThreadsPerBlock=" << attrs.maxThreadsPerBlock
            << " numRegs=" << attrs.numRegs
            << " maxDynamicSharedSizeBytes=" << attrs.maxDynamicSharedSizeBytes
            << std::endl;
}
using namespace facebook::velox::wave;

int main(int argc, char** argv) {
  testing::InitGoogleTest(&argc, argv);
  folly::Init init{&argc, &argv};

  if (int device; hipGetDevice(&device) != hipSuccess) {
    std::cerr << "No CUDA detected, skipping all tests" << std::endl;
    return 0;
  }

  hipDeviceProp_t prop;
  CUDA_CHECK_FATAL(hipGetDeviceProperties(&prop, FLAGS_device_id));
  printf("Running on device: %s\n", prop.name);
  CUDA_CHECK_FATAL(hipSetDevice(FLAGS_device_id));
  if (FLAGS_print_kernels) {
    hipFuncAttributes attrs;
    CUDA_CHECK_FATAL(hipFuncGetAttributes(&attrs, reinterpret_cast<const void*>(detail::decodeGlobal<128>)));
    printFuncAttrs("decode blocksize 128", attrs);
    CUDA_CHECK_FATAL(hipFuncGetAttributes(&attrs, reinterpret_cast<const void*>(detail::decodeGlobal<256>)));
    printFuncAttrs("decode blocksize 256", attrs);
    CUDA_CHECK_FATAL(hipFuncGetAttributes(&attrs, reinterpret_cast<const void*>(detail::decodeGlobal<512>)));
    printFuncAttrs("decode blocksize 512", attrs);
    CUDA_CHECK_FATAL(hipFuncGetAttributes(&attrs, reinterpret_cast<const void*>(detail::decodeGlobal<1024>)));
    printFuncAttrs("decode blocksize 1024", attrs);
    printFuncAttrs("decode2", attrs);

    printKernels();
  }
  return RUN_ALL_TESTS();
}
