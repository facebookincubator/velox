#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <fmt/format.h>
#include <folly/init/Init.h>
#include <gflags/gflags.h>
#include <gtest/gtest.h>
#include "velox/experimental/gpu/Common.h"
#include "velox/experimental/wave/dwio/decode/GpuDecoder.cuh"

DEFINE_int32(device_id, 0, "");
DEFINE_bool(benchmark, false, "");

namespace facebook::velox::wave {
namespace {

using namespace facebook::velox;

// Returns the number of bytes the "values" will occupy after varint encoding.
uint64_t bulkVarintSize(const uint64_t* values, int count) {
  constexpr uint8_t kLookupSizeTable64[64] = {
      10, 9, 9, 9, 9, 9, 9, 9, 8, 8, 8, 8, 8, 8, 8, 7, 7, 7, 7, 7, 7, 7,
      6,  6, 6, 6, 6, 6, 6, 5, 5, 5, 5, 5, 5, 5, 4, 4, 4, 4, 4, 4, 4, 3,
      3,  3, 3, 3, 3, 3, 2, 2, 2, 2, 2, 2, 2, 1, 1, 1, 1, 1, 1, 1};
  uint64_t size = 0;
  for (int i = 0; i < count; ++i) {
    size += kLookupSizeTable64[__builtin_clzll(values[i] | 1ULL)];
  }
  return size;
}

template <typename T>
void writeVarint(T val, char** pos) noexcept {
  while (val >= 128) {
    *((*pos)++) = 0x80 | (val & 0x7f);
    val >>= 7;
  }
  *((*pos)++) = val;
}

template <typename T>
gpu::CudaPtr<T[]> allocate(int n) {
  T* ptr;
  CUDA_CHECK_FATAL(hipMallocManaged(&ptr, n * sizeof(T)));
  return gpu::CudaPtr<T[]>(ptr);
}

template <typename T>
void fillRandom(T* values, int32_t numValues) {
  uint64_t seed = 0xafbe1647deba879LU;
  for (auto i = 0; i < numValues; ++i) {
    values[i] = seed;
    seed = (seed * 0x5def1) ^ (seed >> 21);
  }
}

// Generate random bits with probability "p" being true and "1 - p" being false.
void fillRandomBits(uint8_t* bits, double p, int numValues) {
  for (int i = 0; i < numValues; ++i) {
    setBit(bits, i, (double)rand() / RAND_MAX < p);
  }
}

template <typename T>
inline T* addBytes(T* ptr, int bytes) {
  return reinterpret_cast<T*>(reinterpret_cast<char*>(ptr) + bytes);
}

template <typename T>
inline const T* addBytes(const T* ptr, int bytes) {
  return reinterpret_cast<const T*>(reinterpret_cast<const char*>(ptr) + bytes);
}

template <typename T>
void makeBitpackDict(
    int32_t bitWidth,
    int32_t numValues,
    gpu::CudaPtr<char[]>& cudaPtr,
    T*& dict,
    uint64_t*& bits,
    T*& result,
    int32_t** scatter) {
  int64_t dictBytes = sizeof(T) << bitWidth;
  int64_t bitBytes = (roundUp(numValues * bitWidth, 128) / 8) + 16;
  int64_t resultBytes = numValues * sizeof(T);
  int scatterBytes =
      scatter ? roundUp(numValues * sizeof(int32_t), sizeof(T)) : 0;
  if (scatterBytes) {
    resultBytes += resultBytes / 2;
  }
  cudaPtr = allocate<char>(dictBytes + bitBytes + scatterBytes + resultBytes);
  T* memory = (T*)cudaPtr.get();

  dict = memory;

  static int sequence = 1;
  ++sequence;
  for (auto i = 0; i < dictBytes / sizeof(T); ++i) {
    dict[i] = (10 + sequence) * i;
  }

  // The bit packed data does not start at a word boundary.
  bits = addBytes(reinterpret_cast<uint64_t*>(memory), dictBytes + 1);
  fillRandom(bits, bitBytes / 8);

  if (scatterBytes) {
    // Make a scatter vector that makes gaps in the result sequence.
    *scatter =
        addBytes(reinterpret_cast<int32_t*>(memory), dictBytes + bitBytes);
    for (auto i = 0; i < numValues; ++i) {
      (*scatter)[i] = i + i / 4;
    }
  }
  result = addBytes(
      reinterpret_cast<T*>(memory), dictBytes + bitBytes + scatterBytes);
}

class GpuDecoderTest : public ::testing::Test {
 protected:
  void SetUp() override {
    CUDA_CHECK_FATAL(hipEventCreate(&startEvent_));
    CUDA_CHECK_FATAL(hipEventCreate(&stopEvent_));
  }

  void TearDown() override {
    CUDA_CHECK_FATAL(hipEventDestroy(startEvent_));
    CUDA_CHECK_FATAL(hipEventDestroy(stopEvent_));
  }

  void testCase(
      const std::string& label,
      std::function<void()> func,
      int64_t bytes,
      int32_t numReps) {
    func();
    CUDA_CHECK_FATAL(hipGetLastError());
    if (!FLAGS_benchmark) {
      CUDA_CHECK_FATAL(hipDeviceSynchronize());
      return;
    }
    CUDA_CHECK_FATAL(hipEventRecord(startEvent_, 0));
    for (auto count = 0; count < numReps; ++count) {
      func();
    }
    CUDA_CHECK_FATAL(hipEventRecord(stopEvent_, 0));
    CUDA_CHECK_FATAL(hipEventSynchronize(stopEvent_));
    float ms;
    CUDA_CHECK_FATAL(hipEventElapsedTime(&ms, startEvent_, stopEvent_));
    printf(
        "%s %.2f (%d at %.2f us each)\n",
        label.c_str(),
        bytes * numReps * 1e-6 / ms,
        numReps,
        ms * 1000 / numReps);
  }

  template <typename T, int32_t kBlockSize>
  void testCopyPlan(int64_t numValues, int numBlocks, bool useScatter) {
    auto source = allocate<T>(numValues);
    auto result = allocate<T>(numValues * 4 / 3);
    gpu::CudaPtr<int32_t[]> scatter;
    if (useScatter) {
      scatter = allocate<int32_t>(numValues);
      for (auto i = 0; i < numValues; ++i) {
        scatter[i] = i * 5 / 4;
      }
    }
    fillRandom(source.get(), numValues);
    result[numValues] = 0xdeadbeef;
    int valuesPerOp = roundUp(numValues / numBlocks, kBlockSize);
    int numOps = roundUp(numValues, valuesPerOp) / valuesPerOp;
    auto ops = allocate<GpuDecode>(numOps);
    for (auto i = 0; i < numOps; ++i) {
      int32_t begin = i * valuesPerOp;
      ops[i].step = DecodeStep::kTrivial;
      auto& op = ops[i].data.trivial;
      op.dataType = WaveTypeTrait<T>::typeKind;
      op.begin = begin;
      op.end = std::min<int32_t>(numValues, (i + 1) * valuesPerOp);
      op.result = result.get();
      op.input = source.get();
      op.scatter = scatter.get();
    }
    testCase(
        fmt::format(
            "copy plan {} numValues={} useScatter={}",
            sizeof(T) * 8,
            numValues,
            useScatter),
        [&] { decodeGlobal<kBlockSize>(ops.get(), numOps); },
        numValues * sizeof(T),
        10);
    if (!scatter) {
      EXPECT_EQ(0xdeadbeef, result[numValues]);
    }
    for (auto i = 0; i < numValues; ++i) {
      ASSERT_EQ(source[i], result[scatter ? scatter[i] : i]);
    }
  }

  template <typename T, int kBlockSize>
  void dictTestPlan(
      int32_t bitWidth,
      int64_t numValues,
      int numBlocks,
      bool useScatter) {
    gpu::CudaPtr<char[]> ptr;
    T* dict;
    uint64_t* bits;
    T* result;
    int32_t* scatter = nullptr;
    makeBitpackDict(
        bitWidth,
        numValues,
        ptr,
        dict,
        bits,
        result,
        useScatter ? &scatter : nullptr);
    result[numValues] = 0xdeadbeef;
    int valuesPerOp = roundUp(numValues / numBlocks, kBlockSize);
    int numOps = roundUp(numValues, valuesPerOp) / valuesPerOp;
    auto ops = allocate<GpuDecode>(numOps);
    for (auto i = 0; i < numOps; ++i) {
      int32_t begin = i * valuesPerOp;
      ops[i].step = DecodeStep::kDictionaryOnBitpack;
      auto& op = ops[i].data.dictionaryOnBitpack;
      op.begin = begin;
      op.end = std::min<int>(numValues, (i + 1) * valuesPerOp);
      op.result = result;
      op.bitWidth = bitWidth;
      op.indices = bits;
      op.alphabet = dict;
      op.scatter = scatter;
      op.baseline = 0;
      op.dataType = WaveTypeTrait<T>::typeKind;
    }
    testCase(
        fmt::format(
            "bitpack dictplan {} numValues={} useScatter={}",
            sizeof(T) * 8,
            numValues,
            useScatter),
        [&] { decodeGlobal<kBlockSize>(ops.get(), numOps); },
        numValues * sizeof(T),
        10);
    if (!scatter) {
      EXPECT_EQ(0xdeadbeef, result[numValues]);
    }
    auto mask = (1u << bitWidth) - 1;
    for (auto i = 0; i < numValues; ++i) {
      int32_t bit = i * bitWidth;
      uint64_t word = *addBytes(bits, bit / 8);
      T expected = dict[(word >> (bit & 7)) & mask];
      ASSERT_EQ(result[scatter ? scatter[i] : i], expected) << i;
    }
  }

  template <int kBlockSize>
  void testSparseBool(int numValues, int numBlocks) {
    auto expected = allocate<uint8_t>((numValues + 7) / 8);
    fillRandomBits(expected.get(), 0.99, numValues);
    auto indices = allocate<int32_t>(numValues);
    int indicesCount = 0;
    for (int i = 0; i < numValues; ++i) {
      if (!isSet(expected.get(), i)) {
        indices[indicesCount++] = i;
      }
    }
    auto bools = allocate<bool>(numValues * numBlocks);
    auto resultSize = (numValues + 7) / 8;
    auto result = allocate<uint8_t>(resultSize * numBlocks);
    auto ops = allocate<GpuDecode>(numBlocks);
    for (int i = 0; i < numBlocks; ++i) {
      ops[i].step = DecodeStep::kSparseBool;
      auto& op = ops[i].data.sparseBool;
      op.totalCount = numValues;
      op.sparseValue = false;
      op.sparseIndices = indices.get();
      op.sparseCount = indicesCount;
      op.bools = bools.get() + i * numValues;
      op.result = result.get() + i * resultSize;
    }
    testCase(
        "",
        [&] { decodeGlobal<kBlockSize>(ops.get(), numBlocks); },
        resultSize * numBlocks,
        3);
    for (int j = 0; j < numBlocks; ++j) {
      auto* actual = ops[j].data.sparseBool.result;
      for (int i = 0; i < numValues; ++i) {
        ASSERT_EQ(isSet(actual, i), isSet(expected.get(), i)) << i;
      }
    }
  }

  template <int kBlockSize>
  void testVarint(int numValues, int numBlocks) {
    std::vector<uint64_t> expected(numValues);
    fillRandom(expected.data(), numValues);
    for (int i = 0; i < numValues; ++i) {
      if (i % 100 != 0) {
        expected[i] %= 128;
      }
    }
    auto inputSize = bulkVarintSize(expected.data(), numValues);
    auto input = allocate<char>(inputSize);
    auto* rawInput = input.get();
    for (int i = 0; i < numValues; ++i) {
      writeVarint(expected[i], &rawInput);
    }
    auto ends = allocate<bool>(inputSize * numBlocks);
    auto endPos = allocate<int32_t>(inputSize * numBlocks);
    auto result = allocate<uint64_t>(inputSize * numBlocks);
    auto ops = allocate<GpuDecode>(numBlocks);
    for (int i = 0; i < numBlocks; ++i) {
      ops[i].step = DecodeStep::kVarint;
      auto& op = ops[i].data.varint;
      op.input = input.get();
      op.size = inputSize;
      op.ends = ends.get() + i * inputSize;
      op.endPos = endPos.get() + i * inputSize;
      op.resultType = WaveTypeKind::BIGINT;
      op.result = result.get() + i * inputSize;
    }
    testCase(
        "",
        [&] { decodeGlobal<kBlockSize>(ops.get(), numBlocks); },
        numValues * sizeof(uint64_t) * numBlocks,
        3);
    for (int j = 0; j < numBlocks; ++j) {
      auto& op = ops[j].data.varint;
      ASSERT_EQ(op.resultSize, numValues);
      for (int i = 0; i < numValues; ++i) {
        ASSERT_EQ(reinterpret_cast<const uint64_t*>(op.result)[i], expected[i]);
      }
    }
  }

  template <typename T, int kBlockSize>
  void testMainlyConstant(int numValues, int numBlocks) {
    auto isCommon = allocate<uint8_t>((numValues + 7) / 8);
    fillRandomBits(isCommon.get(), 0.99, numValues);
    auto values = allocate<T>(numValues + 1);
    fillRandom(values.get(), numValues + 1);
    auto otherIndices = allocate<int32_t>(numValues * numBlocks);
    auto result = allocate<T>(numValues * numBlocks);
    auto otherCounts = allocate<int32_t>(numBlocks);
    auto ops = allocate<GpuDecode>(numBlocks);
    for (int i = 0; i < numBlocks; ++i) {
      ops[i].step = DecodeStep::kMainlyConstant;
      auto& op = ops[i].data.mainlyConstant;
      op.dataType = WaveTypeTrait<T>::typeKind;
      op.count = numValues;
      op.commonValue = &values[numValues];
      op.otherValues = values.get();
      op.isCommon = isCommon.get();
      op.otherIndices = otherIndices.get() + i * numValues;
      op.result = result.get() + i * numValues;
      op.otherCount = otherCounts.get() + i;
    }
    testCase(
        "",
        [&] { decodeGlobal<kBlockSize>(ops.get(), numBlocks); },
        numValues * numBlocks * sizeof(T),
        3);
    for (int k = 0; k < numBlocks; ++k) {
      auto& op = ops[k].data.mainlyConstant;
      auto* result = (const T*)op.result;
      int j = 0;
      for (int i = 0; i < numValues; ++i) {
        if (isSet(isCommon.get(), i)) {
          ASSERT_EQ(result[i], values[numValues]);
        } else {
          ASSERT_EQ(result[i], values[j++]);
        }
      }
      ASSERT_EQ(*op.otherCount, j);
    }
  }

  template <int kBlockSize>
  void testRleTotalLength(int numValues, int numBlocks) {
    auto values = allocate<int32_t>(numValues);
    fillRandom(values.get(), numValues);
    int valuesPerOp = (numValues + numBlocks - 1) / numBlocks;
    auto ops = allocate<GpuDecode>(numBlocks);
    auto lengths = allocate<int64_t>(numBlocks);
    for (auto i = 0; i < numBlocks; ++i) {
      ops[i].step = DecodeStep::kRleTotalLength;
      auto& op = ops[i].data.rleTotalLength;
      op.input = values.get() + i * valuesPerOp;
      op.count = std::min(valuesPerOp, numValues - i * valuesPerOp);
      op.result = &lengths[i];
    }
    testCase(
        "",
        [&] { decodeGlobal<kBlockSize>(ops.get(), numBlocks); },
        numValues * sizeof(int32_t),
        5);
    for (int i = 0; i < numBlocks; ++i) {
      auto& op = ops[i].data.rleTotalLength;
      int64_t expected = 0;
      for (int j = 0; j < op.count; ++j) {
        expected += op.input[j];
      }
      ASSERT_EQ(*op.result, expected);
    }
  }

  template <typename T, int kBlockSize>
  void testRle(int numValues, int numBlocks) {
    auto values = allocate<T>(numValues);
    auto lengths = allocate<int32_t>(numValues);
    int totalLength = 0;
    fillRandom(values.get(), numValues);
    fillRandom(lengths.get(), numValues);
    for (int i = 0; i < numValues; ++i) {
      int limit = i % 1000 == 0 ? 1000 : 10;
      lengths[i] = (uint32_t)lengths[i] % limit;
      totalLength += lengths[i];
    }
    auto ops = allocate<GpuDecode>(numBlocks);
    auto results = allocate<int64_t>(numBlocks);
    int valuesPerOp = (numValues + numBlocks - 1) / numBlocks;
    for (int i = 0; i < numBlocks; ++i) {
      ops[i].step = DecodeStep::kRleTotalLength;
      auto& op = ops[i].data.rleTotalLength;
      op.input = lengths.get() + i * valuesPerOp;
      op.count = std::min(valuesPerOp, numValues - i * valuesPerOp);
      op.result = &results[i];
    }
    decodeGlobal<kBlockSize>(ops.get(), numBlocks);
    CUDA_CHECK_FATAL(hipGetLastError());
    CUDA_CHECK_FATAL(hipDeviceSynchronize());
    auto result = allocate<T>(totalLength);
    int lengthSofar = 0;
    for (int i = 0; i < numBlocks; ++i) {
      int subtotal = *ops[i].data.rleTotalLength.result;
      ops[i].step = DecodeStep::kRle;
      auto& op = ops[i].data.rle;
      op.valueType = WaveTypeTrait<T>::typeKind;
      op.values = values.get() + i * valuesPerOp;
      op.lengths = lengths.get() + i * valuesPerOp;
      op.count = std::min(valuesPerOp, numValues - i * valuesPerOp);
      op.result = result.get() + lengthSofar;
      lengthSofar += subtotal;
    }
    testCase(
        "",
        [&] { decodeGlobal<kBlockSize>(ops.get(), numBlocks); },
        totalLength * sizeof(T),
        3);
    for (int i = 0; i < numBlocks; ++i) {
    }
    for (int i = 0, j = 0; i < numValues; ++i) {
      for (int k = 0; k < lengths[i]; ++k) {
        ASSERT_EQ(result[j++], values[i]);
      }
    }
  }

  template <int kBlockSize>
  void testMakeScatterIndices(int numValues, int numBlocks) {
    auto bits = allocate<uint8_t>((numValues * numBlocks + 7) / 8);
    fillRandomBits(bits.get(), 0.5, numValues * numBlocks);
    auto indices = allocate<int32_t>(numValues * numBlocks);
    auto indicesCounts = allocate<int32_t>(numBlocks);
    auto ops = allocate<GpuDecode>(numBlocks);
    for (int i = 0; i < numBlocks; ++i) {
      ops[i].step = DecodeStep::kMakeScatterIndices;
      auto& op = ops[i].data.makeScatterIndices;
      op.bits = bits.get();
      op.findSetBits = true;
      op.begin = i * numValues;
      op.end = op.begin + numValues;
      op.indices = indices.get() + i * numValues;
      op.indicesCount = indicesCounts.get() + i;
    }
    testCase(
        "",
        [&] { decodeGlobal<kBlockSize>(ops.get(), numBlocks); },
        numValues * numBlocks * sizeof(int32_t),
        3);
    for (int i = 0; i < numBlocks; ++i) {
      auto& op = ops[i].data.makeScatterIndices;
      int k = 0;
      for (int j = 0; j < numValues; ++j) {
        if (isSet(bits.get(), j + i * numValues)) {
          ASSERT_LT(k, *op.indicesCount);
          ASSERT_EQ(op.indices[k++], j);
        }
      }
      ASSERT_EQ(k, *op.indicesCount);
    }
  }

  void testMakeScatterIndicesStream(int numValues, int numBlocks) {
    auto bits = allocate<uint8_t>((numValues * numBlocks + 7) / 8);
    fillRandomBits(bits.get(), 0.5, numValues * numBlocks);
    auto indices = allocate<int32_t>(numValues * numBlocks);
    auto indicesCounts = allocate<int32_t>(numBlocks);
    DecodePrograms programs;
    for (int i = 0; i < numBlocks; ++i) {
      programs.programs.emplace_back();
      programs.programs.back().push_back(std::make_unique<GpuDecode>());
      auto opPtr = programs.programs.back().front().get();
      opPtr->step = DecodeStep::kMakeScatterIndices;
      auto& op = opPtr->data.makeScatterIndices;
      op.bits = bits.get();
      op.findSetBits = true;
      op.begin = i * numValues;
      op.end = op.begin + numValues;
      op.indices = indices.get() + i * numValues;
      op.indicesCount = indicesCounts.get() + i;
    }
    auto stream = std::make_unique<Stream>();
    auto arena =
        std::make_unique<GpuArena>(100000000, getAllocator(getDevice()));
    WaveBufferPtr extra;
    launchDecode(programs, arena.get(), extra, stream.get());
    stream->wait();
    for (int i = 0; i < numBlocks; ++i) {
      auto& op = programs.programs[i].front()->data.makeScatterIndices;
      int k = 0;
      for (int j = 0; j < numValues; ++j) {
        if (isSet(bits.get(), j + i * numValues)) {
          ASSERT_LT(k, *op.indicesCount);
          ASSERT_EQ(op.indices[k++], j);
        }
      }
      ASSERT_EQ(k, *op.indicesCount);
    }
  }

 private:
  hipEvent_t startEvent_;
  hipEvent_t stopEvent_;
};

TEST_F(GpuDecoderTest, trivial) {
  testCopyPlan<uint64_t, 128>(40'000'003, 1024, false);
  testCopyPlan<uint64_t, 128>(40'000'003, 1024, true);
}

TEST_F(GpuDecoderTest, dictionaryOnBitpack) {
  dictTestPlan<int32_t, 256>(11, 4'000'037, 1024, false);
  dictTestPlan<int64_t, 256>(11, 4'000'037, 1024, false);
  dictTestPlan<int32_t, 256>(11, 40'000'003, 1024, false);
  dictTestPlan<int64_t, 256>(11, 40'000'003, 1024, false);
  dictTestPlan<int64_t, 256>(11, 40'000'003, 1024, true);
}

TEST_F(GpuDecoderTest, sparseBool) {
  testSparseBool<256>(40013, 1024);
}

TEST_F(GpuDecoderTest, varint) {
  testVarint<256>(4001, 1024);
}

TEST_F(GpuDecoderTest, mainlyConstant) {
  testMainlyConstant<int64_t, 256>(40013, 1024);
}

TEST_F(GpuDecoderTest, rleTotalLength) {
  testRleTotalLength<256>(40'000'003, 1024);
}

TEST_F(GpuDecoderTest, rle) {
  testRle<int64_t, 256>(40'000'003, 1024);
}

TEST_F(GpuDecoderTest, makeScatterIndices) {
  testMakeScatterIndices<256>(40013, 1024);
}

TEST_F(GpuDecoderTest, streamApi) {
  //  One call with few blocks, another with many, to cover inlined and out of
  //  line params.
  testMakeScatterIndicesStream(100, 20);
  testMakeScatterIndicesStream(999, 999);
}
} // namespace
} // namespace facebook::velox::wave

void printFuncAttrs(
    const std::string& heading,
    const hipFuncAttributes& attrs) {
  std::cout << heading << " sharedSizeBytes=" << attrs.sharedSizeBytes
            << " constSizeBytes" << attrs.constSizeBytes
            << " localSizeBytes =" << attrs.localSizeBytes
            << "maxThreadsPerBlock=" << attrs.maxThreadsPerBlock
            << " numRegs=" << attrs.numRegs
            << " maxDynamicSharedSizeBytes=" << attrs.maxDynamicSharedSizeBytes
            << std::endl;
}
using namespace facebook::velox::wave;

int main(int argc, char** argv) {
  testing::InitGoogleTest(&argc, argv);
  folly::Init init{&argc, &argv};

  hipDeviceProp_t prop;
  CUDA_CHECK_FATAL(hipGetDeviceProperties(&prop, FLAGS_device_id));
  printf("Running on device: %s\n", prop.name);
  CUDA_CHECK_FATAL(hipSetDevice(FLAGS_device_id));
  hipFuncAttributes attrs;
  CUDA_CHECK_FATAL(hipFuncGetAttributes(&attrs, reinterpret_cast<const void*>(detail::decodeGlobal<128>)));
  printFuncAttrs("decode blocksize 128", attrs);
  CUDA_CHECK_FATAL(hipFuncGetAttributes(&attrs, reinterpret_cast<const void*>(detail::decodeGlobal<256>)));
  printFuncAttrs("decode blocksize 256", attrs);
  CUDA_CHECK_FATAL(hipFuncGetAttributes(&attrs, reinterpret_cast<const void*>(detail::decodeGlobal<512>)));
  printFuncAttrs("decode blocksize 512", attrs);
  CUDA_CHECK_FATAL(hipFuncGetAttributes(&attrs, reinterpret_cast<const void*>(detail::decodeGlobal<1024>)));
  printFuncAttrs("decode blocksize 1024", attrs);
  return RUN_ALL_TESTS();
}
