#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "velox/experimental/wave/common/Bits.cuh"
#include "velox/experimental/wave/common/Block.cuh"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/HashTable.cuh"
#include "velox/experimental/wave/common/tests/BlockTest.h"
#include "velox/experimental/wave/common/tests/HashTestUtil.h"
#include "velox/experimental/wave/common/tests/Updates.cuh"

namespace facebook::velox::wave {

using ScanAlgorithm = hipcub::BlockScan<int, 256, hipcub::BLOCK_SCAN_RAKING>;

__global__ void
boolToIndicesKernel(uint8_t** bools, int32_t** indices, int32_t* sizes) {
  extern __shared__ char smem[];
  int32_t idx = blockIdx.x;
  // Start cycle timer
  uint8_t* blockBools = bools[idx];
  boolBlockToIndices<256>(
      [&]() { return blockBools[threadIdx.x]; },
      idx * 256,
      indices[idx],
      smem,
      sizes[idx]);
  __syncthreads();
}

void BlockTestStream::testBoolToIndices(
    int32_t numBlocks,
    uint8_t** flags,
    int32_t** indices,
    int32_t* sizes) {
  CUDA_CHECK(hipGetLastError());
  auto tempBytes = sizeof(typename ScanAlgorithm::TempStorage);
  boolToIndicesKernel<<<numBlocks, 256, tempBytes, stream_->stream>>>(
      flags, indices, sizes);
  CUDA_CHECK(hipGetLastError());
}

__global__ void boolToIndicesNoSharedKernel(
    uint8_t** bools,
    int32_t** indices,
    int32_t* sizes,
    void* temp) {
  int32_t idx = blockIdx.x;

  uint8_t* blockBools = bools[idx];
  char* smem = reinterpret_cast<char*>(temp) +
      blockIdx.x * sizeof(typename ScanAlgorithm::TempStorage);
  boolBlockToIndices<256>(
      [&]() { return blockBools[threadIdx.x]; },
      idx * 256,
      indices[idx],
      smem,
      sizes[idx]);
  __syncthreads();
}

void BlockTestStream::testBoolToIndicesNoShared(
    int32_t numBlocks,
    uint8_t** flags,
    int32_t** indices,
    int32_t* sizes,
    void* temp) {
  CUDA_CHECK(hipGetLastError());
  boolToIndicesNoSharedKernel<<<numBlocks, 256, 0, stream_->stream>>>(
      flags, indices, sizes, temp);
  CUDA_CHECK(hipGetLastError());
}

int32_t BlockTestStream::boolToIndicesSize() {
  return sizeof(typename ScanAlgorithm::TempStorage);
}

__global__ void
bool256ToIndicesKernel(uint8_t** bools, int32_t** indices, int32_t* sizes) {
  extern __shared__ char smem[];
  int32_t idx = blockIdx.x;
  auto* bool64 = reinterpret_cast<uint64_t*>(bools[idx]);
  bool256ToIndices(
      [&](int32_t index8) { return bool64[index8]; },
      idx * 256,
      indices[idx],
      sizes[idx],
      smem);
  __syncthreads();
}

void BlockTestStream::testBool256ToIndices(
    int32_t numBlocks,
    uint8_t** flags,
    int32_t** indices,
    int32_t* sizes) {
  CUDA_CHECK(hipGetLastError());
  auto tempBytes = bool256ToIndicesSize();
  bool256ToIndicesKernel<<<numBlocks, 256, tempBytes, stream_->stream>>>(
      flags, indices, sizes);
  CUDA_CHECK(hipGetLastError());
}

__global__ void bool256ToIndicesNoSharedKernel(
    uint8_t** bools,
    int32_t** indices,
    int32_t* sizes,
    void* temp) {
  int32_t idx = blockIdx.x;
  auto* bool64 = reinterpret_cast<uint64_t*>(bools[idx]);
  char* smem = reinterpret_cast<char*>(temp) + blockIdx.x * 80;
  bool256ToIndices(
      [&](int32_t index8) { return bool64[index8]; },
      idx * 256,
      indices[idx],
      sizes[idx],
      smem);
  __syncthreads();
}

void BlockTestStream::testBool256ToIndicesNoShared(
    int32_t numBlocks,
    uint8_t** flags,
    int32_t** indices,
    int32_t* sizes,
    void* temp) {
  CUDA_CHECK(hipGetLastError());
  bool256ToIndicesNoSharedKernel<<<numBlocks, 256, 0, stream_->stream>>>(
      flags, indices, sizes, temp);
  CUDA_CHECK(hipGetLastError());
}

int32_t BlockTestStream::bool256ToIndicesSize() {
  return 80;
}

__global__ void sum64(int64_t* numbers, int64_t* results) {
  extern __shared__ char smem[];
  int32_t idx = blockIdx.x;
  blockSum<256>(
      [&]() { return numbers[idx * 256 + threadIdx.x]; }, smem, results);
  __syncthreads();
}

void BlockTestStream::testSum64(
    int32_t numBlocks,
    int64_t* numbers,
    int64_t* results) {
  auto tempBytes = sizeof(typename hipcub::BlockReduce<int64_t, 256>::TempStorage);
  sum64<<<numBlocks, 256, tempBytes, stream_->stream>>>(numbers, results);
  CUDA_CHECK(hipGetLastError());
}

/// Keys and values are n sections of 8K items. The items in each section get
/// sorted on the key.
void __global__ __launch_bounds__(1024)
    testSort(uint16_t** keys, uint16_t** values) {
  extern __shared__ __align__(16) char smem[];
  auto keyBase = keys[blockIdx.x];
  auto valueBase = values[blockIdx.x];
  blockSort<256, 32>(
      [&](auto i) { return keyBase[i]; },
      [&](auto i) { return valueBase[i]; },
      keys[blockIdx.x],
      values[blockIdx.x],
      smem);
  __syncthreads();
}

void __global__ __launch_bounds__(1024)
    testSortNoShared(uint16_t** keys, uint16_t** values, char* smem) {
  auto keyBase = keys[blockIdx.x];
  auto valueBase = values[blockIdx.x];
  char* tbTemp = smem +
      blockIdx.x *
          sizeof(typename hipcub::BlockRadixSort<uint16_t, 256, 32, uint16_t>::
                     TempStorage);

  blockSort<256, 32>(
      [&](auto i) { return keyBase[i]; },
      [&](auto i) { return valueBase[i]; },
      keys[blockIdx.x],
      values[blockIdx.x],
      tbTemp);
  __syncthreads();
}

int32_t BlockTestStream::sort16SharedSize() {
  return sizeof(
      typename hipcub::BlockRadixSort<uint16_t, 256, 32, uint16_t>::TempStorage);
}

void BlockTestStream::testSort16(
    int32_t numBlocks,
    uint16_t** keys,
    uint16_t** values) {
  auto tempBytes = sizeof(
      typename hipcub::BlockRadixSort<uint16_t, 256, 32, uint16_t>::TempStorage);

  testSort<<<numBlocks, 256, tempBytes, stream_->stream>>>(keys, values);
}

void BlockTestStream::testSort16NoShared(
    int32_t numBlocks,
    uint16_t** keys,
    uint16_t** values,
    char* temp) {
  testSortNoShared<<<numBlocks, 256, 0, stream_->stream>>>(keys, values, temp);
}

/// Calls partitionRows on each thread block of 256 threads. The parameters
/// correspond to 'partitionRows'. Each is an array subscripted by blockIdx.x.
void __global__ partitionShortsKernel(
    uint16_t** keys,
    int32_t* numKeys,
    int32_t numPartitions,
    int32_t** ranks,
    int32_t** partitionStarts,
    int32_t** partitionedRows) {
  partitionRows<256>(
      [&](auto i) { return keys[blockIdx.x][i]; },
      numKeys[blockIdx.x],
      numPartitions,
      ranks[blockIdx.x],
      partitionStarts[blockIdx.x],
      partitionedRows[blockIdx.x]);
  __syncthreads();
}

void BlockTestStream::partitionShorts(
    int32_t numBlocks,
    uint16_t** keys,
    int32_t* numKeys,
    int32_t numPartitions,
    int32_t** ranks,
    int32_t** partitionStarts,
    int32_t** partitionedRows) {
  constexpr int32_t kBlockSize = 256;
  auto shared = partitionRowsSharedSize<kBlockSize>(numPartitions);
  partitionShortsKernel<<<numBlocks, kBlockSize, shared, stream_->stream>>>(
      keys, numKeys, numPartitions, ranks, partitionStarts, partitionedRows);
  CUDA_CHECK(hipGetLastError());
}

/// A mock complex accumulator update function.
ProbeState __device__ arrayAgg64Append(
    ArrayAgg64* accumulator,
    int64_t arg,
    RowAllocator* allocator) {
  auto* last = accumulator->last;
  if (!last || accumulator->numInLast >= sizeof(last->data) / sizeof(int64_t)) {
    auto* next = allocator->allocate<ArrayAgg64::Run>(1);
    if (!next) {
      return ProbeState::kNeedSpace;
    }
    next->next = nullptr;
    if (accumulator->last) {
      accumulator->last->next = next;
      accumulator->last = next;
    } else {
      accumulator->first = accumulator->last = next;
    }
  }
  accumulator->last->data[accumulator->numInLast++] = arg;
  return ProbeState::kDone;
}

/// An mock Ops parameter class to do group by.
class MockGroupByOps {
 public:
  int32_t __device__ blockBase(HashProbe* probe) {
    return probe->numRowsPerThread * blockDim.x * blockIdx.x;
  }

  int32_t __device__ numRowsInBlock(HashProbe* probe) {
    return probe->numRows[blockIdx.x];
  }

  uint64_t __device__ hash(int32_t i, HashProbe* probe) {
    auto key = reinterpret_cast<int64_t**>(probe->keys)[0];
    return hashMix(1, key[i]);
  }

  bool __device__
  compare(GpuHashTable* table, TestingRow* row, int32_t i, HashProbe* probe) {
    return row->key == reinterpret_cast<int64_t**>(probe->keys)[0][i];
  }

  TestingRow* __device__
  newRow(GpuHashTable* table, int32_t partition, int32_t i, HashProbe* probe) {
    auto* allocator = &table->allocators[partition];
    auto row = allocator->allocateRow<TestingRow>();
    if (row) {
      row->key = reinterpret_cast<int64_t**>(probe->keys)[0][i];
      row->flags = 0;
      row->count = 0;
      new (&row->concatenation) ArrayAgg64();
    }
    return row;
  }

  ProbeState __device__ insert(
      GpuHashTable* table,
      int32_t partition,
      GpuBucket* bucket,
      uint32_t misses,
      uint32_t oldTags,
      uint32_t tagWord,
      int32_t i,
      HashProbe* probe,
      TestingRow*& row) {
    if (!row) {
      row = newRow(table, partition, i, probe);
      if (!row) {
        return ProbeState::kNeedSpace;
      }
    }
    auto missShift = __ffs(misses) - 1;
    if (!bucket->addNewTag(tagWord, oldTags, missShift)) {
      return ProbeState::kRetry;
    }
    bucket->store(missShift / 8, row);
    return ProbeState::kDone;
  }

  TestingRow* __device__ getExclusive(
      GpuHashTable* table,
      GpuBucket* bucket,
      TestingRow* row,
      int32_t hitIdx,
      int32_t warp) {
    return row;
    int32_t nanos = 1;
    for (;;) {
      if (atomicTryLock(&row->flags)) {
        return row;
      }
      __nanosleep((nanos + threadIdx.x) & 31);
      nanos += 3;
    }
  }

  void __device__ writeDone(TestingRow* row) {
    // atomicUnlock(&row->flags);
  }

  ProbeState __device__ update(
      GpuHashTable* table,
      GpuBucket* bucket,
      TestingRow* row,
      int32_t i,
      HashProbe* probe) {
    auto* keys = reinterpret_cast<int64_t**>(probe->keys);
    atomicAdd((unsigned long long*)&row->count, (unsigned long long)keys[1][i]);
    return ProbeState::kDone;
    int64_t arg = keys[1][i];
    int32_t part = table->partitionIdx(bucket - table->buckets);
    auto* allocator = &table->allocators[part];
    auto state = arrayAgg64Append(&row->concatenation, arg, allocator);
    row->flags = 0;
    __threadfence();
    return state;
  }
};

void __global__ __launch_bounds__(1024) hashTestKernel(
    GpuHashTable* table,
    HashProbe* probe,
    BlockTestStream::HashCase mode) {
  switch (mode) {
    case BlockTestStream::HashCase::kGroup: {
      table->updatingProbe<TestingRow>(probe, MockGroupByOps());
      break;
    }
    case BlockTestStream::HashCase::kBuild:
    case BlockTestStream::HashCase::kProbe:
      *(long*)0 = 0; // Unimplemented.
  }
  __syncthreads();
}

void BlockTestStream::hashTest(
    GpuHashTableBase* table,
    HashRun& run,
    HashCase mode) {
  int32_t shared = 0;
  if (mode == HashCase::kGroup) {
    shared = GpuHashTable::updatingProbeSharedSize();
  }
  hashTestKernel<<<run.numBlocks, run.blockSize, shared, stream_->stream>>>(
      reinterpret_cast<GpuHashTable*>(table), run.probe, mode);
  CUDA_CHECK(hipGetLastError());
}

void __global__ allocatorTestKernel(
    int32_t numAlloc,
    int32_t numFree,
    int32_t numStr,
    AllocatorTestResult* allResults) {
  auto* result = allResults + threadIdx.x + blockIdx.x * blockDim.x;
  for (;;) {
    int32_t maxRows = sizeof(result->rows) / sizeof(result->rows[0]);
    int32_t maxStrings = sizeof(result->strings) / sizeof(result->strings[0]);
    for (auto count = 0; count < numAlloc; ++count) {
      if (result->numRows >= maxRows) {
        return;
      }
      auto newRow = result->allocator->allocateRow<int64_t>();
      if (newRow == nullptr) {
        return;
      }
      if (reinterpret_cast<uint64_t>(newRow) == result->allocator->base) {
        printf("");
      }

      result->rows[result->numRows++] = newRow;
    }
    for (auto count = 0; count < numFree; ++count) {
      if (result->numRows == 0) {
        return;
      }
      auto* toFree = result->rows[--result->numRows];
      if (reinterpret_cast<uint64_t>(toFree) == result->allocator->base) {
        printf(""); // GPF();
      }
      if (!result->allocator->inRange(toFree)) {
        GPF();
      }
      result->allocator->freeRow(toFree);
    }
    for (auto count = 0; count < numStr; ++count) {
      if (result->numStrings >= maxStrings) {
        return;
      }
      auto str = result->allocator->allocate<char>(11);
      if (!str) {
        return;
      }
      result->strings[result->numStrings++] = reinterpret_cast<int64_t*>(str);
    }
  }
  __syncthreads();
}

void __global__ initAllocatorKernel(RowAllocator* allocator) {
  if (threadIdx.x == 0) {
    if (allocator->freeSet) {
      reinterpret_cast<FreeSet<uint32_t, 1024>*>(allocator->freeSet)->clear();
    }
  }
  __syncthreads();
}

//  static
int32_t BlockTestStream::freeSetSize() {
  return sizeof(FreeSet<uint32_t, 1024>);
}

void BlockTestStream::initAllocator(HashPartitionAllocator* allocator) {
  initAllocatorKernel<<<1, 1, 0, stream_->stream>>>(
      reinterpret_cast<RowAllocator*>(allocator));
  CUDA_CHECK(hipGetLastError());
}

void BlockTestStream::rowAllocatorTest(
    int32_t numBlocks,
    int32_t numAlloc,
    int32_t numFree,
    int32_t numStr,
    AllocatorTestResult* results) {
  allocatorTestKernel<<<numBlocks, 64, 0, stream_->stream>>>(
      numAlloc, numFree, numStr, results);
  CUDA_CHECK(hipGetLastError());
}

#define UPDATE_CASE(name, func, smem)                                      \
  void __global__ name##Kernel(TestingRow* rows, HashProbe* probe) {       \
    func(rows, probe);                                                     \
    __syncthreads();                                                       \
  }                                                                        \
                                                                           \
  void BlockTestStream::name(TestingRow* rows, HashRun& run) {             \
    name##Kernel<<<run.numBlocks, run.blockSize, smem, stream_->stream>>>( \
        rows, run.probe);                                                  \
    CUDA_CHECK(hipGetLastError());                                        \
  }

UPDATE_CASE(updateSum1NoSync, testSumNoSync, 0);
UPDATE_CASE(updateSum1Mtx, testSumMtx, 0);
UPDATE_CASE(updateSum1MtxCoalesce, testSumMtxCoalesce, 0);
UPDATE_CASE(updateSum1Atomic, testSumAtomic, 0);
UPDATE_CASE(updateSum1AtomicCoalesceShfl, testSumAtomicCoalesceShfl, 0);
UPDATE_CASE(
    updateSum1AtomicCoalesceShmem,
    testSumAtomicCoalesceShmem,
    run.blockSize * sizeof(int64_t));
UPDATE_CASE(updateSum1Exch, testSumExch, sizeof(ProbeShared));
UPDATE_CASE(updateSum1Order, testSumOrder, 0);

void __global__ __launch_bounds__(1024) update1PartitionKernel(
    int32_t numRows,
    int32_t numDistinct,
    int32_t numParts,
    int32_t blockStride,
    HashProbe* probe,
    int32_t* temp) {
  auto blockStart = blockStride * blockIdx.x;
  auto keys = reinterpret_cast<int64_t**>(probe->keys);
  auto indices = keys[0];
  partitionRows<256, int32_t>(
      [&](auto i) -> int32_t { return indices[i + blockStart] % numParts; },
      blockIdx.x == blockDim.x - 1 ? numRows - blockStart : blockStride,
      numParts,
      temp + blockIdx.x * blockStride,
      probe->hostRetries + blockStride * blockIdx.x,
      probe->kernelRetries1 + blockStride * blockIdx.x);
  __syncthreads();
}

void __global__ updateSum1PartKernel(
    TestingRow* rows,
    int32_t numParts,
    HashProbe* probe,
    int32_t numGroups,
    int32_t groupStride) {
  testSumPart(
      rows,
      numParts,
      probe,
      probe->kernelRetries1,
      probe->hostRetries,
      numGroups,
      groupStride);
  __syncthreads();
}

void BlockTestStream::updateSum1Part(TestingRow* rows, HashRun& run) {
  auto numParts = std::min<int32_t>(run.numDistinct, 8192);
  auto groupStride = run.numRows / 32;
  auto numGroups = run.numRows / groupStride;
  auto partSmem = partitionRowsSharedSize<256>(numParts);
  // We use probe->kernelRetries1 as the indices array for partitions. We use
  // probe->hostRetries as the array of partition starts. So, if we have 10
  // partitions, then hostRetries[x..y] is the input rows for partition 1 if x
  // is partitionStarts[0] and y is partitionStarts[1].
  update1PartitionKernel<<<numGroups, 256, partSmem, stream_->stream>>>(
      run.numRows,
      run.numDistinct,
      numParts,
      groupStride,
      run.probe,
      run.partitionTemp);
  CUDA_CHECK(hipGetLastError());

  int32_t blockSize = roundUp(std::min<int32_t>(256, numParts), 32);
  int32_t numBlocks = numParts / blockSize;
  // There will be one lane per partition. The last blocks may have empty lanes.
  if (numBlocks * blockSize < numParts) {
    ++numBlocks;
  }
  updateSum1PartKernel<<<numBlocks, blockSize, 0, stream_->stream>>>(
      rows, numParts, run.probe, numGroups, groupStride);
  CUDA_CHECK(hipGetLastError());
}

__global__ void scatterBitsKernel(
    int32_t numSource,
    int32_t numTarget,
    const char* source,
    const uint64_t* targetMask,
    char* target,
    int32_t* temp) {
  if (!temp) {
    extern __shared__ __align__(16) char smem[];
    temp = reinterpret_cast<int32_t*>(smem);
  }
  scatterBitsDevice<4>(numSource, numTarget, source, targetMask, target, temp);
  __syncthreads();
}

//    static
int32_t BlockTestStream::scatterBitsSize(int32_t blockSize) {
  return scatterBitsDeviceSize(blockSize);
}

void BlockTestStream::scatterBits(
    int32_t numSource,
    int32_t numTarget,
    const char* source,
    const uint64_t* targetMask,
    char* target,
    int32_t* temp) {
  scatterBitsKernel<<<
      1,
      256,
      temp ? 0 : scatterBitsDeviceSize(256),
      stream_->stream>>>(
      numSource, numTarget, source, targetMask, target, temp);
}

void __global__ nonNullIndexKernel(
    char* nulls,
    int32_t* rows,
    int32_t numRows,
    int32_t* indices,
    int32_t* temp) {
  if (threadIdx.x == 0) {
    temp[0] = countBits(reinterpret_cast<uint64_t*>(nulls), 0, rows[0]);
    temp[1] = 0;
  }
  __syncthreads();
  for (auto i = 0; i < numRows; i += blockDim.x) {
    auto last = min(i + 256, numRows);
    if (isDense(rows, i, last)) {
      indices[i + threadIdx.x] =
          nonNullIndex256(nulls, rows[i], last - i, temp, temp + 2);
    } else {
      // If a non-contiguous run is followed by a contiguous run, add the
      // non-nulls after between the runs to the total.
      if (threadIdx.x == 0) {
        int32_t nextLast = min(last + 256, numRows);
        // If the next 256 rows are dense, then add the non-nulls between the
        // last of the sparse and the first of the dense.
        if (isDense(rows, last, nextLast)) {
          temp[1] = countBits(
              reinterpret_cast<uint64_t*>(nulls),
              rows[last - 1] + 1,
              rows[last]);
        }
      }
      indices[i + threadIdx.x] =
          nonNullIndex256Sparse(nulls, rows, i, last, temp, temp + 1, temp + 2);
    }
  }
  __syncthreads();
}

void BlockTestStream::nonNullIndex(
    char* nulls,
    int32_t* rows,
    int32_t numRows,
    int32_t* indices,
    int32_t* temp) {
  nonNullIndexKernel<<<1, 256, 0, stream_->stream>>>(
      nulls, rows, numRows, indices, temp);
}

REGISTER_KERNEL("testSort", testSort);
REGISTER_KERNEL("boolToIndices", boolToIndicesKernel);
REGISTER_KERNEL("bool256ToIndices", bool256ToIndicesKernel);
REGISTER_KERNEL("sum64", sum64);
REGISTER_KERNEL("partitionShorts", partitionShortsKernel);
REGISTER_KERNEL("hashTest", hashTestKernel);
REGISTER_KERNEL("allocatorTest", allocatorTestKernel);
REGISTER_KERNEL("sum1atm", updateSum1AtomicKernel);
REGISTER_KERNEL("sum1atmCoaShfl", updateSum1AtomicCoalesceShflKernel);
REGISTER_KERNEL("sum1atmCoaShmem", updateSum1AtomicCoalesceShmemKernel);
REGISTER_KERNEL("sum1Exch", updateSum1ExchKernel);
REGISTER_KERNEL("sum1Part", updateSum1PartKernel);
REGISTER_KERNEL("partSum", update1PartitionKernel);
REGISTER_KERNEL("scatterBits", scatterBitsKernel);

} // namespace facebook::velox::wave
