#include "hip/hip_runtime.h"
#include "velox/experimental/wave/common/Block.cuh"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/tests/BlockTest.h"

namespace facebook::velox::wave {

using ScanAlgorithm = hipcub::BlockScan<int, 256, hipcub::BLOCK_SCAN_RAKING>;

__global__ void boolToIndices(
    uint8_t** bools,
    int32_t** indices,
    int32_t* sizes,
    int64_t* times) {
  extern __shared__ __align__(alignof(ScanAlgorithm::TempStorage)) char smem[];
  int32_t idx = blockIdx.x;
  // Start cycle timer
  clock_t start = clock();
  uint8_t* blockBools = bools[idx];
  boolBlockToIndices<256>(
      [&]() { return blockBools[threadIdx.x]; },
      idx * 256,
      indices[idx],
      smem,
      sizes[idx]);
  clock_t stop = clock();
  if (threadIdx.x == 0) {
    times[idx] = (start > stop) ? start - stop : stop - start;
  }
}

void BlockTestStream::testBoolToIndices(
    int32_t numBlocks,
    uint8_t** flags,
    int32_t** indices,
    int32_t* sizes,
    int64_t* times) {
  CUDA_CHECK(hipGetLastError());
  auto tempBytes = sizeof(typename ScanAlgorithm::TempStorage);
  boolToIndices<<<numBlocks, 256, tempBytes, stream_->stream>>>(
      flags, indices, sizes, times);
  CUDA_CHECK(hipGetLastError());
}

__global__ void sum64(int64_t* numbers, int64_t* results) {
  extern __shared__ __align__(
      alignof(hipcub::BlockReduce<int64_t, 256>::TempStorage)) char smem[];
  int32_t idx = blockIdx.x;
  blockSum<256>(
      [&]() { return numbers[idx * 256 + threadIdx.x]; }, smem, results);
}

void BlockTestStream::testSum64(
    int32_t numBlocks,
    int64_t* numbers,
    int64_t* results) {
  auto tempBytes = sizeof(typename hipcub::BlockReduce<int64_t, 256>::TempStorage);
  sum64<<<numBlocks, 256, tempBytes, stream_->stream>>>(numbers, results);
  CUDA_CHECK(hipGetLastError());
}

} // namespace facebook::velox::wave
