#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>

#include "velox/experimental/wave/common/Cuda.h"
#include "velox/experimental/wave/common/Hash.h"

namespace facebook::velox::wave {
namespace {

__global__ void murmur3(const char* data, size_t len, uint32_t* out) {
  *out = Murmur3::hashBytes(data, len, 42);
}

void testMurmur3(const std::string& s, uint32_t expected) {
  SCOPED_TRACE(s);
  auto* allocator = getAllocator(getDevice());
  auto buf = allocator->allocate<char>(s.size() + 1);
  memcpy(&buf[1], s.data(), s.size());
  auto actual = allocator->allocate<uint32_t>();
  murmur3<<<1, 1>>>(&buf[1], s.size(), actual.get());
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
  ASSERT_EQ(*actual, expected);
}

TEST(HashTest, murmur3) {
  testMurmur3("foo", 1015597510u);
  testMurmur3("foobar", 3446066726u);
}

} // namespace
} // namespace facebook::velox::wave
