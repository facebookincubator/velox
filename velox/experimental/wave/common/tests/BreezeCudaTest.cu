#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#define PLATFORM_CUDA

// clang-format off
#define CUDA_PLATFORM_SPECIALIZATION_HEADER \
  breeze/platforms/specialization/cuda-ptx.cuh
// clang-format on

#include <breeze/functions/reduce.h>
#include <breeze/platforms/platform.h>
#include <breeze/utils/device_vector.h>
#include <breeze/platforms/cuda.cuh>

#include <gtest/gtest.h>

namespace breeze {
namespace {

using namespace functions;
using namespace utils;

constexpr int kBlockThreads = 256;
constexpr int kItemsPerThread = 8;
constexpr int kBlockItems = kBlockThreads * kItemsPerThread;
constexpr int kNumItems = 250'000;
constexpr int kNumBlocks = (kNumItems + kBlockItems - 1) / kBlockItems;

__global__ __launch_bounds__(kBlockThreads) void reduceKernel(int* out) {
  CudaPlatform<kBlockThreads, 32> p;
  using BlockReduceT = BlockReduce<decltype(p), int>;
  __shared__ typename BlockReduceT::Scratch scratch;
  int items[kItemsPerThread];
  for (int i = 0; i < kItemsPerThread; ++i) {
    items[i] = 1;
  }
  int aggregate = BlockReduceT::template Reduce<ReduceOpAdd, kItemsPerThread>(
      p,
      make_slice(items),
      make_slice(&scratch).template reinterpret<SHARED>());
  if (p.thread_idx() == 0) {
    out[p.block_idx()] = aggregate;
  }
}

TEST(BreezeCudaTest, reduce) {
  device_vector<int> result(kNumBlocks);
  reduceKernel<<<kNumBlocks, kBlockThreads>>>(result.data());
  std::vector<int> actual(kNumBlocks);
  result.copy_to_host(actual.data(), actual.size());
  std::vector<int> expected(kNumBlocks, kBlockThreads * kItemsPerThread);
  ASSERT_EQ(actual, expected);
}

} // namespace
} // namespace breeze
