#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <gtest/gtest.h>
#include "velox/experimental/wave/common/Buffer.h"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/Exception.h"
#include "velox/experimental/wave/common/GpuArena.h"
#include "velox/experimental/wave/common/tests/BlockTest.h"

#include <iostream>

namespace facebook::velox::wave {

void testCuCheck(hipError_t result) {
  if (result != hipSuccess) {
    const char* str;
    hipDrvGetErrorString(result, &str);
    waveError(std::string("Cuda error: ") + str);
  }
}

class CompileTest : public testing::Test {
 protected:
  void SetUp() override {
    device_ = getDevice();
    setDevice(device_);
    allocator_ = getAllocator(device_);
    arena_ = std::make_unique<GpuArena>(1 << 28, allocator_);
    streams_.push_back(std::make_unique<BlockTestStream>());
  }

  Device* device_;
  GpuAllocator* allocator_;
  std::unique_ptr<GpuArena> arena_;
  std::vector<std::unique_ptr<BlockTestStream>> streams_;
};

struct KernelParams {
  int32_t* array;
  int32_t size;
};

const char* kernelText =
    "using int32_t = int; //#include <cstdint>\n"
    "namespace facebook::velox::wave {\n"
    "  struct KernelParams {\n"
    "    int32_t* array;\n"
    "    int32_t size;\n"
    "  };\n"
    "\n"
    "  void __global__ add1(KernelParams params) {\n"
    "    for (auto i = threadIdx.x; i < params.size; i += blockDim.x) {\n"
    "      ++params.array[i];\n"
    "    }\n"
    "  }\n"
    "\n"
    "  void __global__ add2(KernelParams params) {\n"
    "    for (auto i = threadIdx.x; i < params.size; i += blockDim.x) {\n"
    "      params.array[i] += 2;\n"
    "    }\n"
    "  }\n"
    "} // namespace\n";

void __global__ add3(KernelParams params) {
  for (auto i = threadIdx.x; i < params.size; i += blockDim.x) {
    params.array[i] += 3;
  }
}

TEST_F(CompileTest, module) {
  KernelSpec spec = KernelSpec{
      kernelText,
      {"facebook::velox::wave::add1", "facebook::velox::wave::add2"},
      "/tmp/add1.cu"};
  auto module = CompiledModule::create(spec);
  int32_t* ptr;
  testCuCheck(hipMallocManaged(
      reinterpret_cast<hipDeviceptr_t*>(&ptr),
      1000 * sizeof(int32_t),
      hipMemAttachGlobal));
  KernelParams record{ptr, 1000};
  memset(ptr, 0, 1000 * sizeof(int32_t));
  void* recordPtr = &record;
  auto impl = std::make_unique<StreamImpl>();
  testCuCheck(hipStreamCreateWithFlags((hipStream_t*)&impl->stream, hipStreamDefault));
  auto stream = std::make_unique<Stream>(std::move(impl));
  module->launch(0, 1, 256, 0, stream.get(), &recordPtr);
  testCuCheck(hipStreamSynchronize((hipStream_t)stream->stream()->stream));
  EXPECT_EQ(1, ptr[0]);
  auto info = module->info(0);
  EXPECT_EQ(1024, info.maxThreadsPerBlock);

  // See if runtime API kernel works on driver API stream.
  add3<<<1, 256, 0, (hipStream_t)stream->stream()->stream>>>(record);
  CUDA_CHECK(hipGetLastError());
  testCuCheck(hipStreamSynchronize((hipStream_t)stream->stream()->stream));
  EXPECT_EQ(4, ptr[0]);

  auto stream2 = std::make_unique<Stream>();
  module->launch(1, 1, 256, 0, stream2.get(), &recordPtr);
  stream2->wait();
  EXPECT_EQ(6, ptr[0]);
}

TEST_F(CompileTest, cache) {
  KernelSpec spec = KernelSpec{
      kernelText,
      {"facebook::velox::wave::add1", "facebook::velox::wave::add2"},
      "/tmp/add1.cu"};
  auto kernel =
      CompiledKernel::getKernel("add1", [&]() -> KernelSpec { return spec; });
  auto buffer = arena_->allocate<int32_t>(1000);
  memset(buffer->as<int32_t>(), 0, sizeof(int32_t) * 1000);
  KernelParams record{buffer->as<int32_t>(), 1000};
  void* recordPtr = &record;
  auto stream = std::make_unique<Stream>();
  kernel->launch(1, 1, 256, 0, stream.get(), &recordPtr);
  stream->wait();
  EXPECT_EQ(2, buffer->as<int32_t>()[0]);
}

} // namespace facebook::velox::wave
