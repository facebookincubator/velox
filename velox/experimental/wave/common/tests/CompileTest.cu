#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <gtest/gtest.h>
#include "velox/experimental/wave/common/Buffer.h"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/Exception.h"
#include "velox/experimental/wave/common/GpuArena.h"
#include "velox/experimental/wave/common/tests/BlockTest.h"

#include <iostream>

namespace facebook::velox::wave {

void testCuCheck(hipError_t result) {
  if (result != hipSuccess) {
    const char* str;
    hipDrvGetErrorString(result, &str);
    waveError(std::string("Cuda error: ") + str);
  }
}

class CompileTest : public testing::Test {
 protected:
  void SetUp() override {
    device_ = getDevice();
    setDevice(device_);
    allocator_ = getAllocator(device_);
    arena_ = std::make_unique<GpuArena>(1 << 28, allocator_);
    streams_.push_back(std::make_unique<BlockTestStream>());
  }

  Device* device_;
  GpuAllocator* allocator_;
  std::unique_ptr<GpuArena> arena_;
  std::vector<std::unique_ptr<BlockTestStream>> streams_;
};

struct KernelParams {
  int32_t* array;
  int32_t size;
};

const char* kernelText =
    "using int32_t = int; //#include <cstdint>\n"
    "namespace facebook::velox::wave {\n"
    "  struct KernelParams {\n"
    "    int32_t* array;\n"
    "    int32_t size;\n"
    "  };\n"
    "\n"
    "  void __global__ add1(KernelParams params) {\n"
    "    for (auto i = threadIdx.x; i < params.size; i += blockDim.x) {\n"
    "      ++params.array[i];\n"
    "    }\n"
    "  }\n"
    "\n"
    "  void __global__ add2(KernelParams params) {\n"
    "    for (auto i = threadIdx.x; i < params.size; i += blockDim.x) {\n"
    "      params.array[i] += 2;\n"
    "    }\n"
    "  }\n"
    "} // namespace\n";

void __global__ add3(KernelParams params) {
  for (auto i = threadIdx.x; i < params.size; i += blockDim.x) {
    params.array[i] += 3;
  }
}

TEST_F(CompileTest, module) {
  KernelSpec spec = KernelSpec{
      kernelText,
      {"facebook::velox::wave::add1", "facebook::velox::wave::add2"},
      "/tmp/add1.cu"};
  auto module = CompiledModule::create(spec);
  int32_t* ptr;
  testCuCheck(hipMallocManaged(
      reinterpret_cast<hipDeviceptr_t*>(&ptr),
      1000 * sizeof(int32_t),
      hipMemAttachGlobal));
  KernelParams record{ptr, 1000};
  memset(ptr, 0, 1000 * sizeof(int32_t));
  void* recordPtr = &record;
  auto impl = std::make_unique<StreamImpl>();
  testCuCheck(hipStreamCreateWithFlags((hipStream_t*)&impl->stream, hipStreamDefault));
  auto stream = std::make_unique<Stream>(std::move(impl));
  module->launch(0, 1, 256, 0, stream.get(), &recordPtr);
  testCuCheck(hipStreamSynchronize((hipStream_t)stream->stream()->stream));
  EXPECT_EQ(1, ptr[0]);
  auto info = module->info(0);
  EXPECT_EQ(1024, info.maxThreadsPerBlock);

  // See if runtime API kernel works on driver API stream.
  add3<<<1, 256, 0, (hipStream_t)stream->stream()->stream>>>(record);
  CUDA_CHECK(hipGetLastError());
  testCuCheck(hipStreamSynchronize((hipStream_t)stream->stream()->stream));
  EXPECT_EQ(4, ptr[0]);

  auto stream2 = std::make_unique<Stream>();
  module->launch(1, 1, 256, 0, stream2.get(), &recordPtr);
  stream2->wait();
  EXPECT_EQ(6, ptr[0]);
}

TEST_F(CompileTest, cache) {
  KernelSpec spec = KernelSpec{
      kernelText,
      {"facebook::velox::wave::add1", "facebook::velox::wave::add2"},
      "/tmp/add1.cu"};
  auto kernel =
      CompiledKernel::getKernel("add1", [&]() -> KernelSpec { return spec; });
  auto buffer = arena_->allocate<int32_t>(1000);
  memset(buffer->as<int32_t>(), 0, sizeof(int32_t) * 1000);
  KernelParams record{buffer->as<int32_t>(), 1000};
  void* recordPtr = &record;
  auto stream = std::make_unique<Stream>();
  kernel->launch(1, 1, 256, 0, stream.get(), &recordPtr);
  stream->wait();
  EXPECT_EQ(2, buffer->as<int32_t>()[0]);
}

TEST_F(CompileTest, scan) {
  // Tests a warp prefix sum across the warp and then across the 8 first lanes
  // of the warp.

  const char* text =
      "#include \"velox/experimental/wave/common/Scan.cuh\"\n"
      "namespace facebook::velox::wave {\n"
      "__global__ void scanKernel32(int32_t* ints) {\n"
      "  using Scan = WarpScan<uint32_t>;\n"
      "uint32_t out;\n"
      " Scan().exclusiveSum(ints[threadIdx.x], out);\n"
      "ints[threadIdx.x] = out;\n"
      "__syncthreads();\n"
      "}\n"
      "__global__ void scanKernel8(int32_t* ints) {\n"
      "  using Scan = WarpScan<uint32_t, 8>;\n"
      "uint32_t out;\n"
      " Scan().exclusiveSum(ints[threadIdx.x], out);\n"
      "ints[threadIdx.x] = out;\n"
      "__syncthreads();\n"
      "}\n"
      "}\n";

  WaveBufferPtr ints = arena_->allocate<uint32_t>(32);
  for (auto i = 0; i < 32; ++i) {
    ints->as<uint32_t>()[i] = i;
  }
  KernelSpec spec = {
      text,
      {"facebook::velox::wave::scanKernel32",
       "facebook::velox::wave::scanKernel8"},
      "scans.cu"};
  auto module = CompiledModule::create(spec);
  auto stream = std::make_unique<Stream>();
  auto rawInts = ints->as<int32_t>();
  void* params = &rawInts;
  module->launch(0, 1, 32, 0, stream.get(), &params);
  stream->wait();
  int32_t sum = 0;
  for (auto i = 0; i < 32; ++i) {
    EXPECT_EQ(rawInts[i], sum);
    sum += i;
  }

  // test prefix sum over the 8 first lanes.
  for (auto i = 0; i < 32; ++i) {
    rawInts[i] = i;
  }
  module->launch(1, 1, 32, 0, stream.get(), &params);
  stream->wait();
  sum = 0;
  for (auto i = 0; i < 8; ++i) {
    EXPECT_EQ(rawInts[i], i < 8 ? sum : i);
    sum += i;
  }
}

TEST_F(CompileTest, reduce) {
  // Tests a warp reduce.

  const char* text =
      "#include \"velox/experimental/wave/common/Scan.cuh\"\n"
      "namespace facebook::velox::wave {\n"
      "template <typename T> __device__ __forceinline__ T add(T x, T y) {return x + y;}\n"
      "__global__ void reduceKernel32(int32_t* ints, int32_t* result) {\n"
      "  using Reduce = WarpReduce<uint32_t>;\n"
      "  result[threadIdx.x] = Reduce().reduce(ints[threadIdx.x], add<int32_t>);\n"
      "__syncthreads();\n"
      "}\n"
      "}\n";

  WaveBufferPtr ints = arena_->allocate<uint32_t>(32);
  for (auto i = 0; i < 32; ++i) {
    ints->as<uint32_t>()[i] = i;
  }
  WaveBufferPtr result = arena_->allocate<uint32_t>(32);

  KernelSpec spec = {
      text, {"facebook::velox::wave::reduceKernel32"}, "reduces.cu"};
  auto module = CompiledModule::create(spec);
  auto ptr1 = ints->as<int32_t>();
  auto ptr2 = result->as<int32_t>();
  auto stream = std::make_unique<Stream>();
  int32_t** arrays[2] = {&ptr1, &ptr2};
  module->launch(0, 1, 32, 0, stream.get(), reinterpret_cast<void**>(arrays));
  stream->wait();
  int32_t sum = 0;
  for (auto i = 0; i < 32; ++i) {
    sum += i;
  }
  EXPECT_EQ(sum, ptr2[0]);
}

} // namespace facebook::velox::wave
