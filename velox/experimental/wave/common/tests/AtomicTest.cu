#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>

#include "velox/experimental/wave/common/Atomic.cuh"
#include "velox/experimental/wave/common/Cuda.h"

namespace facebook::velox::wave {
namespace {

template <typename T, typename U>
inline __device__ Atomic<T, MemoryScope::kDevice>* asDeviceAtomic(U* ptr) {
  return reinterpret_cast<Atomic<T, MemoryScope::kDevice>*>(ptr);
}

template <MemoryOrder Order>
__global__ void loadKernel(int* in, int* out) {
  *out = asDeviceAtomic<int>(in)->template load<Order>();
}

template <MemoryOrder Order>
__global__ void storeKernel(int in, int* out) {
  asDeviceAtomic<int>(out)->template store<Order>(in);
}

template <MemoryOrder Order>
__global__ void
compareExchangeKernel(int* expected, int desired, int* out, bool* was_changed) {
  int expected_value = *expected;
  *was_changed = asDeviceAtomic<int>(out)->template compare_exchange<Order>(
      expected_value, desired);
  *expected = expected_value;
}

template <MemoryOrder Order>
struct AtomicTestType {
  static const MemoryOrder memory_order = Order;
};

struct AtomicTestNameGenerator {
  template <typename T>
  static std::string GetName(int) {
    if constexpr (T::memory_order == MemoryOrder::kRelaxed)
      return "relaxed";
    if constexpr (T::memory_order == MemoryOrder::kAcquire)
      return "acquire";
    if constexpr (T::memory_order == MemoryOrder::kRelease)
      return "release";
    return "?";
  }
};

template <typename TypeParam>
class AtomicLoadTest : public testing::Test {};

using AtomicLoadTestTypes = ::testing::Types<
    AtomicTestType<MemoryOrder::kRelaxed>,
    AtomicTestType<MemoryOrder::kAcquire>>;

TYPED_TEST_SUITE(AtomicLoadTest, AtomicLoadTestTypes, AtomicTestNameGenerator);

TYPED_TEST(AtomicLoadTest, load) {
  auto* allocator = getAllocator(getDevice());
  auto input = allocator->allocate<int>();
  *input = 1234;
  auto output = allocator->allocate<int>();
  loadKernel<TypeParam::memory_order><<<1, 1>>>(input.get(), output.get());
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
  ASSERT_EQ(*output, 1234);
}

template <typename TypeParam>
class AtomicStoreTest : public testing::Test {};

using AtomicStoreTestTypes = ::testing::Types<
    AtomicTestType<MemoryOrder::kRelaxed>,
    AtomicTestType<MemoryOrder::kRelease>>;

TYPED_TEST_SUITE(
    AtomicStoreTest,
    AtomicStoreTestTypes,
    AtomicTestNameGenerator);

TYPED_TEST(AtomicStoreTest, store) {
  auto* allocator = getAllocator(getDevice());
  auto output = allocator->allocate<int>();
  storeKernel<TypeParam::memory_order><<<1, 1>>>(4321, output.get());
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
  ASSERT_EQ(*output, 4321);
}

template <typename TypeParam>
class AtomicCompareExchangeTest : public testing::Test {};

using AtomicCompareExchangeTestTypes = ::testing::Types<
    AtomicTestType<MemoryOrder::kRelaxed>,
    AtomicTestType<MemoryOrder::kAcquire>,
    AtomicTestType<MemoryOrder::kRelease>>;

TYPED_TEST_SUITE(
    AtomicCompareExchangeTest,
    AtomicCompareExchangeTestTypes,
    AtomicTestNameGenerator);

TYPED_TEST(AtomicCompareExchangeTest, compare_exchange) {
  auto* allocator = getAllocator(getDevice());
  auto expected = allocator->allocate<int>();
  *expected = 1234;
  auto output = allocator->allocate<int>();
  *output = 0;
  auto was_changed = allocator->allocate<bool>();
  *was_changed = true;
  compareExchangeKernel<TypeParam::memory_order>
      <<<1, 1>>>(expected.get(), 4321, output.get(), was_changed.get());
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
  ASSERT_EQ(*output, 0);
  ASSERT_EQ(*expected, 0);
  ASSERT_EQ(*was_changed, false);
  *output = 1234;
  *expected = 1234;
  compareExchangeKernel<TypeParam::memory_order>
      <<<1, 1>>>(expected.get(), 4321, output.get(), was_changed.get());
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
  ASSERT_EQ(*output, 4321);
  ASSERT_EQ(*expected, 1234);
  ASSERT_EQ(*was_changed, true);
}

inline __device__ AtomicMutex<MemoryScope::kDevice>* asDeviceAtomicMutex(
    int* ptr) {
  return reinterpret_cast<AtomicMutex<MemoryScope::kDevice>*>(ptr);
}

__global__ void mutexKernel(int* mtx, int* out) {
  asDeviceAtomicMutex(mtx)->acquire();
  *out += 1;
  asDeviceAtomicMutex(mtx)->release();
}

TEST(AtomicMutexTest, basic) {
  auto* allocator = getAllocator(getDevice());
  auto mutex = allocator->allocate<int>();
  *mutex = 1;
  auto output = allocator->allocate<int>();
  *output = 0;
  mutexKernel<<<4, 8>>>(mutex.get(), output.get());
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
  ASSERT_EQ(*mutex, 1);
  ASSERT_EQ(*output, 32);
}

} // namespace
} // namespace facebook::velox::wave
