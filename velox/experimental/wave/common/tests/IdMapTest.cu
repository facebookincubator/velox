#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <folly/init/Init.h>
#include <gtest/gtest.h>
#include <random>

#include "velox/experimental/wave/common/Cuda.h"
#include "velox/experimental/wave/common/IdMap.cuh"

DEFINE_bool(benchmark, false, "");

namespace facebook::velox::wave {
namespace {

constexpr int kBlockSize = 256;

std::random_device::result_type randomSeed() {
  auto seed = std::random_device{}();
  LOG(INFO) << "Random seed: " << seed;
  return seed;
}

template <typename T>
struct IdMapHolder {
  GpuAllocator::UniquePtr<T[]> values;
  GpuAllocator::UniquePtr<int32_t[]> ids;
  GpuAllocator::UniquePtr<IdMap<T>> idMap;
};

template <typename T>
__global__ void initTable(IdMap<T>* idMap) {
  idMap->clearTable();
}

template <typename T>
IdMapHolder<T> createIdMap(GpuAllocator* allocator, int capacity) {
  IdMapHolder<T> holder;
  holder.idMap = allocator->allocate<IdMap<T>>();
  holder.values = allocator->allocate<T>(capacity);
  holder.ids = allocator->allocate<int32_t>(capacity);
  holder.idMap->init(capacity, holder.values.get(), holder.ids.get());
  initTable<<<1, kBlockSize>>>(holder.idMap.get());
  EXPECT_EQ(hipGetLastError(), hipSuccess);
  return holder;
}

template <typename T>
__global__ void
runMakeIds(IdMap<T>* idMap, const T* values, int size, int32_t* output) {
  int step = gridDim.x * blockDim.x;
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i += step) {
    output[i] = idMap->makeId(values[i]);
  }
}

template <typename T>
void makeIds(IdMap<T>* idMap, const T* values, int size, int32_t* output) {
  int numBlocks = (size + kBlockSize - 1) / kBlockSize;
  Stream stream;
  stream.prefetch(getDevice(), const_cast<T*>(values), size * sizeof(T));
  stream.prefetch(getDevice(), output, size * sizeof(int32_t));
  Event start(true), stop(true);
  start.record(stream);
  runMakeIds<<<numBlocks, kBlockSize>>>(idMap, values, size, output);
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  stop.record(stream);
  stop.wait();
  if (FLAGS_benchmark) {
    auto ms = stop.elapsedTime(start);
    LOG(INFO) << std::setprecision(3) << size * 1e-6 / ms << " billion rows/s";
  }
}

template <typename T>
void validate(int size, const T* values, const int32_t* ids) {
  std::unordered_map<T, int32_t, Hasher<T, uint32_t>> sofar;
  for (int i = 0; i < size; ++i) {
    ASSERT_GT(ids[i], 0);
    auto it = sofar.find(values[i]);
    if (it == sofar.end()) {
      sofar[values[i]] = ids[i];
    } else {
      ASSERT_EQ(it->second, ids[i]);
    }
  }
  std::vector<bool> haveId(sofar.size());
  for (auto& [_, id] : sofar) {
    ASSERT_FALSE(haveId[id - 1]);
    haveId[id - 1] = true;
  }
}

TEST(IdMapTest, stringView) {
  constexpr int kCapacity = 64;
  constexpr int kUniqueSize = 26;
  const int valueSize = FLAGS_benchmark ? 40'000'000 : 1009;
  auto* allocator = getAllocator(getDevice());
  auto holder = createIdMap<StringView>(allocator, kCapacity);
  auto uniqueValues =
      allocator->allocate<char>(kUniqueSize * (1 + kUniqueSize) / 2);
  for (int i = 0; i < kUniqueSize; ++i) {
    int j = i * (i + 1) / 2;
    memset(&uniqueValues[j], 'a' + i, i + 1);
  }
  auto values = allocator->allocate<StringView>(valueSize);
  std::default_random_engine gen(randomSeed());
  std::uniform_int_distribution<> dist(1, kUniqueSize);
  for (int i = 0; i < valueSize; ++i) {
    int j = dist(gen);
    int k = (j - 1) * j / 2;
    values[i].init(&uniqueValues[k], j);
  }
  auto output = allocator->allocate<int32_t>(valueSize);
  makeIds(holder.idMap.get(), values.get(), valueSize, output.get());
  validate(valueSize, values.get(), output.get());
}

TEST(IdMapTest, int64) {
  constexpr int kCapacity = 8 << 10;
  const int valueSize = FLAGS_benchmark ? 40'000'000 : 40013;
  auto* allocator = getAllocator(getDevice());
  auto holder = createIdMap<int64_t>(allocator, kCapacity);
  auto values = allocator->allocate<int64_t>(valueSize);
  std::default_random_engine gen(randomSeed());
  std::uniform_int_distribution<> dist(0, kCapacity / 2);
  for (int i = 0; i < valueSize; ++i) {
    values[i] = 23 + 37 * dist(gen);
  }
  auto output = allocator->allocate<int32_t>(valueSize);
  makeIds(holder.idMap.get(), values.get(), valueSize, output.get());
  validate(valueSize, values.get(), output.get());
}

void testOverflow(bool withEmptyMarker) {
  constexpr int kCapacity = 32;
  constexpr int kValueSize = 4001;
  auto* allocator = getAllocator(getDevice());
  auto holder = createIdMap<int64_t>(allocator, kCapacity);
  auto values = allocator->allocate<int64_t>(kValueSize);
  for (int i = 0; i < kValueSize; ++i) {
    values[i] = i + !withEmptyMarker;
  }
  auto output = allocator->allocate<int32_t>(kValueSize);
  makeIds(holder.idMap.get(), values.get(), kValueSize, output.get());
  ASSERT_GT(std::count(output.get(), output.get() + kValueSize, -1), 0);
}

TEST(IdMapTest, overflowWithEmptyMarker) {
  testOverflow(true);
}

TEST(IdMapTest, overflowNoEmptyMarker) {
  testOverflow(false);
}

} // namespace
} // namespace facebook::velox::wave

int main(int argc, char** argv) {
  testing::InitGoogleTest(&argc, argv);
  folly::Init follyInit(&argc, &argv);
  if (int device; hipGetDevice(&device) != hipSuccess) {
    LOG(WARNING) << "No CUDA detected, skipping all tests";
    return 0;
  }
  return RUN_ALL_TESTS();
}
