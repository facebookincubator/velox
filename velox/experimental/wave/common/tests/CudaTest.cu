#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/tests/CudaTest.h"

namespace facebook::velox::wave {

__global__ void addOneKernel(int32_t* numbers, int32_t size, int32_t stride) {
  auto index = blockDim.x * blockIdx.x + threadIdx.x;
  for (; index < size; index += stride) {
    ++numbers[index];
  }
}

void TestStream::addOne(int32_t* numbers, int32_t size) {
  constexpr int32_t kWidth = 10240;
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > kWidth / kBlockSize) {
    stride = kWidth;
    numBlocks = kWidth / kBlockSize;
  }
  addOneKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride);
  CUDA_CHECK(hipGetLastError());
}

} // namespace facebook::velox::wave
