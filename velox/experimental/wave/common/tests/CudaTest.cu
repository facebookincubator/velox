#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "velox/experimental/wave/common/Block.cuh"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/tests/CudaTest.h"

namespace facebook::velox::wave {
constexpr uint32_t kPrime32 = 1815531889;

__global__ void
incOneKernel(int32_t* numbers, int32_t size, int32_t stride, int32_t repeats) {
  for (auto counter = 0; counter < repeats; ++counter) {
    for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
         index += stride) {
      ++numbers[index];
    }
    __syncthreads();
  }
}

__global__ void
addOneKernel(int32_t* numbers, int32_t size, int32_t stride, int32_t repeats) {
  for (auto counter = 0; counter < repeats; ++counter) {
    for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
         index += stride) {
      numbers[index] += index & 31;
    }
    __syncthreads();
  }
}

__global__ void addOneSharedKernel(
    int32_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  extern __shared__ __align__(16) char smem[];
  int32_t* temp = reinterpret_cast<int32_t*>(smem);
  for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
       index += stride) {
    temp[threadIdx.x] = numbers[index];
    for (auto counter = 0; counter < repeats; ++counter) {
      temp[threadIdx.x] += (index + counter) & 31;
    }
    __syncthreads();
    numbers[index] = temp[threadIdx.x];
  }
}

__global__ void addOneRegKernel(
    int32_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
       index += stride) {
    auto temp = numbers[index];
    for (auto counter = 0; counter < repeats; ++counter) {
      temp += (index + counter) & 31;
    }
    __syncthreads();
    numbers[index] = temp;
  }
}

void TestStream::incOne(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  incOneKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

void TestStream::addOne(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

void TestStream::addOneReg(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneRegKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

void TestStream::addOneShared(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneSharedKernel<<<
      numBlocks,
      kBlockSize,
      kBlockSize * sizeof(int32_t),
      stream_->stream>>>(numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

__global__ void addOneWideKernel(WideParams params) {
  auto numbers = params.numbers;
  auto size = params.size;
  auto repeat = params.repeat;
  auto stride = params.stride;
  for (auto counter = 0; counter < repeat; ++counter) {
    for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
         index += stride) {
      ++numbers[index];
    }
  }
}

void TestStream::addOneWide(
    int32_t* numbers,
    int32_t size,
    int32_t repeat,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  WideParams params;
  params.numbers = numbers;
  params.size = size;
  params.stride = stride;
  params.repeat = repeat;
  addOneWideKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(params);
  CUDA_CHECK(hipGetLastError());
}

__global__ void __launch_bounds__(1024) addOneRandomKernel(
    int32_t* numbers,
    const int32_t* lookup,
    uint32_t size,
    int32_t stride,
    int32_t repeats,
    int32_t numLocal,
    int32_t localStride,
    bool emptyWarps,
    bool emptyThreads) {
  for (uint32_t counter = 0; counter < repeats; ++counter) {
    if (emptyWarps) {
      if (((threadIdx.x / 32) & 1) == 0) {
        for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
             index += stride) {
          auto rnd = deviceScale32(index * (counter + 1) * kPrime32, size);
          auto sum = lookup[rnd];
          auto limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index] += sum;

          rnd = deviceScale32((index + 32) * (counter + 1) * kPrime32, size);
          sum = lookup[rnd];
          limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index + 32] += sum;
        }
      }
    } else if (emptyThreads) {
      if ((threadIdx.x & 1) == 0) {
        for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
             index += stride) {
          auto rnd = deviceScale32(index * (counter + 1) * kPrime32, size);
          auto sum = lookup[rnd];
          auto limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index] += sum;

          rnd = deviceScale32((index + 1) * (counter + 1) * kPrime32, size);
          sum = lookup[rnd];
          limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index + 1] += sum;
        }
      }
    } else {
      for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
           index += stride) {
        auto rnd = deviceScale32(index * (counter + 1) * kPrime32, size);
        auto sum = lookup[rnd];
        auto limit = min(rnd + localStride * (1 + numLocal), size);
        for (auto j = rnd + localStride; j < limit; j += localStride) {
          sum += lookup[j];
        }
        numbers[index] += sum;
      }
    }
    __syncthreads();
  }
  __syncthreads();
}

void TestStream::addOneRandom(
    int32_t* numbers,
    const int32_t* lookup,
    int32_t size,
    int32_t repeats,
    int32_t width,
    int32_t numLocal,
    int32_t localStride,
    bool emptyWarps,
    bool emptyThreads) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneRandomKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers,
      lookup,
      size,
      stride,
      repeats,
      numLocal,
      localStride,
      emptyWarps,
      emptyThreads);
  CUDA_CHECK(hipGetLastError());
}

REGISTER_KERNEL("addOne", addOneKernel);
REGISTER_KERNEL("addOneWide", addOneWideKernel);
REGISTER_KERNEL("addOneRandom", addOneRandomKernel);

} // namespace facebook::velox::wave
