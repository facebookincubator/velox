#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "velox/experimental/wave/common/Block.cuh"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/tests/CudaTest.h"

namespace facebook::velox::wave {
constexpr uint32_t kPrime32 = 1815531889;

__global__ void
addOneKernel(int32_t* numbers, int32_t size, int32_t stride, int32_t repeats) {
  for (auto counter = 0; counter < repeats; ++counter) {
    for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
         index += stride) {
      ++numbers[index];
    }
    __syncthreads();
  }
}

__global__ void addOneSharedKernel(
    int32_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  extern __shared__ __align__(16) char smem[];
  int32_t* temp = reinterpret_cast<int32_t*>(smem);
  for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
       index += stride) {
    temp[threadIdx.x] = numbers[blockDim.x * blockIdx.x + threadIdx.x];
    for (auto counter = 0; counter < repeats; ++counter) {
      ++temp[index];
    }
    __syncthreads();
    numbers[blockDim.x * blockIdx.x + threadIdx.x] = temp[threadIdx.x];
  }
}

void TestStream::addOne(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

__global__ void addOneWideKernel(WideParams params) {
  auto numbers = params.numbers;
  auto size = params.size;
  auto repeat = params.repeat;
  auto stride = params.stride;
  for (auto counter = 0; counter < repeat; ++counter) {
    for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
         index += stride) {
      ++numbers[index];
    }
  }
}

void TestStream::addOneWide(
    int32_t* numbers,
    int32_t size,
    int32_t repeat,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  WideParams params;
  params.numbers = numbers;
  params.size = size;
  params.stride = stride;
  params.repeat = repeat;
  addOneWideKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(params);
  CUDA_CHECK(hipGetLastError());
}

void TestStream::addOneShared(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneSharedKernel<<<
      numBlocks,
      kBlockSize,
      sizeof(int32_t) * kBlockSize,
      stream_->stream>>>(numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

__global__ void __launch_bounds__(1024) addOneRandomKernel(
    int32_t* numbers,
    const int32_t* lookup,
    uint32_t size,
    int32_t stride,
    int32_t repeats,
    bool emptyWarps,
    bool emptyThreads) {
  for (uint32_t counter = 0; counter < repeats; ++counter) {
    if (emptyWarps) {
      if (((threadIdx.x / 32) & 1) == 0) {
        for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
             index += stride) {
          auto rnd = deviceScale32(index * (counter + 1) * kPrime32, size);
          numbers[index] += lookup[rnd];
          rnd = deviceScale32((index + 32) * (counter + 1) * kPrime32, size);
          numbers[index + 32] += lookup[rnd];
        }
      }
    } else if (emptyThreads) {
      if ((threadIdx.x & 1) == 0) {
        for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
             index += stride) {
          auto rnd = deviceScale32(index * (counter + 1) * kPrime32, size);
          numbers[index] += lookup[rnd];
          rnd = deviceScale32((index + 1) * (counter + 1) * kPrime32, size);
          numbers[index + 1] += lookup[rnd];
        }
      }
    } else {
#pragma unroll
      for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
           index += stride) {
        auto rnd = deviceScale32(index * (counter + 1) * kPrime32, size);
        numbers[index] += lookup[rnd];
      }
    }
    __syncthreads();
  }
  __syncthreads();
}

void TestStream::addOneRandom(
    int32_t* numbers,
    const int32_t* lookup,
    int32_t size,
    int32_t repeats,
    int32_t width,
    bool emptyWarps,
    bool emptyThreads) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneRandomKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, lookup, size, stride, repeats, emptyWarps, emptyThreads);
  CUDA_CHECK(hipGetLastError());
}

REGISTER_KERNEL("addOne", addOneKernel);
REGISTER_KERNEL("addOneWide", addOneWideKernel);
REGISTER_KERNEL("addOneRandom", addOneRandomKernel);

} // namespace facebook::velox::wave
