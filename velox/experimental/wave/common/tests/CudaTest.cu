#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/tests/CudaTest.h"

namespace facebook::velox::wave {

__global__ void
addOneKernel(int32_t* numbers, int32_t size, int32_t stride, int32_t repeats) {
  auto index = blockDim.x * blockIdx.x + threadIdx.x;
  for (auto counter = 0; counter < repeats; ++counter) {
    for (; index < size; index += stride) {
      ++numbers[index];
    }
    __syncthreads();
  }
}

void TestStream::addOne(int32_t* numbers, int32_t size, int32_t repeats) {
  constexpr int32_t kWidth = 10240;
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > kWidth / kBlockSize) {
    stride = kWidth;
    numBlocks = kWidth / kBlockSize;
  }
  addOneKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

__global__ void addOneWideKernel(WideParams params) {
  auto index = blockDim.x * blockIdx.x + threadIdx.x;
  auto numbers = params.numbers;
  auto size = params.size;
  auto repeat = params.repeat;
  auto stride = params.stride;
  for (auto counter = 0; counter < repeat; ++counter) {
    for (; index < size; index += stride) {
      ++numbers[index];
    }
  }
}

void TestStream::addOneWide(int32_t* numbers, int32_t size, int32_t repeat) {
  constexpr int32_t kWidth = 10240;
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > kWidth / kBlockSize) {
    stride = kWidth;
    numBlocks = kWidth / kBlockSize;
  }
  WideParams params;
  params.numbers = numbers;
  params.size = size;
  params.stride = stride;
  params.repeat = repeat;
  addOneWideKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(params);
  CUDA_CHECK(hipGetLastError());
}

__global__ void addOneRandomKernel(
    int32_t* numbers,
    const int32_t* lookup,
    uint32_t size,
    int32_t stride,
    int32_t repeats) {
  auto index = blockDim.x * blockIdx.x + threadIdx.x;
  for (uint32_t counter = 0; counter < repeats; ++counter) {
    for (; index < size; index += stride) {
      auto rnd = (static_cast<uint64_t>(static_cast<uint32_t>(
                      index * (counter + 1) * 1367836089)) *
                  size) >>
          32;
      numbers[index] += lookup[rnd];
    }
    __syncthreads();
  }
}

void TestStream::addOneRandom(
    int32_t* numbers,
    const int32_t* lookup,
    int32_t size,
    int32_t repeats) {
  constexpr int32_t kWidth = 10240;
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > kWidth / kBlockSize) {
    stride = kWidth;
    numBlocks = kWidth / kBlockSize;
  }
  addOneRandomKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, lookup, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

} // namespace facebook::velox::wave
