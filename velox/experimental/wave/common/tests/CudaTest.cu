#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <assert.h>
#include "velox/experimental/wave/common/Block.cuh"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/tests/CudaTest.h"

namespace facebook::velox::wave {
constexpr uint32_t kPrime32 = 1815531889;

struct ResultPair {
  int64_t n;
  bool f;
};

typedef ResultPair (
    *TestFunc)(int64_t data, int64_t data2, bool& flag, int32_t* ptr);

__device__ TestFunc testFuncs[2];

__device__ ResultPair
testFunc(int64_t data, int64_t data2, bool& flag, int32_t* ptr) {
  return {data + (data2 & 31), false};
}

void __global__ setupFuncs() {
  testFuncs[0] = testFunc;
  testFuncs[1] = testFunc;
}

__global__ void
incOneKernel(int32_t* numbers, int32_t size, int32_t stride, int32_t repeats) {
  for (auto counter = 0; counter < repeats; ++counter) {
    for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
         index += stride) {
      ++numbers[index];
    }
    __syncthreads();
  }
}

__global__ void
addOneKernel(int32_t* numbers, int32_t size, int32_t stride, int32_t repeats) {
  for (auto counter = 0; counter < repeats; ++counter) {
    for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
         index += stride) {
      numbers[index] += index & 31;
    }
    __syncthreads();
  }
}

__global__ void addOneSharedKernel(
    int32_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  extern __shared__ __align__(16) char smem[];
  int32_t* temp = reinterpret_cast<int32_t*>(smem);
  for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
       index += stride) {
    temp[threadIdx.x] = numbers[index];
    for (auto counter = 0; counter < repeats; ++counter) {
      temp[threadIdx.x] += (index + counter) & 31;
    }
    __syncthreads();
    numbers[index] = temp[threadIdx.x];
  }
}

__global__ void addOneRegKernel(
    int32_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
       index += stride) {
    auto temp = numbers[index];
    for (auto counter = 0; counter < repeats; ++counter) {
      temp += (index + counter) & 31;
    }
    __syncthreads();
    numbers[index] = temp;
  }
}

__global__ void addOneFuncKernel(
    int32_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
       index += stride) {
    int32_t* ptr = nullptr;
    bool flag;
    auto temp = numbers[index];
    for (auto counter = 0; counter < repeats; ++counter) {
      auto result = testFuncs[counter & 1](temp, counter, flag, ptr);
      temp = result.n;
    }
    __syncthreads();
    numbers[index] = temp;
  }
}

#define TCASE(nn, m)                                 \
  case nn:                                           \
    temp = m + testFunc(temp, counter, flag, ptr).n; \
    break;

__global__ void addOneSwitchKernel(
    int32_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
       index += stride) {
    int32_t* ptr = nullptr;
    bool flag;
    auto temp = numbers[index];
    for (auto counter = 0; counter < repeats; ++counter) {
      switch (counter & 31) {
        TCASE(0, 1);
        TCASE(1, 82);
        TCASE(2, 91);
        TCASE(3, 181);
        TCASE(4, 28);
        TCASE(5, 36);
        TCASE(6, 18);
        TCASE(7, 13);
        TCASE(8, 21);
        TCASE(9, 32);
        TCASE(10, 31);
        TCASE(11, 191);
        TCASE(12, 181);
        TCASE(13, 151);
        TCASE(14, 121);
        TCASE(15, 111);
        TCASE(16, 1);
        TCASE(17, 82);
        TCASE(18, 91);
        TCASE(19, 181);
        TCASE(20, 28);
        TCASE(21, 36);
        TCASE(22, 18);
        TCASE(23, 13);
        TCASE(24, 21);
        TCASE(25, 32);
        TCASE(26, 31);
        TCASE(27, 191);
        TCASE(28, 181);
        TCASE(29, 151);
        TCASE(30, 121);
        TCASE(31, 111);
      }
    }
    __syncthreads();
    numbers[index] = temp;
  }
}

#define BTCASE(nn, m)                              \
  asm volatile("BLK" nn ":");                      \
  temp = m + testFunc(temp, counter, flag, ptr).n; \
  if (repeats < 1000000000)                        \
    goto end;

__global__ void addOneBranchKernel(
    int32_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
       index += stride) {
    int32_t* ptr = nullptr;
    bool flag;
    auto temp = numbers[index];

#pragma unroll 1
    for (auto counter = 0; counter < repeats; ++counter) {
#if 0
      if (repeats > 100000000) {
      switch (repeats) {
      case 100000001: asm volatile("bra BLK1;");
      case 100000002: asm volatile("bra BLK2;");
	break;
      }
    }
#endif
      uint32_t bits = counter & 31;
      asm volatile(
          "ts: .branchtargets BLK0, BLK1, BLK2, BLK3, BLK4, BLK5, BLK6, BLK7, BLK8, BLK9, BLK10, BLK11, BLK12, BLK13, BLK14, BLK15, BLK16, BLK17, BLK18, BLK19, BLK20, BLK21, BLK22, BLK23, BLK24, BLK25, BLK26, BLK27, BLK28, BLK29, BLK30, BLK31;");
      asm volatile("brx.idx %0, ts;" ::"r"(bits));

      BTCASE("0", 1);
      BTCASE("1", 82);
      BTCASE("2", 91);
      BTCASE("3", 181);
      BTCASE("4", 28);
      BTCASE("5", 36);
      BTCASE("6", 18);
      BTCASE("7", 13);
      BTCASE("8", 21);
      BTCASE("9", 32);
      BTCASE("10", 31);
      BTCASE("11", 191);
      BTCASE("12", 181);
      BTCASE("13", 151);
      BTCASE("14", 121);
      BTCASE("15", 111);
      BTCASE("16", 1);
      BTCASE("17", 82);
      BTCASE("18", 91);
      BTCASE("19", 181);
      BTCASE("20", 28);
      BTCASE("21", 36);
      BTCASE("22", 18);
      BTCASE("23", 13);
      BTCASE("24", 21);
      BTCASE("25", 32);
      BTCASE("26", 31);
      BTCASE("27", 191);
      BTCASE("28", 181);
      BTCASE("29", 151);
      BTCASE("30", 121);
      BTCASE("31", 111);
    end:;
    }

    __syncthreads();
    numbers[index] = temp;
  }
}

__global__ void addOneFuncStoreKernel(
    int32_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  for (auto counter = 0; counter < repeats; ++counter) {
    for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
         index += stride) {
      int32_t* ptr = nullptr;
      bool flag;
      auto temp = numbers[index];
      numbers[index] = testFuncs[counter & 1](temp, counter, flag, ptr).n;
    }
    __syncthreads();
  }
}

void TestStream::incOne(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  incOneKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

void TestStream::addOne(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

void TestStream::addOneReg(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneRegKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

void TestStream::addOneFunc(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  setupFuncs<<<1, 1, 0, stream_->stream>>>();
  CUDA_CHECK(hipGetLastError());
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneFuncKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

void TestStream::addOneFuncStore(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  setupFuncs<<<1, 1, 0, stream_->stream>>>();
  CUDA_CHECK(hipGetLastError());
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneFuncStoreKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

void TestStream::addOneBranch(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  setupFuncs<<<1, 1, 0, stream_->stream>>>();
  CUDA_CHECK(hipGetLastError());
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneBranchKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

void TestStream::addOneSwitch(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneSwitchKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

void TestStream::addOneShared(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneSharedKernel<<<
      numBlocks,
      kBlockSize,
      kBlockSize * sizeof(int32_t),
      stream_->stream>>>(numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

__global__ void addOneWideKernel(WideParams params) {
  auto numbers = params.numbers;
  auto size = params.size;
  auto repeat = params.repeat;
  auto stride = params.stride;
  for (auto counter = 0; counter < repeat; ++counter) {
    for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
         index += stride) {
      ++numbers[index];
    }
  }
}

void TestStream::addOneWide(
    int32_t* numbers,
    int32_t size,
    int32_t repeat,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  WideParams params;
  params.numbers = numbers;
  params.size = size;
  params.stride = stride;
  params.repeat = repeat;
  addOneWideKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(params);
  CUDA_CHECK(hipGetLastError());
}

__global__ void __launch_bounds__(1024) addOneRandomKernel(
    int32_t* numbers,
    const int32_t* lookup,
    uint32_t size,
    int32_t stride,
    int32_t repeats,
    int32_t numLocal,
    int32_t localStride,
    bool emptyWarps,
    bool emptyThreads) {
  for (uint32_t counter = 0; counter < repeats; ++counter) {
    if (emptyWarps) {
      if (((threadIdx.x / 32) & 1) == 0) {
        for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
             index += stride) {
          auto rnd = deviceScale32(index * (counter + 1) * kPrime32, size);
          auto sum = lookup[rnd];
          auto limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index] += sum;

          rnd = deviceScale32((index + 32) * (counter + 1) * kPrime32, size);
          sum = lookup[rnd];
          limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index + 32] += sum;
        }
      }
    } else if (emptyThreads) {
      if ((threadIdx.x & 1) == 0) {
        for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
             index += stride) {
          auto rnd = deviceScale32(index * (counter + 1) * kPrime32, size);
          auto sum = lookup[rnd];
          auto limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index] += sum;

          rnd = deviceScale32((index + 1) * (counter + 1) * kPrime32, size);
          sum = lookup[rnd];
          limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index + 1] += sum;
        }
      }
    } else {
      for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
           index += stride) {
        auto rnd = deviceScale32(index * (counter + 1) * kPrime32, size);
        auto sum = lookup[rnd];
        auto limit = min(rnd + localStride * (1 + numLocal), size);
        for (auto j = rnd + localStride; j < limit; j += localStride) {
          sum += lookup[j];
        }
        numbers[index] += sum;
      }
    }
    __syncthreads();
  }
  __syncthreads();
}

__global__ void __launch_bounds__(1024) addOneRandomPrefetchKernel(
    int32_t* numbers,
    const int32_t* lookup,
    uint32_t size,
    int32_t stride,
    int32_t repeats,
    int32_t numLocal,
    int32_t localStride,
    bool emptyWarps,
    bool emptyThreads) {
  for (uint32_t counter = 0; counter < repeats; ++counter) {
    if (emptyWarps) {
      if (((threadIdx.x / 32) & 1) == 0) {
        for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
             index += stride) {
          auto rnd = deviceScale32(index * (counter + 1) * kPrime32, size);
          auto sum = lookup[rnd];
          auto limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index] += sum;

          rnd = deviceScale32((index + 32) * (counter + 1) * kPrime32, size);
          sum = lookup[rnd];
          limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index + 32] += sum;
        }
      }
    } else if (emptyThreads) {
      if ((threadIdx.x & 1) == 0) {
        for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
             index += stride) {
          auto rnd = deviceScale32(index * (counter + 1) * kPrime32, size);
          auto sum = lookup[rnd];
          auto limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index] += sum;

          rnd = deviceScale32((index + 1) * (counter + 1) * kPrime32, size);
          sum = lookup[rnd];
          limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index + 1] += sum;
        }
      }
    } else {
      for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
           index += stride) {
        auto rnd = deviceScale32(index * (counter + 1) * kPrime32, size);
        auto sum = lookup[rnd];
        auto limit = min(rnd + localStride * (1 + numLocal), size);
        for (auto j = rnd + localStride; j < limit; j += localStride) {
          sum += lookup[j];
        }
        numbers[index] += sum;
      }
    }
    __syncthreads();
  }
  __syncthreads();
}

void TestStream::addOneRandom(
    int32_t* numbers,
    const int32_t* lookup,
    int32_t size,
    int32_t repeats,
    int32_t width,
    int32_t numLocal,
    int32_t localStride,
    bool emptyWarps,
    bool emptyThreads) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneRandomKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers,
      lookup,
      size,
      stride,
      repeats,
      numLocal,
      localStride,
      emptyWarps,
      emptyThreads);
  CUDA_CHECK(hipGetLastError());
}

/// Memory width and stride
__global__ void addOne64Kernel(
    int64_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  for (auto counter = 0; counter < repeats; ++counter) {
    for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
         index += stride) {
      numbers[index] += index & 31;
    }
    __syncthreads();
  }
}

__global__ void addOne4x64ConsecKernel(
    int64_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  for (auto counter = 0; counter < repeats; ++counter) {
    for (auto index = 4 * (blockDim.x * blockIdx.x + threadIdx.x); index < size;
         index += stride) {
      long2 l1 = *addCast<long2>(numbers, sizeof(int64_t) * index);
      long2 l2 = *addCast<long2>(numbers, sizeof(int64_t) * (index + 2));
      l1.x += index & 31;
      l1.y += (index + 1) & 31;
      l2.x += (index + 2) & 31;
      l2.y += (index + 3) & 31;
      *addCast<long2>(numbers, sizeof(int64_t) * index) = l1;
      *addCast<long2>(numbers, sizeof(int64_t) * (index + 2)) = l2;
    }
    __syncthreads();
  }
}

__global__ void addOne1x64Kernel(
    int64_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  int32_t stride1 = stride / 4;
  for (auto counter = 0; counter < repeats; ++counter) {
    for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
         index += stride1) {
      numbers[index] += 31 & index;
    }
    __syncthreads();
  }
}

__global__ void addOne4Ox64Kernel(
    int64_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  int32_t stride1 = stride / 4;
  for (auto counter = 0; counter < repeats; ++counter) {
    for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
         index += stride) {
      numbers[index] += 31 & index;
      numbers[index + stride1] += 31 & index;
      numbers[index + stride1 * 2] += 31 & index;
      numbers[index + stride1 * 3] += 31 & index;
    }
    __syncthreads();
  }
}

__global__ void addOne4x64CoaKernel(
    int64_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  int32_t halfStride = stride / 2;
  for (auto counter = 0; counter < repeats; ++counter) {
    for (auto index = 4 * (blockDim.x * blockIdx.x + threadIdx.x); index < size;
         index += stride) {
      long2 l1 = *addCast<long2>(numbers, sizeof(int64_t) * index);
      long2 l2 =
          *addCast<long2>(numbers, sizeof(int64_t) * (index + halfStride));
      l1.x += index & 31;
      l1.y += (index + 1) & 31;
      l2.x += (index + halfStride) & 31;
      l2.y += (index + halfStride + 1) & 31;
      *addCast<long2>(numbers, sizeof(int64_t) * index) = l1;
      *addCast<long2>(numbers, sizeof(int64_t) * (index + halfStride)) = l2;
    }
    __syncthreads();
  }
}

struct Params1 {
  int64_t n;
  bool flag;
};

struct Params4 {
  long2 l1;
  long2 l2;
  int32_t flags;
};

typedef Params4 (*TestFunc4)(Params4 params, int32_t index, long* args);

__device__ Params4 testFunc4(Params4 params, int32_t index, long* args) {
  params.l1.x += 32 & index;
  params.l1.y = 31 & (index + 1);
  params.l2.x += 31 & (index + 2);
  params.l2.y += 31 & (index + 3);
  return params;
}

__device__ TestFunc4 testFuncs4[2];

typedef void (*TestFunc4SMem)(int64_t* params, int32_t index, int64_t* args);
__device__ void testFunc4SMem(int64_t* smem, int32_t index, long* args) {
  long2 l1 = *addCast<long2>(smem, 0);
  long2 l2 = *addCast<long2>(smem, 16);
  l1.x += 32 & index;
  l1.y = 31 & (index + 1);
  l2.x += 31 & (index + 2);
  l2.y += 31 & (index + 3);
  *addCast<long2>(smem, 0) = l1;
  *addCast<long2>(smem, 16) = l2;
}

__device__ TestFunc4SMem testFuncs4SMem[2];

__global__ void setupFuncs4() {
  testFuncs[0] = testFunc;
  testFuncs[1] = testFunc;
  testFuncs4[0] = testFunc4;
  testFuncs4[1] = testFunc4;
  testFuncs4SMem[0] = testFunc4SMem;
  testFuncs4SMem[1] = testFunc4SMem;
}

__global__ void addOne4x64RegKernel(
    int64_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  int32_t halfStride = stride / 2;
  for (auto index = 4 * (blockDim.x * blockIdx.x + threadIdx.x); index < size;
       index += stride) {
    long2 l1 = *addCast<long2>(numbers, sizeof(int64_t) * index);
    long2 l2 = *addCast<long2>(numbers, sizeof(int64_t) * (index + halfStride));

    for (auto counter = 0; counter < repeats; ++counter) {
      l1.x += index & 31;
      l1.y += (index + 1) & 31;
      l2.x += (index + halfStride) & 31;
      l2.y += (index + halfStride + 1) & 31;
    }
    *addCast<long2>(numbers, sizeof(int64_t) * (index)) = l1;
    *addCast<long2>(numbers, sizeof(int64_t) * (index + halfStride)) = l2;

    __syncthreads();
  }
}

#define BTCASE4(nn, m)                            \
  asm volatile("BLK" nn ":");                     \
  params = testFunc4(params, index + m, nullptr); \
  if (repeats < 1000000000)                       \
    goto end;

void __global__ __launch_bounds__(1024) addOne4x64BranchKernel(
    int64_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  int32_t halfStride = stride / 2;
  for (auto index = 4 * (blockDim.x * blockIdx.x + threadIdx.x); index < size;
       index += stride) {
    Params4 params;
    params.l1 = *addCast<long2>(numbers, sizeof(int64_t) * index);
    params.l2 =
        *addCast<long2>(numbers, sizeof(int64_t) * (index + halfStride));
    for (auto counter = 0; counter < repeats; ++counter) {
      asm volatile(
          "ts: .branchtargets BLK0, BLK1, BLK2, BLK3, BLK4, BLK5, BLK6, BLK7, BLK8, BLK9, BLK10, BLK11, BLK12, BLK13, BLK14, BLK15, BLK16, BLK17, BLK18, BLK19, BLK20, BLK21, BLK22, BLK23, BLK24, BLK25, BLK26, BLK27, BLK28, BLK29, BLK30, BLK31;");
      asm volatile("brx.idx %0, ts;" ::"r"(counter & 31));

      BTCASE4("0", 1);
      BTCASE4("1", 82);
      BTCASE4("2", 91);
      BTCASE4("3", 181);
      BTCASE4("4", 28);
      BTCASE4("5", 36);
      BTCASE4("6", 18);
      BTCASE4("7", 13);
      BTCASE4("8", 21);
      BTCASE4("9", 32);
      BTCASE4("10", 31);
      BTCASE4("11", 191);
      BTCASE4("12", 181);
      BTCASE4("13", 151);
      BTCASE4("14", 121);
      BTCASE4("15", 111);
      BTCASE4("16", 1);
      BTCASE4("17", 82);
      BTCASE4("18", 91);
      BTCASE4("19", 181);
      BTCASE4("20", 28);
      BTCASE4("21", 36);
      BTCASE4("22", 18);
      BTCASE4("23", 13);
      BTCASE4("24", 21);
      BTCASE4("25", 32);
      BTCASE4("26", 31);
      BTCASE4("27", 191);
      BTCASE4("28", 181);
      BTCASE4("29", 151);
      BTCASE4("30", 121);
      BTCASE4("31", 111);
    end:;
    }
    *addCast<long2>(numbers, sizeof(int64_t) * (index)) = params.l1;
    *addCast<long2>(numbers, sizeof(int64_t) * (index + halfStride)) =
        params.l2;

    __syncthreads();
  }
}

__global__ void addOne4x64FuncKernel(
    int64_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  int32_t halfStride = stride / 2;
  for (auto index = 4 * (blockDim.x * blockIdx.x + threadIdx.x); index < size;
       index += stride) {
    Params4 params;
    params.l1 = *addCast<long2>(numbers, sizeof(int64_t) * index);
    params.l2 =
        *addCast<long2>(numbers, sizeof(int64_t) * (index + halfStride));
    for (auto counter = 0; counter < repeats; ++counter) {
      params = testFuncs4[counter & 1](params, index, nullptr);
    }
    *addCast<long2>(numbers, sizeof(int64_t) * (index)) = params.l1;
    *addCast<long2>(numbers, sizeof(int64_t) * (index + halfStride)) =
        params.l2;

    __syncthreads();
  }
}

__global__ void addOne1x64FuncKernel(
    int64_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  int32_t stride1 = stride / 4;
  for (auto index = (blockDim.x * blockIdx.x + threadIdx.x); index < size;
       index += stride1) {
    ResultPair r;
    r.n = numbers[index];
    bool flag;
    for (auto counter = 0; counter < repeats; ++counter) {
      r = testFuncs[counter & 1](r.n, 0, flag, nullptr);
    }
    numbers[index] = r.n;
    __syncthreads();
  }
}

__global__ void addOne4x64SMemFuncKernel(
    int64_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  int32_t halfStride = stride / 2;
  extern __shared__ __align__(16) char smemBase[];

  for (auto index = 4 * (blockDim.x * blockIdx.x + threadIdx.x); index < size;
       index += stride) {
    long2* smem = addCast<long2>(smemBase, threadIdx.x * 4 * sizeof(int64_t));
    *smem = *addCast<long2>(numbers, sizeof(int64_t) * index);
    smem[1] = *addCast<long2>(numbers, sizeof(int64_t) * (index + halfStride));
    for (auto counter = 0; counter < repeats; ++counter) {
      testFuncs4SMem[counter & 1](
          reinterpret_cast<int64_t*>(smem), index, nullptr);
    }
    *addCast<long2>(numbers, sizeof(int64_t) * (index)) = smem[0];
    *addCast<long2>(numbers, sizeof(int64_t) * (index + halfStride)) = smem[1];
    __syncthreads();
  }
}

void TestStream::addOne4x64(
    int64_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width,
    Add64Mode mode) {
  constexpr int32_t kBlockSize = 256;
  constexpr int32_t kNumPerThread = 4;
  setupFuncs4<<<1, 1, 0, stream_->stream>>>();
  CUDA_CHECK(hipGetLastError());
  int32_t numThreads = size / kNumPerThread;
  auto numBlocks = roundUp(size / kNumPerThread, kBlockSize) / kBlockSize;
  int32_t stride = numThreads;
  if (numBlocks > (width / (kBlockSize * kNumPerThread))) {
    stride = width * kNumPerThread;
    numBlocks = width / kBlockSize;
  }
  int32_t smem = 0;
  if (mode == Add64Mode::k4SMem || mode == Add64Mode::k4SMemFunc) {
    smem = 5 * kBlockSize * sizeof(int64_t);
  }
  switch (mode) {
    case Add64Mode::k4Seq:
      addOne4x64ConsecKernel<<<numBlocks, kBlockSize, smem, stream_->stream>>>(
          numbers, size, stride, repeats);
      break;
    case Add64Mode::k4Add:
      addOne4Ox64Kernel<<<numBlocks, kBlockSize, smem, stream_->stream>>>(
          numbers, size, stride, repeats);
      break;

    case Add64Mode::k4Coa:
      addOne4x64CoaKernel<<<numBlocks, kBlockSize, smem, stream_->stream>>>(
          numbers, size, stride, repeats);
      break;
    case Add64Mode::k4Reg:
      addOne4x64RegKernel<<<numBlocks, kBlockSize, smem, stream_->stream>>>(
          numbers, size, stride, repeats);
      break;
    case Add64Mode::k4Branch:
      addOne4x64BranchKernel<<<numBlocks, kBlockSize, smem, stream_->stream>>>(
          numbers, size, stride, repeats);
      break;

    case Add64Mode::k4Func:
      addOne4x64FuncKernel<<<numBlocks, kBlockSize, smem, stream_->stream>>>(
          numbers, size, stride, repeats);
      break;
    case Add64Mode::k1Func:
      addOne1x64FuncKernel<<<numBlocks, kBlockSize, smem, stream_->stream>>>(
          numbers, size, stride, repeats);
      break;

    case Add64Mode::k4SMemFunc:
      addOne4x64SMemFuncKernel<<<
          numBlocks,
          kBlockSize,
          smem,
          stream_->stream>>>(numbers, size, stride, repeats);
      break;
    case Add64Mode::k1Add:
      addOne1x64Kernel<<<numBlocks, kBlockSize, smem, stream_->stream>>>(
          numbers, size, stride, repeats);
      break;

    default:
      assert(false);
  }
  CUDA_CHECK(hipGetLastError());
}

REGISTER_KERNEL("addOne", addOneKernel);
REGISTER_KERNEL("addOneFunc", addOneFuncKernel);
REGISTER_KERNEL("addOneWide", addOneWideKernel);
REGISTER_KERNEL("addOneRandom", addOneRandomKernel);
REGISTER_KERNEL("add4x64branch", addOne4x64BranchKernel);
REGISTER_KERNEL("add4x64func", addOne4x64FuncKernel);
REGISTER_KERNEL("add4x64smemfunc", addOne4x64SMemFuncKernel);

} // namespace facebook::velox::wave
