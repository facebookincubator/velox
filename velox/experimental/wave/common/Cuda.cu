/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <fmt/format.h>
#include "velox/experimental/wave/common/Cuda.h"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/Exception.h"

namespace facebook::velox::wave {

void cudaCheck(hipError_t err, const char* file, int line) {
  if (err == hipSuccess) {
    return;
  }
  waveError(
      fmt::format("Cuda error: {}:{} {}", file, line, hipGetErrorString(err)));
}

namespace {
class CudaManagedAllocator : public GpuAllocator {
 public:
  void* allocate(size_t size) override {
    void* ret;
    CUDA_CHECK(hipMallocManaged(&ret, size));
    return ret;
  }

  void free(void* ptr, size_t /*size*/) override {
    hipFree(ptr);
  }
};
} // namespace

GpuAllocator* getAllocator(Device* /*device*/) {
  static auto* allocator = new CudaManagedAllocator();
  return allocator;
}

// Always returns device 0.
Device* getDevice(int32_t /*preferredDevice*/) {
  static Device device(0);
  return &device;
}

void setDevice(Device* device) {
  CUDA_CHECK(hipSetDevice(device->deviceId));
}

Stream::Stream() {
  stream = std::make_unique<StreamImpl>();
  CUDA_CHECK(hipStreamCreate(&stream->stream));
}

Stream::~Stream() {
  hipStreamDestroy(stream->stream);
}

void Stream::wait() {
  CUDA_CHECK(hipStreamSynchronize(stream->stream));
}

void Stream::prefetch(Device* device, void* ptr, size_t size) {
  CUDA_CHECK(hipMemPrefetchAsync(
      ptr, size, device ? device->deviceId : hipCpuDeviceId, stream->stream));
}

} // namespace facebook::velox::wave
