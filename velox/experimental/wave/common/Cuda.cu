#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <fmt/format.h>
#include <iostream>
#include "velox/experimental/wave/common/Cuda.h"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/Exception.h"

#include <mutex>
#include <sstream>

namespace facebook::velox::wave {

void cuCheck(hipError_t result, const char* file, int32_t line) {
  if (result != hipSuccess) {
    const char* str;
    hipDrvGetErrorString(result, &str);
    waveError(fmt::format("Cuda error: {}:{} {}", file, line, str));
  }
}

void cudaCheck(hipError_t err, const char* file, int line) {
  if (err == hipSuccess) {
    return;
  }
  waveError(
      fmt::format("Cuda error: {}:{} {}", file, line, hipGetErrorString(err)));
}

void cudaCheckFatal(hipError_t err, const char* file, int line) {
  if (err == hipSuccess) {
    return;
  }
  auto error =
      fmt::format("Cuda error: {}:{} {}", file, line, hipGetErrorString(err));
  std::cerr << err << std::endl;
  exit(1);
}

namespace {
std::mutex ctxMutex;
bool driverInited = false;

// A context for each device. Each is initialized on first use and made the
// primary context for the device.
std::vector<hipCtx_t> contexts;
// Device structs to 1:1 to contexts.
std::vector<std::unique_ptr<Device>> devices;

Device* setDriverDevice(int32_t deviceId) {
  if (!driverInited) {
    std::lock_guard<std::mutex> l(ctxMutex);
    CU_CHECK(hipInit(0));
    int32_t cnt;
    CU_CHECK(hipGetDeviceCount(&cnt));
    contexts.resize(cnt);
    devices.resize(cnt);
    if (cnt == 0) {
      waveError("No Cuda devices found");
    }
  }
  if (deviceId >= contexts.size()) {
    waveError(fmt::format("Bad device id {}", deviceId));
  }
  if (contexts[deviceId] != nullptr) {
    hipCtxSetCurrent(contexts[deviceId]);
    return devices[deviceId].get();
  }
  {
    std::lock_guard<std::mutex> l(ctxMutex);
    hipDevice_t dev;
    CU_CHECK(hipDeviceGet(&dev, deviceId));
    CU_CHECK(hipDevicePrimaryCtxRetain(&contexts[deviceId], dev));
    devices[deviceId] = std::make_unique<Device>(deviceId);
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, deviceId));
    auto& device = devices[deviceId];
    device->model = prop.name;
    device->major = prop.major;
    device->minor = prop.minor;
    device->globalMB = prop.totalGlobalMem >> 20;
    device->numSM = prop.multiProcessorCount;
    device->sharedMemPerSM = prop.sharedMemPerMultiprocessor;
    device->L2Size = prop.l2CacheSize;
    device->persistingL2MaxSize = prop.persistingL2CacheMaxSize;
  }
  CU_CHECK(hipCtxSetCurrent(contexts[deviceId]));
  return devices[deviceId].get();
}

} // namespace

Device* currentDevice() {
  hipCtx_t ctx;
  CU_CHECK(hipCtxGetCurrent(&ctx));
  if (!ctx) {
    return nullptr;
  }
  for (auto i = 0; i < contexts.size(); ++i) {
    if (contexts[i] == ctx) {
      return devices[i].get();
    }
  }
  waveError("Device context not found. Inconsistent state.");
  return nullptr;
}

Device* getDevice(int32_t deviceId) {
  Device* save = nullptr;
  if (driverInited) {
    save = currentDevice();
  }
  auto* dev = setDriverDevice(deviceId);
  if (save) {
    setDevice(save);
  }
  return dev;
}

void setDevice(Device* device) {
  setDriverDevice(device->deviceId);
  CUDA_CHECK(hipSetDevice(device->deviceId));
}

namespace {
class CudaManagedAllocator : public GpuAllocator {
 public:
  void* allocate(size_t size) override {
    void* ret;
    CUDA_CHECK(hipMallocManaged(&ret, size));
    return ret;
  }

  void free(void* ptr, size_t /*size*/) override {
    hipFree(ptr);
  }
};

class CudaDeviceAllocator : public GpuAllocator {
 public:
  void* allocate(size_t size) override {
    void* ret;
    CUDA_CHECK(hipMalloc(&ret, size));
    return ret;
  }

  void free(void* ptr, size_t /*size*/) override {
    hipFree(ptr);
  }
  bool isDevice() const override {
    return true;
  }
};

class CudaHostAllocator : public GpuAllocator {
 public:
  void* allocate(size_t size) override {
    void* ret;
    CUDA_CHECK(hipHostMalloc(&ret, size));
    return ret;
  }

  void free(void* ptr, size_t /*size*/) override {
    hipHostFree(ptr);
  };

  bool isHost() const override {
    return true;
  }
};

} // namespace

GpuAllocator* getAllocator(Device* /*device*/) {
  static auto* allocator = new CudaManagedAllocator();
  return allocator;
}

GpuAllocator* getDeviceAllocator(Device* /*device*/) {
  static auto* allocator = new CudaDeviceAllocator();
  return allocator;
}
GpuAllocator* getHostAllocator(Device* /*device*/) {
  static auto* allocator = new CudaHostAllocator();
  return allocator;
}

Stream::Stream(std::unique_ptr<StreamImpl> impl) : stream_(std::move(impl)) {}

Stream::Stream() {
  stream_ = std::make_unique<StreamImpl>();
  CUDA_CHECK(hipStreamCreate(&stream_->stream));
}

Stream::~Stream() {
  if (stream_->stream) {
    hipStreamDestroy(stream_->stream);
  }
}

void Stream::wait() {
  CUDA_CHECK(hipStreamSynchronize(stream_->stream));
}

void Stream::prefetch(Device* device, void* ptr, size_t size) {
  CUDA_CHECK(hipMemPrefetchAsync(
      ptr, size, device ? device->deviceId : hipCpuDeviceId, stream_->stream));
}

void Stream::memset(void* ptr, int32_t value, size_t size) {
  CUDA_CHECK(hipMemsetAsync(ptr, value, size, stream_->stream));
}

void Stream::hostToDeviceAsync(
    void* deviceAddress,
    const void* hostAddress,
    size_t size) {
  CUDA_CHECK(hipMemcpyAsync(
      deviceAddress,
      hostAddress,
      size,
      hipMemcpyHostToDevice,
      stream_->stream));
  isTransfer_ = true;
}

void Stream::deviceToHostAsync(
    void* hostAddress,
    const void* deviceAddress,
    size_t size) {
  CUDA_CHECK(hipMemcpyAsync(
      hostAddress,
      deviceAddress,
      size,
      hipMemcpyDeviceToHost,
      stream_->stream));
}

namespace {
struct CallbackData {
  CallbackData(std::function<void()> callback)
      : callback(std::move(callback)){};
  std::function<void()> callback;
};

void readyCallback(void* voidData) {
  std::unique_ptr<CallbackData> data(reinterpret_cast<CallbackData*>(voidData));
  data->callback();
}
} // namespace

void Stream::addCallback(std::function<void()> callback) {
  auto cdata = new CallbackData(std::move(callback));
  CUDA_CHECK(hipLaunchHostFunc(stream_->stream, readyCallback, cdata));
}

struct EventImpl {
  ~EventImpl() {
    auto err = hipEventDestroy(event);
    if (err != hipSuccess) {
      // Do not throw because it can shadow other more important exceptions.  As
      // a rule of thumb, we should not throw in any destructors.
      LOG(ERROR) << "hipEventDestroy: " << hipGetErrorString(err);
    }
  }
  hipEvent_t event;
};

Event::Event(bool withTime) : hasTiming_(withTime) {
  event_ = std::make_unique<EventImpl>();
  CUDA_CHECK(hipEventCreateWithFlags(
      &event_->event, withTime ? 0 : hipEventDisableTiming));
}

Event::~Event() {}

void Event::record(Stream& stream) {
  CUDA_CHECK(hipEventRecord(event_->event, stream.stream_->stream));
  recorded_ = true;
}

void Event::wait() {
  CUDA_CHECK(hipEventSynchronize(event_->event));
}

bool Event::query() const {
  auto rc = hipEventQuery(event_->event);
  if (rc == ::hipErrorNotReady) {
    return false;
  }
  CUDA_CHECK(rc);
  return true;
}

void Event::wait(Stream& stream) {
  CUDA_CHECK(hipStreamWaitEvent(stream.stream_->stream, event_->event));
}

/// Returns time in ms betweene 'this' and an earlier 'start'. Both events must
/// enable timing.
float Event::elapsedTime(const Event& start) const {
  float ms;
  if (!hasTiming_ || !start.hasTiming_) {
    waveError("Event timing not enabled");
  }
  CUDA_CHECK(hipEventElapsedTime(&ms, start.event_->event, event_->event));
  return ms;
}
namespace {
struct KernelEntry {
  const char* name;
  const void* func;
};

int32_t numKernelEntries = 0;
KernelEntry kernelEntries[200];
} // namespace

bool registerKernel(const char* name, const void* func) {
  kernelEntries[numKernelEntries].name = name;
  kernelEntries[numKernelEntries].func = func;
  ++numKernelEntries;
  if (numKernelEntries >= sizeof(kernelEntries) / sizeof(kernelEntries[0])) {
    LOG(ERROR) << "Reserve more space in kernelEntries";
    exit(1);
  }
  return true;
}

KernelInfo kernelInfo(const void* func) {
  hipFuncAttributes attrs;
  CUDA_CHECK_FATAL(hipFuncGetAttributes(&attrs, reinterpret_cast<const void*>(func)));
  KernelInfo info;
  info.numRegs = attrs.numRegs;
  info.maxThreadsPerBlock = attrs.maxThreadsPerBlock;
  info.sharedMemory = attrs.sharedSizeBytes;
  int max;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&max, func, 256, 0);
  info.maxOccupancy0 = max;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&max, func, 256, 256 * 32);
  info.maxOccupancy32 = max;

  return info;
}

std::string KernelInfo::toString() const {
  std::stringstream out;
  out << "NumRegs=" << numRegs << " maxThreadsPerBlock= " << maxThreadsPerBlock
      << " sharedMemory=" << sharedMemory
      << " occupancy 256,  0=" << maxOccupancy0
      << " occupancy 256,32=" << maxOccupancy32;
  return out.str();
}

KernelInfo getRegisteredKernelInfo(const char* name) {
  for (auto i = 0; i < numKernelEntries; ++i) {
    if (strcmp(name, kernelEntries[i].name) == 0) {
      return kernelInfo(kernelEntries[i].func);
    }
  }
  return KernelInfo();
}

void printKernels() {
  for (auto i = 0; i < numKernelEntries; ++i) {
    std::cout << kernelEntries[i].name << " - "
              << getRegisteredKernelInfo(kernelEntries[i].name).toString()
              << std::endl;
  }
}

} // namespace facebook::velox::wave
