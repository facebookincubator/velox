/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <fmt/format.h>
#include "velox/experimental/wave/common/Cuda.h"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/Exception.h"

namespace facebook::velox::wave {

void cudaCheck(hipError_t err, const char* file, int line) {
  if (err == hipSuccess) {
    return;
  }
  waveError(
      fmt::format("Cuda error: {}:{} {}", file, line, hipGetErrorString(err)));
}

namespace {
class CudaManagedAllocator : public GpuAllocator {
 public:
  void* allocate(size_t size) override {
    void* ret;
    CUDA_CHECK(hipMallocManaged(&ret, size));
    return ret;
  }

  void free(void* ptr, size_t /*size*/) override {
    hipFree(ptr);
  }
};

class CudaDeviceAllocator : public GpuAllocator {
 public:
  void* allocate(size_t size) override {
    void* ret;
    CUDA_CHECK(hipMalloc(&ret, size));
    return ret;
  }

  void free(void* ptr, size_t /*size*/) override {
    hipFree(ptr);
  }
};

class CudaHostAllocator : public GpuAllocator {
 public:
  void* allocate(size_t size) override {
    void* ret;
    CUDA_CHECK(hipHostMalloc(&ret, size));
    return ret;
  }

  void free(void* ptr, size_t /*size*/) override {
    hipHostFree(ptr);
  };
};

} // namespace

GpuAllocator* getAllocator(Device* /*device*/) {
  static auto* allocator = new CudaManagedAllocator();
  return allocator;
}

GpuAllocator* getDeviceAllocator(Device* /*device*/) {
  static auto* allocator = new CudaDeviceAllocator();
  return allocator;
}
GpuAllocator* getHostAllocator(Device* /*device*/) {
  static auto* allocator = new CudaHostAllocator();
  return allocator;
}

// Always returns device 0.
Device* getDevice(int32_t /*preferredDevice*/) {
  static Device device(0);
  return &device;
}

void setDevice(Device* device) {
  CUDA_CHECK(hipSetDevice(device->deviceId));
}

Stream::Stream() {
  stream_ = std::make_unique<StreamImpl>();
  CUDA_CHECK(hipStreamCreate(&stream_->stream));
}

Stream::~Stream() {
  hipStreamDestroy(stream_->stream);
}

void Stream::wait() {
  CUDA_CHECK(hipStreamSynchronize(stream_->stream));
}

void Stream::prefetch(Device* device, void* ptr, size_t size) {
  CUDA_CHECK(hipMemPrefetchAsync(
      ptr, size, device ? device->deviceId : hipCpuDeviceId, stream_->stream));
}

void Stream::hostToDeviceAsync(
    void* deviceAddress,
    const void* hostAddress,
    size_t size) {
  CUDA_CHECK(hipMemcpyAsync(
      deviceAddress,
      hostAddress,
      size,
      hipMemcpyHostToDevice,
      stream_->stream));
}

void Stream::deviceToHostAsync(
    void* hostAddress,
    const void* deviceAddress,
    size_t size) {
  CUDA_CHECK(hipMemcpyAsync(
      hostAddress,
      deviceAddress,
      size,
      hipMemcpyDeviceToHost,
      stream_->stream));
}

namespace {
struct CallbackData {
  CallbackData(std::function<void()> callback)
      : callback(std::move(callback)){};
  std::function<void()> callback;
};

void readyCallback(void* voidData) {
  std::unique_ptr<CallbackData> data(reinterpret_cast<CallbackData*>(voidData));
  data->callback();
}
} // namespace

void Stream::addCallback(std::function<void()> callback) {
  auto cdata = new CallbackData(std::move(callback));
  CUDA_CHECK(hipLaunchHostFunc(stream_->stream, readyCallback, cdata));
}

struct EventImpl {
  ~EventImpl() {
    auto err = hipEventDestroy(event);
    if (err != hipSuccess) {
      // Do not throw because it can shadow other more important exceptions.  As
      // a rule of thumb, we should not throw in any destructors.
      LOG(ERROR) << "hipEventDestroy: " << hipGetErrorString(err);
    }
  }
  hipEvent_t event;
};

Event::Event(bool withTime) : hasTiming_(withTime) {
  event_ = std::make_unique<EventImpl>();
  CUDA_CHECK(hipEventCreateWithFlags(
      &event_->event, withTime ? 0 : hipEventDisableTiming));
}

Event::~Event() {}

void Event::record(Stream& stream) {
  CUDA_CHECK(hipEventRecord(event_->event, stream.stream_->stream));
  recorded_ = true;
}

void Event::wait() {
  CUDA_CHECK(hipEventSynchronize(event_->event));
}

bool Event::query() const {
  auto rc = hipEventQuery(event_->event);
  if (rc == ::hipErrorNotReady) {
    return false;
  }
  CUDA_CHECK(rc);
  return true;
}

void Event::wait(Stream& stream) {
  CUDA_CHECK(hipStreamWaitEvent(stream.stream_->stream, event_->event));
}

/// Returns time in ms betweene 'this' and an earlier 'start'. Both events must
/// enable timing.
float Event::elapsedTime(const Event& start) const {
  float ms;
  if (!hasTiming_ || !start.hasTiming_) {
    waveError("Event timing not enabled");
  }
  CUDA_CHECK(hipEventElapsedTime(&ms, start.event_->event, event_->event));
  return ms;
}

} // namespace facebook::velox::wave
