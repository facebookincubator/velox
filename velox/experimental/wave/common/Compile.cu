#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <fmt/format.h>
#include <gflags/gflags.h>
#include <hip/hiprtc.h>
#include "velox/experimental/wave/common/Cuda.h"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/Exception.h"

DEFINE_string(
    wavegen_architecture,
    "compute_70",
    "--gpu-architecture flag for generated code");

namespace facebook::velox::wave {

void nvrtcCheck(hiprtcResult result) {
  if (result != HIPRTC_SUCCESS) {
    waveError(hiprtcGetErrorString(result));
  }
}

class CompiledModuleImpl : public CompiledModule {
 public:
  CompiledModuleImpl(hipModule_t module, std::vector<hipFunction_t> kernels)
      : module_(module), kernels_(std::move(kernels)) {}

  ~CompiledModuleImpl() {
    auto result = hipModuleUnload(module_);
    if (result != hipSuccess) {
      LOG(ERROR) << "Error in unloading module " << result;
    }
  }

  void launch(
      int32_t kernelIdx,
      int32_t numBlocks,
      int32_t numThreads,
      int32_t shared,
      Stream* stream,
      void** args) override;

  KernelInfo info(int32_t kernelIdx) override;

 private:
  hipModule_t module_;
  std::vector<hipFunction_t> kernels_;
};

void addFlag(
    const char* flag,
    const char* value,
    int32_t length,
    std::vector<std::string>& data) {
  std::string str(flag);
  str.resize(str.size() + length + 1);
  memcpy(str.data() + strlen(flag), value, length);
  str.back() = 0;
  data.push_back(std::move(str));
}

// Gets compiler options from the environment and appends  them  to 'opts''. The
// memory is owned by  'data'.
void getNvrtcOptions(
    std::vector<const char*>& opts,
    std::vector<std::string>& data) {
  const char* includes = getenv("WAVE_NVRTC_INCLUDE_PATH");
  if (includes && strlen(includes) > 0) {
    for (;;) {
      const char* end = strchr(includes, ':');
      if (!end) {
        addFlag("-I", includes, strlen(includes), data);
        break;
      }
      addFlag("-I", includes, end - includes, data);
      includes = end + 1;
    }
  }
  const char* flags = getenv("WAVE_NVRTC_FLAGS");
  if (flags && strlen(flags)) {
    for (;;) {
      auto end = strchr(flags, ' ');
      if (!end) {
        addFlag("", flags, strlen(flags), data);
        break;
      }
      addFlag("", flags, end - flags, data);
      flags = end + 1;
    }
  }
  for (auto& str : data) {
    opts.push_back(str.data());
  }
}

std::shared_ptr<CompiledModule> CompiledModule::create(const KernelSpec& spec) {
  hiprtcProgram prog;
  hiprtcCreateProgram(
      &prog,
      spec.code.c_str(), // buffer
      spec.filePath.c_str(), // name
      spec.numHeaders, // numHeaders
      spec.headers, // headers
      spec.headerNames); // includeNames
  for (auto& name : spec.entryPoints) {
    nvrtcCheck(hiprtcAddNameExpression(prog, name.c_str()));
  }
  std::vector<const char*> opts;
  std::vector<std::string> optsData;
#ifndef NDEBUG
  optsData.push_back("-G");
#else
  optsData.push_back("-O3");
#endif
  getNvrtcOptions(opts, optsData);

  auto compileResult = hiprtcCompileProgram(
      prog, // prog
      opts.size(), // numOptions
      opts.data()); // options

  size_t logSize;

  hiprtcGetProgramLogSize(prog, &logSize);
  std::string log;
  log.resize(logSize);
  hiprtcGetProgramLog(prog, log.data());

  if (compileResult != HIPRTC_SUCCESS) {
    hiprtcDestroyProgram(&prog);
    waveError(std::string("Cuda compilation error: ") + log);
  }
  // Obtain PTX from the program.
  size_t ptxSize;
  nvrtcCheck(hiprtcGetCodeSize(prog, &ptxSize));
  std::string ptx;
  ptx.resize(ptxSize);
  nvrtcCheck(hiprtcGetCode(prog, ptx.data()));
  std::vector<std::string> loweredNames;
  for (auto& entry : spec.entryPoints) {
    const char* temp;
    nvrtcCheck(hiprtcGetLoweredName(prog, entry.c_str(), &temp));
    loweredNames.push_back(std::string(temp));
  }

  hiprtcDestroyProgram(&prog);
  hipJitOption options[] = {
      hipJitOptionInfoLogBuffer,
      hipJitOptionInfoLogBufferSizeBytes,
      hipJitOptionErrorLogBuffer,
      hipJitOptionErrorLogBufferSizeBytes};
  char info[1024];
  char error[1024];
  uint32_t infoSize = sizeof(info);
  uint32_t errorSize = sizeof(error);
  void* values[] = {info, &infoSize, error, &errorSize};

  hipModule_t module;
  auto loadResult = hipModuleLoadDataEx(
      &module, ptx.data(), sizeof(values) / sizeof(void*), options, values);
  if (loadResult != hipSuccess) {
    LOG(ERROR) << "Load error " << errorSize << " " << infoSize;
    waveError(fmt::format("Error in load module: {} {}", info, error));
  }
  std::vector<hipFunction_t> funcs;
  for (auto& name : loweredNames) {
    funcs.emplace_back();
    CU_CHECK(hipModuleGetFunction(&funcs.back(), module, name.c_str()));
  }
  return std::make_shared<CompiledModuleImpl>(module, std::move(funcs));
}

void CompiledModuleImpl::launch(
    int32_t kernelIdx,
    int32_t numBlocks,
    int32_t numThreads,
    int32_t shared,
    Stream* stream,
    void** args) {
  auto result = hipModuleLaunchKernel(
      kernels_[kernelIdx],
      numBlocks,
      1,
      1, // grid dim
      numThreads,
      1,
      1, // block dim
      shared,
      reinterpret_cast<hipStream_t>(stream->stream()->stream),
      args,
      0);
  CU_CHECK(result);
};

KernelInfo CompiledModuleImpl::info(int32_t kernelIdx) {
  KernelInfo info;
  auto f = kernels_[kernelIdx];
  hipFuncGetAttribute(&info.numRegs, HIP_FUNC_ATTRIBUTE_NUM_REGS, f);
  hipFuncGetAttribute(
      &info.sharedMemory, HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, f);
  hipFuncGetAttribute(
      &info.maxThreadsPerBlock, HIP_FUNC_ATTRIBUTE_MAX_THREADS_PER_BLOCK, f);
  int32_t max;
  hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(&max, f, 256, 0);
  info.maxOccupancy0 = max;
  hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(&max, f, 256, 256 * 32);
  info.maxOccupancy32 = max;
  return info;
}

} // namespace facebook::velox::wave
