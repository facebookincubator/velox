#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <folly/init/Init.h>
#include <gflags/gflags.h>
#include <cuda/atomic> // @manual
#include "velox/experimental/gpu/Common.h"

DEFINE_int32(grid_size, 1024, "");
DEFINE_int32(node_count, 4000, "");
DEFINE_int32(host_threads, 10, "");
DEFINE_int32(running_time_seconds, 5, "");
DEFINE_int32(backoff_initial_nanoseconds, 8, "");

struct Node {
  cuda::atomic<int32_t> dependencies;
};

struct States {
  int backoffInitialNanoseconds;
  cuda::atomic<bool> running;
  cuda::atomic<int64_t> start;
  cuda::atomic<int64_t> finish;
};

__device__ int findReadyNode(Node* nodes, int size, States& states) {
  int backoff = states.backoffInitialNanoseconds;
  while (states.running) {
    for (int i = 0; i < size; ++i) {
      int ready = 0;
      if (nodes[i].dependencies.compare_exchange_strong(ready, -1)) {
        return i;
      }
    }
    __nanosleep(backoff);
    backoff *= 2;
  }
  return -1;
}

int findAvailableNode(Node* nodes, int size, States& states) {
  int backoff = states.backoffInitialNanoseconds;
  while (states.running) {
    for (int i = 0; i < size; ++i) {
      int avail = -1;
      if (nodes[i].dependencies.compare_exchange_strong(avail, 0)) {
        ++states.start;
        return i;
      }
    }
    std::this_thread::sleep_for(std::chrono::nanoseconds(backoff));
    backoff *= 2;
  }
  return -1;
}

__global__ void pickUpWork(Node* nodes, int size, States& states) {
  __shared__ int nodeIndex;
  // if (threadIdx.x == 0) {
  //   printf("Starting running block %d\n", blockIdx.x);
  // }
  while (states.running) {
    if (threadIdx.x == 0) {
      nodeIndex = findReadyNode(nodes, size, states);
    }
    __syncthreads();
    if (nodeIndex == -1) {
      return;
    }
    // printf("Pick up work for node %d on block %d thread %d\n", nodeIndex,
    // blockIdx.x, threadIdx.x);
    __syncthreads();
    if (threadIdx.x == 0) {
      ++states.finish;
    }
  }
}

void schedule(Node* nodes, int size, States* states) {
  while (states->running) {
    findAvailableNode(nodes, size, *states);
  }
}

int main(int argc, char** argv) {
  using namespace facebook::velox::gpu;
  folly::init(&argc, &argv);
  Node* nodes;
  CUDA_CHECK_FATAL(hipMallocManaged(&nodes, FLAGS_node_count * sizeof(Node)));
  for (int i = 0; i < FLAGS_node_count; ++i) {
    nodes[i].dependencies = -1;
  }
  States* states;
  CUDA_CHECK_FATAL(hipMallocManaged(&states, sizeof(States)));
  states->backoffInitialNanoseconds = FLAGS_backoff_initial_nanoseconds;
  states->running = true;
  states->start = 0;
  states->finish = 0;
  pickUpWork<<<FLAGS_grid_size, 32>>>(nodes, FLAGS_node_count, *states);
  CUDA_CHECK_FATAL(hipGetLastError());
  std::vector<std::thread> threads;
  for (int i = 0; i < FLAGS_host_threads; ++i) {
    threads.emplace_back(schedule, nodes, FLAGS_node_count, states);
  }
  std::chrono::seconds runningTime(FLAGS_running_time_seconds);
  std::this_thread::sleep_for(runningTime);
  states->running = false;
  for (auto& t : threads) {
    t.join();
  }
  CUDA_CHECK_FATAL(hipDeviceSynchronize());
  printf(
      "Started: %ld, Finished: %ld\n",
      states->start.load(),
      states->finish.load());
  printf(
      "%.2f ns per node\n",
      1.0 * std::chrono::nanoseconds(runningTime).count() /
          states->finish.load());
  CUDA_CHECK_LOG(hipFree(states));
  CUDA_CHECK_LOG(hipFree(nodes));
  return 0;
}
