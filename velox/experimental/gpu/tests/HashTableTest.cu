#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <folly/init/Init.h>
#include <gflags/gflags.h>
#include <algorithm>
#include <cassert>
#include <hipcub/hipcub.hpp> // @manual
#include <random>

#include "velox/experimental/gpu/Common.h"

DEFINE_int32(device, 0, "");
DEFINE_int64(table_size, 8 << 20, "For large table, use 268435456");
DEFINE_bool(use_tags, false, "");
DEFINE_bool(partitioned, false, "");
DEFINE_double(occupancy, 0.5, "");
DEFINE_double(matching_rate, 0.5, "");

namespace facebook::velox::gpu {
namespace {

constexpr int kBlockSize = 256;

__device__ uint32_t jenkinsRevMix32(uint32_t key) {
  key += (key << 12); // key *= (1 + (1 << 12))
  key ^= (key >> 22);
  key += (key << 4); // key *= (1 + (1 << 4))
  key ^= (key >> 9);
  key += (key << 10); // key *= (1 + (1 << 10))
  key ^= (key >> 2);
  // key *= (1 + (1 << 7)) * (1 + (1 << 12))
  key += (key << 7);
  key += (key << 12);
  return key;
}

__device__ uint64_t twangMix64(uint64_t key) {
  key = (~key) + (key << 21); // key *= (1 << 21) - 1; key -= 1;
  key = key ^ (key >> 24);
  key = key + (key << 3) + (key << 8); // key *= 1 + (1 << 3) + (1 << 8)
  key = key ^ (key >> 14);
  key = key + (key << 2) + (key << 4); // key *= 1 + (1 << 2) + (1 << 4)
  key = key ^ (key >> 28);
  key = key + (key << 31); // key *= 1 + (1 << 31)
  return key;
}

// Must be avalanching to ensure tag performance.
template <typename T>
__device__ uint64_t hashInt(T value) {
  return twangMix64(value);
}

__device__ uint8_t hashTag(uint64_t hash) {
  return static_cast<uint8_t>(hash >> 32) | 0x80;
}

__device__ uint8_t
atomicCASByte(uint8_t* address, uint8_t compare, uint8_t val) {
  using T = uint8_t;
  using T_int = unsigned int;

  T_int shift = ((reinterpret_cast<size_t>(address) & 3) * 8);
  T_int* address_uint32 = reinterpret_cast<T_int*>(
      address - (reinterpret_cast<size_t>(address) & 3));

  // the 'target_value' in `old` can be different from `compare`
  // because other thread may update the value
  // before fetching a value from `address_uint32` in this function
  T_int old = *address_uint32;
  T_int assumed;
  T target_value;

  do {
    assumed = old;
    target_value = T((old >> shift) & 0xff);
    // have to compare `target_value` and `compare` before calling atomicCAS
    // the `target_value` in `old` can be different with `compare`
    if (target_value != compare) {
      break;
    }

    T_int new_value = (old & ~(0x000000ff << shift)) | (T_int(val) << shift);
    old = atomicCAS(address_uint32, assumed, new_value);
  } while (assumed != old);

  return target_value;
}

CudaPtr<uint64_t[]> generateBuildKeys(
    std::default_random_engine& gen,
    std::vector<uint64_t>& keys) {
  std::iota(keys.begin(), keys.end(), 0);
  std::shuffle(keys.begin(), keys.end(), gen);
  auto keysD = allocateDeviceMemory<uint64_t>(keys.size());
  CUDA_CHECK_FATAL(hipMemcpy(
      keysD.get(),
      keys.data(),
      keys.size() * sizeof(uint64_t),
      hipMemcpyHostToDevice));
  return keysD;
}

void generateProbeKeys(
    std::default_random_engine& gen,
    std::vector<uint64_t>& keysHost,
    uint64_t* keysDevice) {
  std::shuffle(keysHost.begin(), keysHost.end(), gen);
  std::uniform_int_distribution<uint64_t> dist(keysHost.size());
  for (auto i = keysHost.size() * FLAGS_matching_rate; i < keysHost.size();
       ++i) {
    keysHost[i] = dist(gen);
  }
  std::shuffle(keysHost.begin(), keysHost.end(), gen);
  CUDA_CHECK_FATAL(hipMemcpy(
      keysDevice,
      keysHost.data(),
      keysHost.size() * sizeof(uint64_t),
      hipMemcpyHostToDevice));
}

struct HashTable {
  int64_t size;
  uint64_t* keys;
  uint64_t* values;
  uint64_t emptyMarker;
  int hasEmptyValue;
  uint64_t emptyValue;
  uint8_t* tags;
};

template <bool kUseTags>
__global__ void init(HashTable* table) {
  int64_t step = gridDim.x * blockDim.x;
  for (int64_t i = threadIdx.x + blockIdx.x * blockDim.x; i < table->size;
       i += step) {
    table->keys[i] = table->emptyMarker;
    if constexpr (kUseTags) {
      table->tags[i] = 0;
    }
  }
}

template <bool kUseTags>
__global__ void build(
    HashTable* table,
    const uint64_t* keys,
    const uint64_t* values,
    int64_t size);

template <>
__global__ void build<false>(
    HashTable* table,
    const uint64_t* keys,
    const uint64_t* values,
    int64_t size) {
  uint64_t tableSizeMask = table->size - 1;
  int64_t step = gridDim.x * blockDim.x;
  for (int64_t i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i += step) {
    if (__builtin_expect(keys[i] == table->emptyMarker, false)) {
      if (atomicCAS(&table->hasEmptyValue, 0, 1) == 0) {
        table->emptyValue = values[i];
      } else {
        printf("ERROR: Duplicate key %llu\n", keys[i]);
      }
      continue;
    }
    auto hash = hashInt(keys[i]);
    for (auto j = hash & tableSizeMask;; j = (j + 1) & tableSizeMask) {
      if (table->keys[j] == table->emptyMarker &&
          atomicCAS(
              (unsigned long long*)&table->keys[j],
              table->emptyMarker,
              keys[i]) == table->emptyMarker) {
        table->values[j] = values[i];
        break;
      }
      if (table->keys[j] == keys[i]) {
        printf("ERROR: Duplicate key %llu\n", keys[i]);
        break;
      }
    }
  }
}

template <>
__global__ void build<true>(
    HashTable* table,
    const uint64_t* keys,
    const uint64_t* values,
    int64_t size) {
  uint64_t tableSizeMask = table->size - 1;
  int64_t step = gridDim.x * blockDim.x;
  for (int64_t i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i += step) {
    auto hash = hashInt(keys[i]);
    auto tag = hashTag(hash);
    for (auto j = hash & tableSizeMask;; j = (j + 1) & tableSizeMask) {
      if (table->tags[j] == 0 && atomicCASByte(&table->tags[j], 0, tag) == 0) {
        table->keys[j] = keys[i];
        table->values[j] = values[i];
        break;
      }
      if (table->keys[j] == keys[i]) {
        printf("ERROR: Duplicate key %llu\n", keys[i]);
        break;
      }
    }
  }
}

template <bool kUseTags>
__global__ void probe(
    const HashTable* table,
    const uint64_t* keys,
    uint64_t* values,
    bool* hasValue,
    int64_t size);

template <>
__global__ void probe<false>(
    const HashTable* table,
    const uint64_t* keys,
    uint64_t* values,
    bool* hasValue,
    int64_t size) {
  uint64_t tableSizeMask = table->size - 1;
  int64_t step = gridDim.x * blockDim.x;
  for (int64_t i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i += step) {
    if (__builtin_expect(keys[i] == table->emptyMarker, false)) {
      hasValue[i] = table->hasEmptyValue;
      if (table->hasEmptyValue) {
        values[i] = table->emptyValue;
      }
      continue;
    }
    auto hash = hashInt(keys[i]);
    for (auto j = hash & tableSizeMask;; j = (j + 1) & tableSizeMask) {
      if (table->keys[j] == keys[i]) {
        hasValue[i] = true;
        values[i] = table->values[j];
        break;
      }
      if (table->keys[j] == table->emptyMarker) {
        hasValue[i] = false;
        break;
      }
    }
  }
}

template <>
__global__ void probe<true>(
    const HashTable* table,
    const uint64_t* keys,
    uint64_t* values,
    bool* hasValue,
    int64_t size) {
  uint64_t tableSizeMask = table->size - 1;
  int64_t step = gridDim.x * blockDim.x;
  for (int64_t i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i += step) {
    auto hash = hashInt(keys[i]);
    uint32_t tag = hashTag(hash);
    tag = tag | (tag << 8);
    tag = tag | (tag << 16);
    hash &= tableSizeMask;
    auto rem = hash % sizeof(uint32_t);
    int64_t j = hash - rem;
    uint32_t cmpMask = 0xffffffff << (rem * 8);
    for (;;) {
      auto hits = __vcmpeq4(*(uint32_t*)&table->tags[j], tag) & cmpMask;
      while (hits) {
        auto jj = j + (__ffs(hits) - 1) / 8;
        if (table->keys[jj] == keys[i]) {
          hasValue[i] = true;
          values[i] = table->values[jj];
          goto end;
        }
        hits &= hits - 1;
      }
      if (__vcmpeq4(*(uint32_t*)&table->tags[j], 0) & cmpMask) {
        hasValue[i] = false;
        goto end;
      }
      j = (j + sizeof(uint32_t)) & tableSizeMask;
      cmpMask = 0xffffffff;
    }
  end:
  }
}

__global__ void validate(
    const uint64_t* keys,
    const uint64_t* result,
    const bool* hasResult,
    int64_t size) {
  int64_t step = gridDim.x * blockDim.x;
  for (int64_t i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i += step) {
    if (keys[i] < (uint64_t)size) {
      if (!hasResult[i]) {
        printf("ERROR: Result missing %llu\n", keys[i]);
      } else if (result[i] != keys[i]) {
        printf("ERROR: Result mismatch %llu != %llu\n", result[i], keys[i]);
      }
    } else if (hasResult[i]) {
      printf("ERROR: Unexpected result %llu\n", keys[i]);
    }
  }
}

template <bool kUseTags>
void run() {
  auto tableKeys = allocateDeviceMemory<uint64_t>(FLAGS_table_size);
  auto tableValues = allocateDeviceMemory<uint64_t>(FLAGS_table_size);
  CudaPtr<uint8_t[]> tags;
  auto table = allocateManagedMemory<HashTable>();
  table->size = FLAGS_table_size;
  table->keys = tableKeys.get();
  table->values = tableValues.get();
  table->emptyMarker = 0xdeadbeefbadefeedULL;
  table->hasEmptyValue = 0;
  if constexpr (kUseTags) {
    tags = allocateDeviceMemory<uint8_t>(FLAGS_table_size);
    table->tags = tags.get();
  }

  float time;
  auto startEvent = createCudaEvent();
  auto stopEvent = createCudaEvent();
  int64_t numKeys = FLAGS_table_size * FLAGS_occupancy;
  auto numBlocks = FLAGS_table_size / kBlockSize;
  std::default_random_engine gen(std::random_device{}());

  std::vector<uint64_t> keysHost(numKeys);
  auto keys = generateBuildKeys(gen, keysHost);
  init<kUseTags><<<numBlocks, kBlockSize>>>(table.get());

  CUDA_CHECK_FATAL(hipEventRecord(startEvent.get()));
  build<kUseTags>
      <<<numBlocks, kBlockSize>>>(table.get(), keys.get(), keys.get(), numKeys);
  CUDA_CHECK_FATAL(hipEventRecord(stopEvent.get()));
  CUDA_CHECK_FATAL(hipEventSynchronize(stopEvent.get()));
  CUDA_CHECK_FATAL(
      hipEventElapsedTime(&time, startEvent.get(), stopEvent.get()));
  printf("Hash build: %.2f billion rows/s\n", numKeys * 1e-6 / time);

  generateProbeKeys(gen, keysHost, keys.get());
  auto result = allocateDeviceMemory<uint64_t>(numKeys);
  auto hasResult = allocateDeviceMemory<bool>(numKeys);

  CUDA_CHECK_FATAL(hipEventRecord(startEvent.get()));
  probe<kUseTags><<<numBlocks, kBlockSize>>>(
      table.get(), keys.get(), result.get(), hasResult.get(), numKeys);
  CUDA_CHECK_FATAL(hipEventRecord(stopEvent.get()));
  CUDA_CHECK_FATAL(hipEventSynchronize(stopEvent.get()));
  CUDA_CHECK_FATAL(
      hipEventElapsedTime(&time, startEvent.get(), stopEvent.get()));
  printf("Hash probe: %.2f billion rows/s\n", numKeys * 1e-6 / time);

  validate<<<numBlocks, kBlockSize>>>(
      keys.get(), result.get(), hasResult.get(), numKeys);
  CUDA_CHECK_FATAL(hipGetLastError());
  CUDA_CHECK_FATAL(hipEventRecord(stopEvent.get()));
  CUDA_CHECK_FATAL(hipEventSynchronize(stopEvent.get()));
}

__global__ void computeHistogram(
    const uint64_t* keys,
    int64_t size,
    uint64_t mask,
    int64_t* hist) {
  int64_t step = gridDim.x * blockDim.x;
  for (int64_t i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i += step) {
    atomicAdd((unsigned long long*)&hist[hashInt(keys[i]) & mask], 1);
  }
}

__global__ void shuffle(
    const uint64_t* keys,
    int64_t size,
    uint64_t mask,
    int64_t* offsets,
    uint64_t* out) {
  int64_t step = gridDim.x * blockDim.x;
  for (int64_t i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i += step) {
    auto k = keys[i];
    auto j = atomicAdd((unsigned long long*)&offsets[hashInt(k) & mask], 1);
    __stwt(&out[j], k);
  }
}

void partitionKeys(
    const uint64_t* keys,
    int64_t size,
    int64_t numPartitions,
    int64_t* hist,
    int64_t* offsets,
    uint64_t* out,
    char* tmp,
    size_t tmpSize) {
  CUDA_CHECK_FATAL(hipMemset(hist, 0, numPartitions * sizeof(int64_t)));
  computeHistogram<<<(size + kBlockSize - 1) / kBlockSize, kBlockSize>>>(
      keys, size, numPartitions - 1, hist);
  CUDA_CHECK_FATAL(hipMemset(offsets, 0, sizeof(int64_t)));
  CUDA_CHECK_FATAL(hipcub::DeviceScan::InclusiveSum(
      tmp, tmpSize, hist, offsets + 1, numPartitions));
  CUDA_CHECK_FATAL(hipMemcpy(
      hist,
      offsets,
      numPartitions * sizeof(int64_t),
      hipMemcpyDeviceToDevice));
  shuffle<<<(size + kBlockSize - 1) / kBlockSize, kBlockSize>>>(
      keys, size, numPartitions - 1, hist, out);
}

// One block per partition.
__global__ void validatePartition(
    int maxPartitionSize,
    int64_t numKeys,
    const uint64_t* keys,
    const int64_t* offsets) {
  if (threadIdx.x == 0) {
    if (offsets[blockIdx.x] >= offsets[blockIdx.x + 1]) {
      printf("ERROR: Bad offsets\n");
      return;
    }
    if (offsets[blockIdx.x + 1] - offsets[blockIdx.x] > maxPartitionSize) {
      printf("ERROR: Partition overflow\n");
      return;
    }
    if (blockIdx.x == gridDim.x - 1 && offsets[gridDim.x] != numKeys) {
      printf("ERROR: Wrong total size\n");
      return;
    }
  }
  uint64_t mask = gridDim.x - 1;
  for (auto i = threadIdx.x + offsets[blockIdx.x]; i < offsets[blockIdx.x + 1];
       i += blockDim.x) {
    if ((hashInt(keys[i]) & mask) != blockIdx.x) {
      printf("ERROR: Key %llu in wrong partition %d\n", keys[i], blockIdx.x);
      return;
    }
  }
}

// One block per partition.
template <bool kUseTags>
__global__ void buildPartitioned(
    HashTable* table,
    int partitionSize,
    int shift,
    const uint64_t* keys,
    const uint64_t* values,
    const int64_t* offsets);

template <>
__global__ void buildPartitioned<false>(
    HashTable* table,
    int partitionSize,
    int shift,
    const uint64_t* keys,
    const uint64_t* values,
    const int64_t* offsets) {
  uint64_t tableSizeMask = partitionSize - 1;
  for (auto i = threadIdx.x + offsets[blockIdx.x]; i < offsets[blockIdx.x + 1];
       i += blockDim.x) {
    if (__builtin_expect(keys[i] == table->emptyMarker, false)) {
      if (atomicCAS_block(&table->hasEmptyValue, 0, 1) == 0) {
        table->emptyValue = values[i];
      } else {
        printf("ERROR: Duplicate key %llu\n", keys[i]);
      }
      continue;
    }
    auto hash = hashInt(keys[i]) >> shift;
    for (auto j = hash & tableSizeMask;; j = (j + 1) & tableSizeMask) {
      auto jj = j + partitionSize * blockIdx.x;
      if (table->keys[jj] == table->emptyMarker &&
          atomicCAS_block(
              (unsigned long long*)&table->keys[jj],
              table->emptyMarker,
              keys[i]) == table->emptyMarker) {
        table->values[jj] = values[i];
        break;
      }
      if (table->keys[jj] == keys[i]) {
        printf("ERROR: Duplicate key %llu\n", keys[i]);
        break;
      }
    }
  }
}

template <>
__global__ void buildPartitioned<true>(
    HashTable* table,
    int partitionSize,
    int shift,
    const uint64_t* keys,
    const uint64_t* values,
    const int64_t* offsets) {
  uint64_t tableSizeMask = partitionSize - 1;
  for (auto i = threadIdx.x + offsets[blockIdx.x]; i < offsets[blockIdx.x + 1];
       i += blockDim.x) {
    auto hash = hashInt(keys[i]) >> shift;
    auto tag = hashTag(hash);
    for (auto j = hash & tableSizeMask;; j = (j + 1) & tableSizeMask) {
      auto jj = j + partitionSize * blockIdx.x;
      if (table->tags[jj] == 0 &&
          atomicCASByte(&table->tags[jj], 0, tag) == 0) {
        table->keys[jj] = keys[i];
        table->values[jj] = values[i];
        break;
      }
      if (table->keys[jj] == keys[i]) {
        printf("ERROR: Duplicate key %llu\n", keys[i]);
        break;
      }
    }
  }
}

// One block per partition.
template <bool kUseTags>
__global__ void probePartitioned(
    const HashTable* table,
    int partitionSize,
    int shift,
    const uint64_t* keys,
    const int64_t* offsets,
    uint64_t* values,
    bool* hasValue);

template <>
__global__ void probePartitioned<false>(
    const HashTable* table,
    int partitionSize,
    int shift,
    const uint64_t* keys,
    const int64_t* offsets,
    uint64_t* values,
    bool* hasValue) {
  extern __shared__ uint64_t tableKeys[];
  for (auto i = threadIdx.x; i < partitionSize; i += blockDim.x) {
    tableKeys[i] = table->keys[i + partitionSize * blockIdx.x];
  }
  __syncthreads();
  uint64_t tableSizeMask = partitionSize - 1;
  for (auto i = threadIdx.x + offsets[blockIdx.x]; i < offsets[blockIdx.x + 1];
       i += blockDim.x) {
    if (__builtin_expect(keys[i] == table->emptyMarker, false)) {
      hasValue[i] = table->hasEmptyValue;
      if (table->hasEmptyValue) {
        values[i] = table->emptyValue;
      }
      continue;
    }
    auto hash = hashInt(keys[i]) >> shift;
    for (auto j = hash & tableSizeMask;; j = (j + 1) & tableSizeMask) {
      auto jj = j + partitionSize * blockIdx.x;
      if (tableKeys[j] == keys[i]) {
        hasValue[i] = true;
        values[i] = table->values[jj];
        break;
      }
      if (tableKeys[j] == table->emptyMarker) {
        hasValue[i] = false;
        break;
      }
    }
  }
}

template <>
__global__ void probePartitioned<true>(
    const HashTable* table,
    int partitionSize,
    int shift,
    const uint64_t* keys,
    const int64_t* offsets,
    uint64_t* values,
    bool* hasValue) {
  extern __shared__ uint8_t tableTags[];
  for (auto i = threadIdx.x; i < partitionSize; i += blockDim.x) {
    tableTags[i] = table->tags[i + partitionSize * blockIdx.x];
  }
  __syncthreads();
  uint64_t tableSizeMask = partitionSize - 1;
  for (auto i = threadIdx.x + offsets[blockIdx.x]; i < offsets[blockIdx.x + 1];
       i += blockDim.x) {
    auto hash = hashInt(keys[i]) >> shift;
    uint32_t tag = hashTag(hash);
    tag = tag | (tag << 8);
    tag = tag | (tag << 16);
    hash &= tableSizeMask;
    auto rem = hash % sizeof(uint32_t);
    int64_t j = hash - rem;
    uint32_t cmpMask = 0xffffffff << (rem * 8);
    for (;;) {
      auto hits = __vcmpeq4(*(uint32_t*)&tableTags[j], tag) & cmpMask;
      while (hits) {
        auto jj = j + (__ffs(hits) - 1) / 8 + partitionSize * blockIdx.x;
        if (table->keys[jj] == keys[i]) {
          hasValue[i] = true;
          values[i] = table->values[jj];
          goto end;
        }
        hits &= hits - 1;
      }
      if (__vcmpeq4(*(uint32_t*)&tableTags[j], 0) & cmpMask) {
        hasValue[i] = false;
        goto end;
      }
      j = (j + sizeof(uint32_t)) & tableSizeMask;
      cmpMask = 0xffffffff;
    }
  end:
  }
}

template <bool kUseTags>
void runPartitioned() {
  constexpr int kSharedMemorySize = 1 << (kUseTags ? 12 : 16);
  constexpr int kPartitionSize =
      kSharedMemorySize / (kUseTags ? 1 : sizeof(uint64_t));

  float time;
  auto startEvent = createCudaEvent();
  auto stopEvent = createCudaEvent();
  int64_t numKeys = FLAGS_table_size * FLAGS_occupancy;
  int64_t numPartitions = FLAGS_table_size / kPartitionSize;

  auto tableKeys = allocateDeviceMemory<uint64_t>(FLAGS_table_size);
  auto tableValues = allocateDeviceMemory<uint64_t>(FLAGS_table_size);
  auto table = allocateManagedMemory<HashTable>();
  table->size = FLAGS_table_size;
  table->keys = tableKeys.get();
  table->values = tableValues.get();
  table->emptyMarker = 0xdeadbeefbadefeedULL;
  table->hasEmptyValue = 0;
  CudaPtr<uint8_t[]> tags;
  if constexpr (kUseTags) {
    tags = allocateDeviceMemory<uint8_t>(FLAGS_table_size);
    table->tags = tags.get();
  }
  init<kUseTags><<<FLAGS_table_size / kBlockSize, kBlockSize>>>(table.get());

  std::default_random_engine gen(std::random_device{}());
  std::vector<uint64_t> keysHost(numKeys);
  auto keys = generateBuildKeys(gen, keysHost);

  auto hist = allocateDeviceMemory<int64_t>(numPartitions);
  auto offsets = allocateDeviceMemory<int64_t>(numPartitions + 1);
  size_t tmpSize;
  CUDA_CHECK_FATAL(hipcub::DeviceScan::InclusiveSum(
      nullptr, tmpSize, hist.get(), offsets.get(), numPartitions));
  auto tmp = allocateDeviceMemory<char>(tmpSize);
  auto shuffledKeys = allocateDeviceMemory<uint64_t>(numKeys);

  CUDA_CHECK_FATAL(hipEventRecord(startEvent.get()));
  partitionKeys(
      keys.get(),
      numKeys,
      numPartitions,
      hist.get(),
      offsets.get(),
      shuffledKeys.get(),
      tmp.get(),
      tmpSize);
  buildPartitioned<kUseTags><<<numPartitions, 1024>>>(
      table.get(),
      kPartitionSize,
      __builtin_ctz(numPartitions),
      shuffledKeys.get(),
      shuffledKeys.get(),
      offsets.get());
  CUDA_CHECK_FATAL(hipEventRecord(stopEvent.get()));
  CUDA_CHECK_FATAL(hipEventSynchronize(stopEvent.get()));
  CUDA_CHECK_FATAL(
      hipEventElapsedTime(&time, startEvent.get(), stopEvent.get()));
  printf("Hash build: %.2f billion rows/s\n", numKeys * 1e-6 / time);

  validatePartition<<<numPartitions, 1024>>>(
      kPartitionSize, numKeys, shuffledKeys.get(), offsets.get());
  CUDA_CHECK_FATAL(hipGetLastError());
  generateProbeKeys(gen, keysHost, keys.get());
  auto result = allocateDeviceMemory<uint64_t>(numKeys);
  auto hasResult = allocateDeviceMemory<bool>(numKeys);

  CUDA_CHECK_FATAL(hipEventRecord(startEvent.get()));
  partitionKeys(
      keys.get(),
      numKeys,
      numPartitions,
      hist.get(),
      offsets.get(),
      shuffledKeys.get(),
      tmp.get(),
      tmpSize);
  CUDA_CHECK_FATAL(
      hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
  CUDA_CHECK_FATAL(hipFuncSetAttribute(reinterpret_cast<const void*>(
      probePartitioned<kUseTags>),
      hipFuncAttributeMaxDynamicSharedMemorySize,
      kSharedMemorySize));
  probePartitioned<kUseTags><<<numPartitions, 1024, kSharedMemorySize>>>(
      table.get(),
      kPartitionSize,
      __builtin_ctz(numPartitions),
      shuffledKeys.get(),
      offsets.get(),
      result.get(),
      hasResult.get());
  CUDA_CHECK_FATAL(hipEventRecord(stopEvent.get()));
  CUDA_CHECK_FATAL(hipEventSynchronize(stopEvent.get()));
  CUDA_CHECK_FATAL(
      hipEventElapsedTime(&time, startEvent.get(), stopEvent.get()));
  printf("Hash probe: %.2f billion rows/s\n", numKeys * 1e-6 / time);

  validatePartition<<<numPartitions, 1024>>>(
      numKeys, numKeys, shuffledKeys.get(), offsets.get());
  CUDA_CHECK_FATAL(hipGetLastError());
  validate<<<(numKeys + kBlockSize - 1) / kBlockSize, kBlockSize>>>(
      shuffledKeys.get(), result.get(), hasResult.get(), numKeys);
  CUDA_CHECK_FATAL(hipGetLastError());
  CUDA_CHECK_FATAL(hipEventRecord(stopEvent.get()));
  CUDA_CHECK_FATAL(hipEventSynchronize(stopEvent.get()));
}

} // namespace
} // namespace facebook::velox::gpu

int main(int argc, char** argv) {
  using namespace facebook::velox::gpu;
  folly::init(&argc, &argv);
  assert(__builtin_popcount(FLAGS_table_size) == 1);
  assert(FLAGS_table_size % kBlockSize == 0);
  CUDA_CHECK_FATAL(hipSetDevice(FLAGS_device));
  if (FLAGS_partitioned) {
    if (FLAGS_use_tags) {
      runPartitioned<true>();
    } else {
      runPartitioned<false>();
    }
  } else {
    if (FLAGS_use_tags) {
      run<true>();
    } else {
      run<false>();
    }
  }
  return 0;
}
