#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <folly/init/Init.h>
#include <gflags/gflags.h>
#include <hipcub/hipcub.hpp> // @manual
#include <numeric>
#include <thread>
#include "velox/experimental/gpu/Common.h"

DEFINE_int64(buffer_size, 32 << 20, "");
DEFINE_int32(repeat, 100, "");
DEFINE_string(devices, "0", "Comma-separate list of device ids");
DEFINE_bool(validate, false, "");
DEFINE_int32(num_strides, 32, "");

constexpr int kBlockSize = 256;

namespace facebook::velox::gpu {
namespace {

__device__ uint64_t hashMix(uint64_t upper, uint64_t lower) {
  constexpr uint64_t kMul = 0x9ddfea08eb382d69ULL;
  uint64_t a = (lower ^ upper) * kMul;
  a ^= (a >> 47);
  uint64_t b = (upper ^ a) * kMul;
  b ^= (b >> 47);
  b *= kMul;
  return b;
}

__global__ void strideMemory(const int64_t* data, int strides, int64_t* out) {
  using Reduce = hipcub::BlockReduce<int64_t, kBlockSize>;
  __shared__ Reduce::TempStorage tmp;
  int64_t i = threadIdx.x + 1ll * blockIdx.x * blockDim.x;
  auto ans = i;
  for (int j = 0; j < strides; ++j, i = data[i]) {
    ans = hashMix(ans, data[i]);
  }
  out[blockIdx.x] = Reduce(tmp).Reduce(ans, hashMix);
}

void testCudaEvent(int deviceId) {
  CUDA_CHECK_FATAL(hipSetDevice(deviceId));
  auto elementCount = FLAGS_buffer_size / sizeof(int64_t);
  if (__builtin_popcount(elementCount) != 1) {
    abort();
  }
  int64_t* hostBuffer;
  int64_t* deviceBuffer[2];
  int64_t* outputBuffer;
  CUDA_CHECK_FATAL(hipHostMalloc(&hostBuffer, FLAGS_buffer_size));
  CUDA_CHECK_FATAL(hipMalloc(&deviceBuffer[0], FLAGS_buffer_size));
  CUDA_CHECK_FATAL(hipMalloc(&deviceBuffer[1], FLAGS_buffer_size));
  CUDA_CHECK_FATAL(
      hipMalloc(&outputBuffer, sizeof(int64_t) * elementCount / kBlockSize));
  for (int64_t i = 0, j = 1; j <= elementCount; ++j) {
    hostBuffer[i] = (i + j) % elementCount;
    i = hostBuffer[i];
  }
  CudaStream streams[] = {
      createCudaStream(),
      createCudaStream(),
  };
  CudaEvent bufferReady[] = {
      createCudaEvent(),
      createCudaEvent(),
  };
  CudaEvent processDone[] = {
      createCudaEvent(),
      createCudaEvent(),
  };

  auto startEvent = createCudaEvent();
  CUDA_CHECK_FATAL(hipEventRecord(startEvent.get()));
  int loading = 0;
  int processing = 1;
  for (int i = 0; i < FLAGS_repeat; ++i) {
    std::swap(loading, processing);
    if (i > 0) {
      CUDA_CHECK_FATAL(hipEventSynchronize(processDone[loading].get()));
    }
    CUDA_CHECK_FATAL(hipMemcpyAsync(
        deviceBuffer[loading],
        hostBuffer,
        FLAGS_buffer_size,
        hipMemcpyHostToDevice,
        streams[loading].get()));
    CUDA_CHECK_FATAL(
        hipEventRecord(bufferReady[loading].get(), streams[loading].get()));
    if (i > 0) {
      CUDA_CHECK_FATAL(hipEventSynchronize(bufferReady[processing].get()));
      strideMemory<<<elementCount / kBlockSize, kBlockSize>>>(
          deviceBuffer[processing], FLAGS_num_strides, outputBuffer);
      CUDA_CHECK_FATAL(hipGetLastError());
    }
    CUDA_CHECK_FATAL(hipEventRecord(
        processDone[processing].get(), streams[processing].get()));
  }
  auto stopEvent = createCudaEvent();
  CUDA_CHECK_FATAL(hipEventRecord(stopEvent.get()));
  CUDA_CHECK_FATAL(hipEventSynchronize(stopEvent.get()));
  float time;
  CUDA_CHECK_FATAL(
      hipEventElapsedTime(&time, startEvent.get(), stopEvent.get()));
  printf(
      "Device %d memcpy throughput: %.2f GB/s\n",
      deviceId,
      FLAGS_buffer_size * FLAGS_repeat * 1e-6 / time);

  CUDA_CHECK_FATAL(hipEventRecord(startEvent.get()));
  for (int i = 0; i < FLAGS_repeat; ++i) {
    strideMemory<<<elementCount / kBlockSize, kBlockSize>>>(
        deviceBuffer[processing], FLAGS_num_strides, outputBuffer);
  }
  CUDA_CHECK_FATAL(hipEventRecord(stopEvent.get()));
  CUDA_CHECK_FATAL(hipEventSynchronize(stopEvent.get()));
  CUDA_CHECK_FATAL(
      hipEventElapsedTime(&time, startEvent.get(), stopEvent.get()));
  printf(
      "Device %d device memory random read throughput: %.2f GB/s\n",
      deviceId,
      FLAGS_buffer_size * FLAGS_num_strides * FLAGS_repeat * 1e-6 / time);

  CUDA_CHECK_LOG(hipFree(outputBuffer));
  CUDA_CHECK_LOG(hipFree(deviceBuffer[0]));
  CUDA_CHECK_LOG(hipFree(deviceBuffer[1]));
  CUDA_CHECK_LOG(hipHostFree(hostBuffer));
}

} // namespace
} // namespace facebook::velox::gpu

int main(int argc, char** argv) {
  using namespace facebook::velox::gpu;
  folly::init(&argc, &argv);
  int deviceCount;
  CUDA_CHECK_FATAL(hipGetDeviceCount(&deviceCount));
  printf("Device count: %d\n", deviceCount);
  std::vector<int> devices;
  for (int i = 0, deviceId = 0; i < FLAGS_devices.size(); ++i) {
    char c = FLAGS_devices[i];
    if (c != ',') {
      deviceId = 10 * deviceId + (c - '0');
    }
    if (c == ',' || i + 1 == FLAGS_devices.size()) {
      devices.push_back(deviceId);
      deviceId = 0;
    }
  }
  std::vector<std::thread> threads;
  for (int deviceId : devices) {
    threads.emplace_back(testCudaEvent, deviceId);
  }
  for (auto& t : threads) {
    t.join();
  }
  return 0;
}
