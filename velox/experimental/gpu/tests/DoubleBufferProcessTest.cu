#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <folly/init/Init.h>
#include <gflags/gflags.h>
#include <hipcub/hipcub.hpp> // @manual
#include <numeric>
#include "velox/experimental/gpu/Common.h"

DEFINE_int64(buffer_size, 24 << 20, "");
DEFINE_int32(repeat, 100, "");
DEFINE_string(devices, "0", "Comma-separate list of device ids");
DEFINE_bool(validate, false, "");
DEFINE_int32(num_strides, 32, "");

constexpr int kBlockSize = 256;
constexpr int kItemsPerThread = 32;
constexpr int kItemsPerBlock = kBlockSize * kItemsPerThread;

namespace facebook::velox::gpu {
namespace {

struct BlockStorage {
  using Load = hipcub::BlockLoad<
      int64_t,
      kBlockSize,
      kItemsPerThread,
      hipcub::BLOCK_LOAD_TRANSPOSE>;

  using Store = hipcub::BlockStore<
      int64_t,
      kBlockSize,
      kItemsPerThread,
      hipcub::BLOCK_STORE_TRANSPOSE>;

  using Sort = hipcub::BlockRadixSort<int64_t, kBlockSize, kItemsPerThread>;

  union {
    Load::TempStorage load;
    Store::TempStorage store;
    Sort::TempStorage sort;
  };
};

__global__ void sortBlock(int64_t* data) {
  extern __shared__ BlockStorage sharedStorage[];
  int64_t threadKeys[kItemsPerThread];
  int offset = blockIdx.x * kItemsPerBlock;
  BlockStorage::Load(sharedStorage[0].load).Load(data + offset, threadKeys);
  __syncthreads();
  BlockStorage::Sort(sharedStorage[0].sort).Sort(threadKeys);
  __syncthreads();
  BlockStorage::Store(sharedStorage[0].store).Store(data + offset, threadKeys);
}

__device__ uint64_t hashMix(uint64_t upper, uint64_t lower) {
  constexpr uint64_t kMul = 0x9ddfea08eb382d69ULL;
  uint64_t a = (lower ^ upper) * kMul;
  a ^= (a >> 47);
  uint64_t b = (upper ^ a) * kMul;
  b ^= (b >> 47);
  b *= kMul;
  return b;
}

__global__ void
strideMemory(const int64_t* data, int64_t count, int strides, int64_t* out) {
  using Reduce = hipcub::BlockReduce<int64_t, kBlockSize>;
  __shared__ Reduce::TempStorage tmp;
  auto i = threadIdx.x + blockIdx.x * blockDim.x;
  int64_t ans = 0;
  for (int j = 0; j < strides; ++j) {
    ans = hashMix(ans, data[i]);
    i = (i + data[i]) % count;
  }
  out[blockIdx.x] = Reduce(tmp).Reduce(ans, hashMix);
}

void testCudaEvent(int deviceId) {
  CUDA_CHECK_FATAL(hipSetDevice(deviceId));
  CUDA_CHECK_FATAL(hipFuncSetAttribute(reinterpret_cast<const void*>(
      sortBlock),
      hipFuncAttributeMaxDynamicSharedMemorySize,
      sizeof(BlockStorage)));
  auto elementCount = FLAGS_buffer_size / sizeof(int64_t);
  if (!(elementCount >= kItemsPerBlock && elementCount % kItemsPerBlock == 0)) {
    abort();
  }
  int64_t* hostBuffer;
  int64_t* deviceBuffer[2];
  int64_t* outputBuffer;
  CUDA_CHECK_FATAL(hipHostMalloc(&hostBuffer, FLAGS_buffer_size));
  CUDA_CHECK_FATAL(hipMalloc(&deviceBuffer[0], FLAGS_buffer_size));
  CUDA_CHECK_FATAL(hipMalloc(&deviceBuffer[1], FLAGS_buffer_size));
  CUDA_CHECK_FATAL(
      hipMalloc(&outputBuffer, sizeof(int64_t) * elementCount / kBlockSize));
  std::iota(hostBuffer, hostBuffer + elementCount, 0);
  std::reverse(hostBuffer, hostBuffer + elementCount);
  CudaStream streams[] = {
      createCudaStream(),
      createCudaStream(),
  };
  CudaEvent bufferReady[] = {
      createCudaEvent(),
      createCudaEvent(),
  };
  CudaEvent processDone[] = {
      createCudaEvent(),
      createCudaEvent(),
  };
  auto startEvent = createCudaEvent();
  CUDA_CHECK_FATAL(hipEventRecord(startEvent.get()));
  int loading = 0;
  int processing = 1;
  for (int i = 0; i < FLAGS_repeat; ++i) {
    std::swap(loading, processing);
    if (i > 0) {
      CUDA_CHECK_FATAL(hipEventSynchronize(processDone[loading].get()));
    }
    CUDA_CHECK_FATAL(hipMemcpyAsync(
        deviceBuffer[loading],
        hostBuffer,
        FLAGS_buffer_size,
        hipMemcpyHostToDevice,
        streams[loading].get()));
    CUDA_CHECK_FATAL(
        hipEventRecord(bufferReady[loading].get(), streams[loading].get()));
    if (i > 0) {
      CUDA_CHECK_FATAL(hipEventSynchronize(bufferReady[processing].get()));
      sortBlock<<<
          elementCount / kItemsPerBlock,
          kBlockSize,
          sizeof(BlockStorage),
          streams[processing].get()>>>(deviceBuffer[processing]);
      CUDA_CHECK_FATAL(hipGetLastError());
      strideMemory<<<elementCount / kBlockSize, kBlockSize>>>(
          deviceBuffer[processing],
          elementCount,
          FLAGS_num_strides,
          outputBuffer);
      CUDA_CHECK_FATAL(hipGetLastError());
    }
    CUDA_CHECK_FATAL(hipEventRecord(
        processDone[processing].get(), streams[processing].get()));
  }
  auto stopEvent = createCudaEvent();
  CUDA_CHECK_FATAL(hipEventRecord(stopEvent.get()));
  CUDA_CHECK_FATAL(hipEventSynchronize(stopEvent.get()));
  float time;
  CUDA_CHECK_FATAL(
      hipEventElapsedTime(&time, startEvent.get(), stopEvent.get()));
  printf(
      "Device %d throughput: %.2f GB/s\n",
      deviceId,
      FLAGS_buffer_size * FLAGS_repeat * 1e-6 / time);
  if (FLAGS_validate) {
    CUDA_CHECK_FATAL(hipMemcpy(
        hostBuffer,
        deviceBuffer[processing],
        FLAGS_buffer_size,
        hipMemcpyDeviceToHost));
    for (int64_t i = 0; i < elementCount; i += kItemsPerBlock) {
      for (int64_t j = elementCount - i - kItemsPerBlock, di = 0;
           j < elementCount - i;
           ++j, ++di) {
        if (hostBuffer[i + di] != j) {
          fprintf(
              stderr,
              "hostBuffer[%ld]: %ld != %ld\n",
              i + di,
              hostBuffer[i + di],
              j);
          abort();
        }
      }
    }
  }
  CUDA_CHECK_LOG(hipFree(outputBuffer));
  CUDA_CHECK_LOG(hipFree(deviceBuffer[0]));
  CUDA_CHECK_LOG(hipFree(deviceBuffer[1]));
  CUDA_CHECK_LOG(hipHostFree(hostBuffer));
}

} // namespace
} // namespace facebook::velox::gpu

int main(int argc, char** argv) {
  using namespace facebook::velox::gpu;
  folly::init(&argc, &argv);
  int deviceCount;
  CUDA_CHECK_FATAL(hipGetDeviceCount(&deviceCount));
  printf("Device count: %d\n", deviceCount);
  std::vector<int> devices;
  for (int i = 0, deviceId = 0; i < FLAGS_devices.size(); ++i) {
    char c = FLAGS_devices[i];
    if (c != ',') {
      deviceId = 10 * deviceId + (c - '0');
    }
    if (c == ',' || i + 1 == FLAGS_devices.size()) {
      devices.push_back(deviceId);
      deviceId = 0;
    }
  }
  std::vector<std::thread> threads;
  for (int deviceId : devices) {
    threads.emplace_back(testCudaEvent, deviceId);
  }
  for (auto& t : threads) {
    t.join();
  }
  return 0;
}
