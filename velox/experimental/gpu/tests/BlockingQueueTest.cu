#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <folly/init/Init.h>
#include <gflags/gflags.h>
#include <algorithm>
#include <mutex>
#include "velox/experimental/gpu/BlockingQueue.h"
#include "velox/experimental/gpu/Common.h"

DEFINE_int32(gpu_grid_size, 1024, "");
DEFINE_int32(host_threads, 10, "");
DEFINE_int32(queue_size, 2048, "");
DEFINE_int32(running_time_seconds, 5, "");

DEFINE_int32(
    sync_every,
    50,
    "Host threads will wait for this number of messages"
    " from GPU before send this number of messages again");

namespace facebook::velox::gpu {
namespace {

void checkDeviceProperties() {
  int device;
  CUDA_CHECK_FATAL(hipGetDevice(&device));
  hipDeviceProp_t prop;
  CUDA_CHECK_FATAL(hipGetDeviceProperties(&prop, device));
  assert(prop.concurrentManagedAccess);
}

template <typename T, cuda::thread_scope kScope>
T* xCudaMalloc(int count) {
  T* ptr;
  if constexpr (kScope == cuda::thread_scope_system) {
    CUDA_CHECK_FATAL(hipMallocManaged(&ptr, count * sizeof(T)));
  } else {
    CUDA_CHECK_FATAL(hipMalloc(&ptr, count * sizeof(T)));
  }
  return ptr;
}

template <typename T, cuda::thread_scope kScope>
struct Queue {
  CudaPtr<T[]> data;
  CudaPtr<BlockingQueue<T, kScope>> queue;
};

template <typename T, cuda::thread_scope kScope>
__global__ void
initBlockingQueue(BlockingQueue<T, kScope>* queue, T* data, size_t capacity) {
  new (queue) BlockingQueue<T, kScope>(data, capacity);
}

template <typename T, cuda::thread_scope kScope>
Queue<T, kScope> createQueue(int capacity) {
  Queue<T, kScope> ans;
  ans.data.reset(xCudaMalloc<T, kScope>(capacity));
  ans.queue.reset(xCudaMalloc<BlockingQueue<T, kScope>, kScope>(1));
  if constexpr (kScope == cuda::thread_scope_system) {
    new (ans.queue.get()) BlockingQueue<T, kScope>(ans.data.get(), capacity);
  } else {
    initBlockingQueue<<<1, 1>>>(ans.queue.get(), ans.data.get(), capacity);
    CUDA_CHECK_FATAL(hipDeviceSynchronize());
  }
  return ans;
}

std::atomic_int64_t nextId;
std::chrono::steady_clock::time_point startTime;
std::vector<int64_t> received;
std::mutex receivedMutex;

void runCpu(BlockingQueue<int64_t>* rx, BlockingQueue<int64_t>* tx) {
  int messageCount = 0;
  std::chrono::seconds duration(FLAGS_running_time_seconds);
#ifndef NDEBUG
  std::vector<int64_t> localReceived;
#endif
  for (;;) {
    auto message = rx->dequeue();
#ifndef NDEBUG
    localReceived.push_back(message);
#else
    (void)message;
#endif
    if (++messageCount < FLAGS_sync_every) {
      continue;
    }
    if (std::chrono::steady_clock::now() >= startTime + duration) {
#ifndef NDEBUG
      {
        std::lock_guard<std::mutex> lock(receivedMutex);
        (void)lock;
        std::copy(
            localReceived.begin(),
            localReceived.end(),
            std::back_inserter(received));
      }
#else
      (void)receivedMutex;
#endif
      return;
    }
    messageCount = 0;
    for (int i = 0; i < FLAGS_sync_every; ++i) {
      tx->enqueue(nextId++);
    }
  }
}

__global__ void echo(BlockingQueue<int64_t>& rx, BlockingQueue<int64_t>& tx) {
  __shared__ int64_t message;
  for (;;) {
    if (threadIdx.x == 0) {
      message = rx.dequeue();
    }
    __syncthreads();
    if (message == -1) {
      return;
    }
    if (threadIdx.x == 0) {
      tx.enqueue(message);
    }
  }
}

void runCpuGpuPingPong() {
  auto cpu2gpu =
      createQueue<int64_t, cuda::thread_scope_system>(FLAGS_queue_size);
  auto gpu2cpu =
      createQueue<int64_t, cuda::thread_scope_system>(FLAGS_queue_size);
  for (int i = 0; i < FLAGS_queue_size; ++i) {
    cpu2gpu.queue->enqueue(nextId++);
  }
  std::vector<std::thread> threads;
  for (int i = 0; i < FLAGS_host_threads; ++i) {
    threads.emplace_back(runCpu, gpu2cpu.queue.get(), cpu2gpu.queue.get());
  }
  startTime = std::chrono::steady_clock::now();
  echo<<<FLAGS_gpu_grid_size, 1>>>(*cpu2gpu.queue, *gpu2cpu.queue);
  for (auto& t : threads) {
    t.join();
  }
  for (int i = 0; i < FLAGS_gpu_grid_size; ++i) {
    cpu2gpu.queue->enqueue(-1);
  }
  CUDA_CHECK_FATAL(hipDeviceSynchronize());
  auto endTime = std::chrono::steady_clock::now();
  assert(cpu2gpu.queue->size() == 0);
  size_t remaining = 0;
  int64_t message;
  while (gpu2cpu.queue->tryDequeue(message)) {
#ifndef NDEBUG
    received.push_back(message);
#endif
    ++remaining;
  }
  printf("CPU-GPU Ping Pong\n");
  printf("=================\n");
  printf("Total send: %lu\n", nextId.load());
  printf("Total received: %lu\n", nextId - remaining);
  printf(
      "%.2f ns per element\n",
      1.0 * (endTime - startTime).count() / (nextId - remaining));
  assert(gpu2cpu.queue->size() == 0);
  assert(received.size() == nextId);
#ifndef NDEBUG
  std::sort(received.begin(), received.end());
  for (int64_t i = 0; i < nextId; ++i) {
    if (received[i] != i) {
      printf("received[%ld] = %ld\n", i, received[i]);
      abort();
    }
  }
#else
  (void)received;
#endif
}

__global__ void runGpu2Gpu(
    BlockingQueue<int64_t, cuda::thread_scope_device>& q1,
    BlockingQueue<int64_t, cuda::thread_scope_device>& q2,
    cuda::atomic<int64_t, cuda::thread_scope_device>& nextId,
    int capacity,
    int limit) {
  __shared__ struct { int64_t message; } shared;
  if (blockIdx.x == 0) {
    nextId = 0;
    for (int i = 0; i < capacity; ++i) {
      q2.enqueue(nextId++);
    }
  }
  if (blockIdx.x % 2 == 0) {
    // Read from q1.
    for (;;) {
      if (threadIdx.x == 0) {
        shared.message = q1.dequeue();
      }
      __syncthreads();
      if (nextId > limit) {
        q2.enqueue(-1);
        return;
      }
      q2.enqueue(nextId++);
    }
  } else {
    // Read from q2.
    for (;;) {
      if (threadIdx.x == 0) {
        shared.message = q2.dequeue();
      }
      __syncthreads();
      if (shared.message == -1) {
        return;
      }
      if (threadIdx.x == 0) {
        q1.enqueue(shared.message);
      }
    }
  }
}

void runGpuGpuPingPong() {
  constexpr int kLimit = 40'000;
  auto gpu1 = createQueue<int64_t, cuda::thread_scope_device>(FLAGS_queue_size);
  auto gpu2 = createQueue<int64_t, cuda::thread_scope_device>(FLAGS_queue_size);
  CudaPtr<cuda::atomic<int64_t, cuda::thread_scope_device>> nextId(
      xCudaMalloc<
          cuda::atomic<int64_t, cuda::thread_scope_device>,
          cuda::thread_scope_device>(1));
  startTime = std::chrono::steady_clock::now();
  runGpu2Gpu<<<FLAGS_gpu_grid_size, 1>>>(
      *gpu1.queue, *gpu2.queue, *nextId, FLAGS_queue_size, kLimit);
  CUDA_CHECK_FATAL(hipDeviceSynchronize());
  auto endTime = std::chrono::steady_clock::now();
  printf("GPU-GPU Ping Pong\n");
  printf("=================\n");
  printf("%.2f ns per element\n", 1.0 * (endTime - startTime).count() / kLimit);
}

__global__ void runSimpleKernel(int64_t m, int64_t& out) {
  __shared__ int64_t message;
  if (threadIdx.x == 0) {
    message = m;
  }
  __syncthreads();
  out ^= message ^ threadIdx.x;
}

void runKernelLaunches() {
  startTime = std::chrono::steady_clock::now();
  CudaPtr<int64_t> out(xCudaMalloc<int64_t, cuda::thread_scope_system>(1));
  for (int i = 0; i < nextId; ++i) {
    runSimpleKernel<<<1, 1>>>(i, *out);
  }
  CUDA_CHECK_FATAL(hipDeviceSynchronize());
  printf("Simple Kernel Launch\n");
  printf("====================\n");
  printf(
      "%.2f ns per kernel launch\n",
      1.0 * (std::chrono::steady_clock::now() - startTime).count() / nextId);
}

} // namespace
} // namespace facebook::velox::gpu

int main(int argc, char** argv) {
  using namespace facebook::velox::gpu;
  folly::init(&argc, &argv);
  checkDeviceProperties();
  runCpuGpuPingPong();
  printf("\n");
#ifdef NDEBUG
  // For unknown reason BlockingQueue constructor is not called in dev mode.
  runGpuGpuPingPong();
  printf("\n");
#endif
  runKernelLaunches();
  return 0;
}
