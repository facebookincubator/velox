/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <chrono>
#include <thread>
#include <vector>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline hipError_t checkCuda(hipError_t result) {
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

__global__ void kernel(float* a, int offset) {
  int i = offset + threadIdx.x + blockIdx.x * blockDim.x;
  a[i] += i;
}

int64_t millis(std::chrono::steady_clock::time_point start) {
  return std::chrono::duration_cast<std::chrono::milliseconds>(
             std::chrono::steady_clock::now() - start)
      .count();
}

// Arguments: [-1] n-repeats n-streams kbytesperstream dev1 dev2 ...
int main(int argc, char** argv) {
  int nRepeats = 1;
  int nStreams = 4;
  int n256floats = 4096;
  if (argc < 4) {
    printf(
        "Usage CudaMemMeter [-1] numRepeats  numStreams KB [dev-1 dev-2 ...]\n"
        " Copies KB bytes to device and back numRepeats times. If doing "
        "this with multiple streams, uses numstreams streams.\n"
        "Runs the above for each of the devices listed after KB:\n"
        "CudaMemMeter 100 10e 200 0 0 1 1 will do 100 repeats of a 200K transfer to and "
        "from \n"
        "device. Each of these transfers will be divided into 10 parallel\n"
        "streams. This will be run twice on two host threads going to \n"
        "device 0 and two host threads going to device 1.");
    return 1;
  }
  int32_t firstArg = argv[1][0] == '-' && argv[1][1] == '1' ? 1 : 0;
  bool oneWay = firstArg == 1;
  float multiplier = oneWay ? 1.0 : 2.0;
  if (argc >= firstArg + 4) {
    nRepeats = atoi(argv[firstArg + 1]);
    nStreams = atoi(argv[firstArg + 2]);
    n256floats = atoi(argv[firstArg + 3]);
  }
  const int blockSize = 256;
  const long n = n256floats * blockSize * nStreams;
  const long streamSize = n / nStreams;
  const long streamBytes = streamSize * sizeof(float);
  const long bytes = n * sizeof(float);

  int devId = 0;
  if (argc > firstArg + 4) {
    devId = atoi(argv[firstArg + 4]);
  }
  std::vector<int32_t> devices;
  devices.push_back(devId);
  printf("Devices: %d", devId);
  for (auto i = firstArg + 5; i < argc; ++i) {
    devices.push_back(atoi(argv[i]));
    printf(", %d", devices.back());
  }
  printf("\n");
  hipDeviceProp_t prop;
  checkCuda(hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  std::vector<std::thread> threads;
  auto start = std::chrono::steady_clock::now();
  for (auto dev : devices) {
    threads.push_back(std::thread([dev,
                                   n,
                                   start,
                                   streamSize,
                                   blockSize,
                                   bytes,
                                   streamBytes,
                                   nRepeats,
                                   nStreams,
                                   oneWay,
                                   multiplier]() {
      checkCuda(hipSetDevice(dev));
      // allocate pinned host memory and device memory
      float *a, *d_a;
      printf("%ldKB\n", bytes / 1024);
      checkCuda(hipHostMalloc((void**)&a, bytes, hipHostMallocDefault)); // host pinned
      checkCuda(hipMalloc((void**)&d_a, bytes)); // device

      float ms; // elapsed time in milliseconds
      float singleMs = 0;
      float async1Ms = 0;
      float async2Ms = 0;

      // create events and streams
      hipEvent_t startEvent, stopEvent, dummyEvent;
      hipStream_t* stream =
          (hipStream_t*)malloc(nStreams * sizeof(hipStream_t));
      checkCuda(hipEventCreate(&startEvent));
      checkCuda(hipEventCreate(&stopEvent));
      checkCuda(hipEventCreate(&dummyEvent));
      for (int i = 0; i < nStreams; ++i) {
        checkCuda(hipStreamCreate(&stream[i]));
      }
      // baseline case - sequential transfer and execute
      memset(a, 0, bytes);
      for (int repeat = 0; repeat < nRepeats; ++repeat) {
        checkCuda(hipEventRecord(startEvent, 0));
        checkCuda(hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice));
        kernel<<<n / blockSize, blockSize>>>(d_a, 0);
        checkCuda(hipGetLastError());
        if (!oneWay) {
          checkCuda(hipMemcpy(a, d_a, bytes, hipMemcpyDeviceToHost));
        }
        checkCuda(hipEventRecord(stopEvent, 0));
        checkCuda(hipEventSynchronize(stopEvent));
        checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
        singleMs += ms;
      }
      float volume = bytes * nRepeats;
      printf(
          "%d at %ld: Time for sequential transfer and execute (ms): %f %f GB/s\n",
          dev,
          millis(start),
          singleMs,
          multiplier * (volume / (1 << 30)) / (singleMs / 1000));

      // asynchronous version 1: loop over {copy, kernel, copy}
      memset(a, 0, bytes);
      for (int repeat = 0; repeat < nRepeats; ++repeat) {
        checkCuda(hipEventRecord(startEvent, 0));
        for (int i = 0; i < nStreams; ++i) {
          int offset = i * streamSize;
          checkCuda(hipMemcpyAsync(
              &d_a[offset],
              &a[offset],
              streamBytes,
              hipMemcpyHostToDevice,
              stream[i]));
          kernel<<<streamSize / blockSize, blockSize, 0, stream[i]>>>(
              d_a, offset);
          checkCuda(hipGetLastError());
          if (!oneWay) {
            checkCuda(hipMemcpyAsync(
                &a[offset],
                &d_a[offset],
                streamBytes,
                hipMemcpyDeviceToHost,
                stream[i]));
          }
        }
        checkCuda(hipEventRecord(stopEvent, 0));
        checkCuda(hipEventSynchronize(stopEvent));
        checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
        async1Ms += ms;
      }
      volume = streamBytes * nRepeats * nStreams;
      printf(
          "%d at %ld: Time for asynchronous V1 transfer and execute (ms): %f %f GB/s\n",
          dev,
          millis(start),
          async1Ms,
          multiplier * (volume / (1 << 30)) / (async1Ms / 1000));

      // asynchronous version 2:
      // loop over copy, loop over kernel, loop over copy
      memset(a, 0, bytes);
      for (int repeat = 0; repeat < nRepeats; ++repeat) {
        checkCuda(hipEventRecord(startEvent, 0));
        for (int i = 0; i < nStreams; ++i) {
          int offset = i * streamSize;
          checkCuda(hipMemcpyAsync(
              &d_a[offset],
              &a[offset],
              streamBytes,
              hipMemcpyHostToDevice,
              stream[i]));
        }
        for (int i = 0; i < nStreams; ++i) {
          int offset = i * streamSize;
          kernel<<<streamSize / blockSize, blockSize, 0, stream[i]>>>(
              d_a, offset);
          checkCuda(hipGetLastError());
        }
        for (int i = 0; i < nStreams; ++i) {
          int offset = i * streamSize;
          if (!oneWay) {
            checkCuda(hipMemcpyAsync(
                &a[offset],
                &d_a[offset],
                streamBytes,
                hipMemcpyDeviceToHost,
                stream[i]));
          }
        }
        checkCuda(hipEventRecord(stopEvent, 0));
        checkCuda(hipEventSynchronize(stopEvent));
        checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
        async2Ms += ms;
      }
      volume = streamBytes * nRepeats * nStreams;
      printf(
          "%d at %ld: Time for asynchronous V2 transfer and execute (ms): %f %f GB/s\n",
          dev,
          millis(start),
          async2Ms,
          multiplier * (volume / (1 << 30)) / (async2Ms / 1000));

      // cleanup
      checkCuda(hipEventDestroy(startEvent));
      checkCuda(hipEventDestroy(stopEvent));
      checkCuda(hipEventDestroy(dummyEvent));
      for (int i = 0; i < nStreams; ++i)
        checkCuda(hipStreamDestroy(stream[i]));
      checkCuda(hipFree(d_a));
      checkCuda(hipHostFree(a));
    }));
  }
  for (auto& thread : threads) {
    thread.join();
  }
  printf("At %ld: Completed %ld threads", millis(start), threads.size());
  return 0;
}
