#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * Copyright (c) 2024 by Rivos Inc.
 * Licensed under the Apache License, Version 2.0, see LICENSE for details.
 * SPDX-License-Identifier: Apache-2.0
 */

#include <gtest/gtest.h>

#include <fstream>
#include <iostream>
#include <memory>
#include <string>
#include <vector>

#include "breeze/algorithms/sort.h"
#include "breeze/functions/load.h"
#include "breeze/functions/reduce.h"
#include "breeze/functions/scan.h"
#include "breeze/functions/store.h"
#include "breeze/platforms/cuda.cuh"
#include "breeze/utils/device_vector.h"
#include "breeze/utils/types.h"
#include "perftest/perftest.h"
#include "perftest/queries/caching_device_allocator.cuh"
#include "perftest/queries/device_column.h"

namespace breeze {
namespace test {
namespace kernels {

enum { CUDA_WARP_THREADS = 32 };

template <int BLOCK_THREADS, int ITEMS_PER_THREAD, int TILE_SIZE,
          int RADIX_BITS, typename T, typename U>
__global__ __launch_bounds__(BLOCK_THREADS) void BuildRadixSortHistogram(
    T* in_buffers[2], const int* in_selector, U* histogram, int num_items) {
  using namespace algorithms;
  using namespace utils;

  CudaPlatform<BLOCK_THREADS, CUDA_WARP_THREADS> p;
  using RadixSortHistogramT = DeviceRadixSortHistogram<RADIX_BITS, T>;
  __shared__ typename RadixSortHistogramT::Scratch scratch;

  const T* in = in_buffers[*in_selector];
  RadixSortHistogramT::template Build<ITEMS_PER_THREAD, TILE_SIZE>(
      p, make_slice<GLOBAL>(in), make_slice<GLOBAL>(histogram),
      make_slice(&scratch).template reinterpret<SHARED>(), num_items);
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD, int RADIX_BITS, typename U>
__global__
__launch_bounds__(BLOCK_THREADS) void RadixSortHistogramExclusiveScan(
    const U* in, U* out, int* buffer_advancement) {
  using namespace functions;
  using namespace utils;

  enum {
    NUM_BINS = 1 << RADIX_BITS,
  };

  CudaPlatform<BLOCK_THREADS, CUDA_WARP_THREADS> p;
  using BlockReduceT = BlockReduce<decltype(p), U>;
  using BlockScanT = BlockScan<decltype(p), U, ITEMS_PER_THREAD>;
  __shared__ struct {
    typename BlockReduceT::Scratch reduce_sum;
    typename BlockReduceT::Scratch reduce_max;
    typename BlockScanT::Scratch scan;
  } scratch;

  U items[ITEMS_PER_THREAD];

  // load counts
  const U* it = in + p.block_idx() * NUM_BINS;
  BlockLoad<BLOCK_THREADS, ITEMS_PER_THREAD>(p, make_slice<GLOBAL>(it),
                                             make_slice(items), NUM_BINS);

  // reductions to determine if all items are in the same bin
  U sum = BlockReduceT::template Reduce<ReduceOpAdd, ITEMS_PER_THREAD>(
      p, make_slice(items), make_slice<SHARED>(&scratch.reduce_sum), NUM_BINS);
  U max = BlockReduceT::template Reduce<ReduceOpMax, ITEMS_PER_THREAD>(
      p, make_slice(items), make_slice<SHARED>(&scratch.reduce_max), NUM_BINS);

  // advance buffer unless all items are in the same bin (sum == max) and
  // pass can be skipped
  if (p.thread_idx() == 0) {
    buffer_advancement[p.block_idx()] = sum == max ? 0 : 1;
  }

  // inclusive scan
  U offsets[ITEMS_PER_THREAD];
  BlockScanT::template Scan<ScanOpAdd>(
      p, make_slice(items), make_slice(offsets),
      make_slice<SHARED>(&scratch.scan), NUM_BINS);

  // convert inclusive scan to exclusive scan
#pragma unroll
  for (int i = 0; i < ITEMS_PER_THREAD; ++i) {
    offsets[i] -= items[i];
  }

  // store results
  U* out_it = out + p.block_idx() * NUM_BINS;
  BlockStore<BLOCK_THREADS, ITEMS_PER_THREAD>(
      p, make_slice(offsets), make_slice<GLOBAL>(out_it), NUM_BINS);
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ __launch_bounds__(BLOCK_THREADS) void UpdateBufferSelectors(
    const int* buffer_advancements, int* input_selector,
    int* buffer_selectors) {
  // get initial index from input selector
  int current_selector = *input_selector;

  // we have 2 selectors per pass; first selector determines the input and
  // second determines the output
  for (int i = 0; i < ITEMS_PER_THREAD; ++i) {
    buffer_selectors[i * 2] = current_selector;
    current_selector = (current_selector + buffer_advancements[i]) % 2;
    buffer_selectors[i * 2 + 1] = current_selector;
  }

  // update input selector to the final output selector
  *input_selector = current_selector;
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD, int RADIX_BITS, typename T,
          typename U, typename BlockT>
__global__ __launch_bounds__(BLOCK_THREADS) void RadixSort(
    const int* in_buffer_selectors, const U* in_offsets, int start_bit,
    int num_pass_bits, T* buffers[2], int* next_block_idx, BlockT* blocks,
    int num_items) {
  using namespace algorithms;
  using namespace utils;

  CudaPlatform<BLOCK_THREADS, CUDA_WARP_THREADS> p;
  using RadixSortT =
      DeviceRadixSort<decltype(p), ITEMS_PER_THREAD, RADIX_BITS, T>;
  __shared__ typename RadixSortT::Scratch scratch;

  // load buffer selectors
  int current_selector = in_buffer_selectors[0];
  int alternate_selector = in_buffer_selectors[1];

  // sorting pass is only needed if input and output selectors are different
  if (current_selector != alternate_selector) {
    const T* in = buffers[current_selector];
    T* out = buffers[alternate_selector];

    RadixSortT::template Sort<BlockT>(
        p, make_slice<GLOBAL>(in), make_slice<GLOBAL>(in_offsets), start_bit,
        num_pass_bits, make_slice<GLOBAL>(out),
        make_slice<GLOBAL>(next_block_idx), make_slice<GLOBAL>(blocks),
        make_slice(&scratch).template reinterpret<SHARED>(), num_items);
  }
}

}  // namespace kernels

using OrderByConfig = PerfTestArrayConfig<11>;

const OrderByConfig kConfig = {{
    {"num_key_rows", "400000"},
    {"num_key_rows_short", "6400"},
    {"num_key_rows_grande", "6400000"},
    {"num_key_rows_venti", "64000000"},
    {"key_generate_method", "RANDOM"},
    {"key_random_engine", "MT19937"},
    {"key_random_shuffle", "1"},
    {"key_random_stride", "1000"},
    {"key_random_stride_short", "10"},
    {"key_random_stride_grande", "100000"},
    {"key_random_stride_venti", "100000"},
}};

template <typename TypeParam>
class OrderByPerfTest : public PerfTest<OrderByConfig>, public testing::Test {
 public:
  template <typename T>
  T GetConfigValue(const char* key, T default_value) const {
    return kConfig.get<T>(key, default_value);
  }
  template <typename T>
  std::vector<T> GetConfigColumn(const char* prefix) const {
    return kConfig.get_column<T>(prefix);
  }
};

template <int N>
struct RadixBits {
  enum {
    VALUE = N,
  };
};

template <typename LaunchParamsAndItemTypeT, typename RadixBitsT>
struct OrderByTestType {
  using launch_params_and_item_type_type = LaunchParamsAndItemTypeT;
  using item_type = typename LaunchParamsAndItemTypeT::item_type;
  using key_type = typename item_type::type;
  using launch_params = typename LaunchParamsAndItemTypeT::launch_params;

  enum {
    BLOCK_THREADS = launch_params::BLOCK_THREADS,
    ITEMS_PER_THREAD = launch_params::ITEMS_PER_THREAD,
    BLOCK_ITEMS = BLOCK_THREADS * ITEMS_PER_THREAD,
    RADIX_BITS = RadixBitsT::VALUE,
    NUM_BINS = 1 << RADIX_BITS,
    END_BIT = sizeof(key_type) * /*BITS_PER_BYTE=*/8,
    NUM_PASSES = utils::DivideAndRoundUp<END_BIT, RADIX_BITS>::VALUE,
    HISTOGRAM_SIZE = NUM_BINS * NUM_PASSES,
    BINS_PER_THREAD = utils::DivideAndRoundUp<NUM_BINS, BLOCK_THREADS>::VALUE,
    HISTOGRAM_ITEMS_PER_THREAD = 8,
    HISTOGRAM_TILE_SIZE = 16,
    HISTOGRAM_BLOCK_ITEMS = BLOCK_THREADS * HISTOGRAM_ITEMS_PER_THREAD,
    HISTOGRAM_TILE_ITEMS = HISTOGRAM_TILE_SIZE * HISTOGRAM_BLOCK_ITEMS,
  };

  static std::string GetName() {
    return launch_params_and_item_type_type::GetName() + ".RadixBits" +
           std::to_string(RADIX_BITS);
  }

  static size_t GlobalMemoryLoads(size_t num_keys) {
    int num_histogram_blocks =
        (num_keys + HISTOGRAM_TILE_ITEMS - 1) / HISTOGRAM_TILE_ITEMS;
    // count each atomic add as 1 load + 1 store
    int num_atomic_loads = HISTOGRAM_SIZE * num_histogram_blocks;
    // 1N global memory loads for histogram + 1N for each sorting pass
    return (num_keys * (1ll + NUM_PASSES)) * sizeof(key_type) +
           num_atomic_loads * sizeof(unsigned);
  }

  static size_t GlobalMemoryStores(size_t num_keys) {
    int num_histogram_blocks =
        (num_keys + HISTOGRAM_TILE_ITEMS - 1) / HISTOGRAM_TILE_ITEMS;
    // count the store of each atomic add
    int num_atomic_stores = HISTOGRAM_SIZE * num_histogram_blocks;
    // 1N global memory stores for each sorting pass
    return (num_keys * NUM_PASSES) * sizeof(key_type) +
           num_atomic_stores * sizeof(unsigned);
  }

  template <typename Allocator>
  static void SortKeys(device_column_buffered<key_type>& keys,
                       const Allocator& allocator) {
    using namespace utils;

    // constant size temporary storage that needs to be zero initialized
    struct TempStorage {
      unsigned histogram[HISTOGRAM_SIZE];
      int next_block_idx[NUM_PASSES];
    };

    int num_blocks = (keys.size() + BLOCK_ITEMS - 1) / BLOCK_ITEMS;
    int num_histogram_blocks =
        (keys.size() + HISTOGRAM_TILE_ITEMS - 1) / HISTOGRAM_TILE_ITEMS;

    typedef
        typename Allocator::template rebind<TempStorage>::other temp_alloc_type;
    typedef typename Allocator::template rebind<int>::other int_alloc_type;
    typedef
        typename Allocator::template rebind<unsigned>::other uint_alloc_type;

    auto temp_allocator = temp_alloc_type(allocator);
    auto int_allocator = int_alloc_type(allocator);
    auto uint_allocator = uint_alloc_type(allocator);

    // temporary storage
    device_vector<TempStorage, temp_alloc_type> temp_storage(1, temp_allocator);
    device_vector<unsigned, uint_alloc_type> offsets(HISTOGRAM_SIZE,
                                                     uint_allocator);
    device_vector<unsigned, uint_alloc_type> blocks(
        num_blocks * NUM_BINS * NUM_PASSES, uint_allocator);
    device_vector<int, int_alloc_type> buffer_advancements(NUM_PASSES,
                                                           int_allocator);
    device_vector<int, int_alloc_type> buffer_selectors(2 * NUM_PASSES,
                                                        int_allocator);

    // initialize temporary storage
    hipMemsetAsync(temp_storage.data(), 0, sizeof(TempStorage));
    hipMemsetAsync(blocks.data(), 0, sizeof(unsigned) * blocks.size());

    kernels::BuildRadixSortHistogram<BLOCK_THREADS, HISTOGRAM_ITEMS_PER_THREAD,
                                     HISTOGRAM_TILE_SIZE, RADIX_BITS>
        <<<num_histogram_blocks, BLOCK_THREADS>>>(
            keys.ptrs().data(), keys.selector().data(),
            temp_storage.data()->histogram, keys.size());

    // exclusive scan of histogram and set buffer advancements
    kernels::RadixSortHistogramExclusiveScan<BLOCK_THREADS, BINS_PER_THREAD,
                                             RADIX_BITS>
        <<<NUM_PASSES, BLOCK_THREADS>>>(temp_storage.data()->histogram,
                                        offsets.data(),
                                        buffer_advancements.data());

    // update buffer selectors using buffer advancements
    kernels::UpdateBufferSelectors<
        /*BLOCK_THREADS=*/1,
        /*ITEMS_PER_THREAD=*/NUM_PASSES>
        <<</*num_blocks=*/1, /*BLOCK_THREADS=*/1>>>(buffer_advancements.data(),
                                                    keys.selector().data(),
                                                    buffer_selectors.data());

    // start from lsb and loop until no bits are left
    int start_bit = 0;
    for (int pass = 0; pass < NUM_PASSES; ++pass) {
      int num_pass_bits =
          std::min(static_cast<int>(RADIX_BITS), END_BIT - start_bit);

      // input for this sorting pass
      int* pass_next_block_idx = &temp_storage.data()->next_block_idx[pass];
      unsigned* pass_blocks = blocks.data() + pass * num_blocks * NUM_BINS;
      const int* pass_buffer_selectors = buffer_selectors.data() + pass * 2;
      const unsigned* pass_offsets = offsets.data() + pass * NUM_BINS;

      // radix sorting pass
      kernels::RadixSort<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS>
          <<<num_blocks, BLOCK_THREADS>>>(
              pass_buffer_selectors, pass_offsets, start_bit, num_pass_bits,
              keys.ptrs().data(), pass_next_block_idx, pass_blocks,
              keys.size());

      // advance start bit for next pass
      start_bit += RADIX_BITS;
    }
  }
};

using LaunchParamsTypes =
    std::tuple<LaunchParams<256, 8>, LaunchParams<256, 16>,
               LaunchParams<256, 24>>;

using LaunchParamsAndItemTypes =
    CombineLaunchParamsAndTypes<LaunchParamsAndItemType, LaunchParamsTypes, int,
                                unsigned, long long, unsigned long long>;

using TestTypes =
    MakeTestTypes<CombineTestTypes<OrderByTestType, LaunchParamsAndItemTypes,
                                   RadixBits<8>>>::types;

TYPED_TEST_SUITE(OrderByPerfTest, TestTypes, TestTypeNames);

TYPED_TEST(OrderByPerfTest, SelectKeysOrderByKeys) {
  using key_type = typename TypeParam::key_type;
  using indices_type = utils::size_type;

  auto items = this->template GetConfigColumn<key_type>("key");
  ASSERT_NE(items.size(), 0u);

  auto check_result = this->GetConfigValue("check_result", true);
  auto result_file = this->GetConfigValue("result_file", std::string());

  device_column_buffered<key_type> d_items(items.size());
  int input_selector = 0;
  d_items.buffer(input_selector).copy_from_host(items.data(), items.size());
  d_items.selector().copy_from_host(&input_selector, 1);

  auto free_list = std::make_shared<
      caching_device_allocator<utils::size_type>::free_list_type>();
  caching_device_allocator<utils::size_type> allocator(free_list);

  // provide throughput information
  this->set_element_count(items.size());
  this->set_element_size(sizeof(key_type));
  this->set_elements_per_thread(TypeParam::launch_params::ITEMS_PER_THREAD);
  this->set_global_memory_loads(TypeParam::GlobalMemoryLoads(items.size()));
  this->set_global_memory_stores(TypeParam::GlobalMemoryStores(items.size()));

  this->Measure(kConfig, [&]() { TypeParam::SortKeys(d_items, allocator); });

  if (check_result) {
    int output_selector;
    d_items.selector().copy_to_host(&output_selector, 1);
    std::vector<key_type> h_sorted_items(items.size());
    d_items.buffer(output_selector)
        .copy_to_host(h_sorted_items.data(), items.size());
    std::vector<key_type> expected_sorted_items = items;
    std::stable_sort(expected_sorted_items.begin(),
                     expected_sorted_items.end());
    EXPECT_EQ(h_sorted_items, expected_sorted_items);
  }

  if (!result_file.empty()) {
    int output_selector;
    d_items.selector().copy_to_host(&output_selector, 1);
    std::vector<key_type> h_sorted_items(items.size());
    d_items.buffer(output_selector)
        .copy_to_host(h_sorted_items.data(), items.size());

    std::ofstream result_out;
    result_out.open(result_file);
    ASSERT_TRUE(result_out.is_open())
        << "failed to open result file: " << result_file;
    result_out << "sorted_item" << std::endl;
    for (size_t i = 0; i < h_sorted_items.size(); ++i) {
      result_out << h_sorted_items[i] << std::endl;
    }
    result_out.close();
  }

  for (auto entry : *free_list) {
    hipFree(entry.second);
  }
}

}  // namespace test
}  // namespace breeze
