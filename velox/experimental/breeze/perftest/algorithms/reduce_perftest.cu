#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * Copyright (c) 2024 by Rivos Inc.
 * Licensed under the Apache License, Version 2.0, see LICENSE for details.
 * SPDX-License-Identifier: Apache-2.0
 */

#include <gtest/gtest.h>

#include <limits>
#include <vector>

#include "breeze/algorithms/reduce.h"
#include "breeze/platforms/platform.h"
#include "breeze/utils/device_vector.h"
#include "perftest/perftest.h"

namespace breeze {

using namespace algorithms;
using namespace utils;

namespace test {
namespace kernels {

enum { CUDA_WARP_THREADS = 32 };

template <typename Op, int BLOCK_THREADS, int ITEMS_PER_THREAD, typename T,
          typename U>
__global__ __launch_bounds__(BLOCK_THREADS) void Reduce(const T* in, U* out,
                                                        int num_items) {
  CudaPlatform<BLOCK_THREADS, CUDA_WARP_THREADS> p;
  using DeviceReduceT = DeviceReduce<decltype(p), U>;
  __shared__ typename DeviceReduceT::Scratch scratch;
  DeviceReduceT::template Reduce<Op, ITEMS_PER_THREAD>(
      p, make_slice<GLOBAL>(in), make_slice<GLOBAL>(out),
      make_slice(&scratch).template reinterpret<SHARED>(), num_items);
}

}  // namespace kernels

using ReduceConfig = PerfTestArrayConfig<12>;

const ReduceConfig kConfig = {{
    {"num_input_rows", "1048000"},
    {"num_input_rows_short", "200000"},
    {"num_input_rows_grande", "16768000"},
    {"num_input_rows_venti", "200000000"},
    {"input_generate_method", "FILL"},
    {"input_fill_value", "1"},
    {"expected_sum", "1048000"},
    {"expected_sum_short", "200000"},
    {"expected_sum_grande", "16768000"},
    {"expected_sum_venti", "200000000"},
    {"expected_min", "1"},
    {"expected_max", "1"},
}};

template <typename TypeParam>
class ReducePerfTest : public PerfTest<ReduceConfig>, public testing::Test {
 public:
  template <typename T>
  T GetConfigValue(const char* key, T default_value) const {
    return kConfig.get<T>(key, default_value);
  }
  template <typename T>
  T GetSizedConfigValue(const char* key, T default_value) const {
    return kConfig.get_sized<T>(key, default_value);
  }
  template <typename T>
  std::vector<T> GetConfigColumn(const char* prefix) const {
    return kConfig.get_column<T>(prefix);
  }
};

using LaunchParamsTypes =
    std::tuple<LaunchParams<256, 8>, LaunchParams<256, 16>>;

using TestTypes = MakePerfTestTypes<LaunchParamsTypes, int, long long>::types;

TYPED_TEST_SUITE(ReducePerfTest, TestTypes, TestTypeNames);

TYPED_TEST(ReducePerfTest, Add) {
  using value_type = typename TypeParam::item_type::type;

  auto input = this->template GetConfigColumn<value_type>("input");
  ASSERT_NE(input.size(), 0u);

  auto check_result = this->GetConfigValue("check_result", true);
  auto expected_sum = this->GetSizedConfigValue("expected_sum", 0);

  std::vector<value_type> input_identity(1);

  device_vector<value_type> items(input.size());
  device_vector<value_type> identity(1);
  device_vector<value_type> result(1);

  constexpr int kBlockThreads = TypeParam::launch_params::BLOCK_THREADS;
  constexpr int kItemsPerThread = TypeParam::launch_params::ITEMS_PER_THREAD;
  constexpr int kBlockItems = kBlockThreads * kItemsPerThread;

  int num_blocks = (input.size() + kBlockItems - 1) / kBlockItems;

  // copy input to device memory
  items.copy_from_host(input.data(), input.size());
  identity.copy_from_host(input_identity.data(), input_identity.size());

  // provide throughput information
  this->set_element_count(input.size());
  this->set_element_size(sizeof(value_type));
  this->set_elements_per_thread(kItemsPerThread);
  this->template set_global_memory_loads<value_type>(items.size());

  this->MeasureWithSetup(
      kConfig,
      [&]() {
        hipMemcpyAsync(result.data(), identity.data(), sizeof(value_type),
                        hipMemcpyDeviceToDevice);
      },
      [&]() {
        kernels::Reduce<ReduceOpAdd, kBlockThreads, kItemsPerThread>
            <<<num_blocks, kBlockThreads>>>(items.data(), result.data(),
                                            items.size());
      });

  if (check_result) {
    value_type sum = 0;
    result.copy_to_host(&sum, 1);
    EXPECT_EQ(sum, expected_sum);
  }
}

TYPED_TEST(ReducePerfTest, Min) {
  using value_type = typename TypeParam::item_type::type;

  auto input = this->template GetConfigColumn<value_type>("input");
  ASSERT_NE(input.size(), 0u);

  auto check_result = this->GetConfigValue("check_result", true);
  auto expected_min = this->GetConfigValue("expected_min", 0);

  std::vector<value_type> input_identity(
      1, std::numeric_limits<value_type>::max());

  device_vector<value_type> items(input.size());
  device_vector<value_type> identity(1);
  device_vector<value_type> result(1);

  constexpr int kBlockThreads = TypeParam::launch_params::BLOCK_THREADS;
  constexpr int kItemsPerThread = TypeParam::launch_params::ITEMS_PER_THREAD;
  constexpr int kBlockItems = kBlockThreads * kItemsPerThread;

  int num_blocks = (input.size() + kBlockItems - 1) / kBlockItems;

  // copy input to device memory
  items.copy_from_host(input.data(), input.size());
  identity.copy_from_host(input_identity.data(), input_identity.size());

  // provide throughput information
  this->set_element_count(input.size());
  this->set_element_size(sizeof(value_type));
  this->set_elements_per_thread(kItemsPerThread);
  this->template set_global_memory_loads<value_type>(items.size());

  this->MeasureWithSetup(
      kConfig,
      [&]() {
        hipMemcpyAsync(result.data(), identity.data(), sizeof(value_type),
                        hipMemcpyDeviceToDevice);
      },
      [&]() {
        kernels::Reduce<ReduceOpMin, kBlockThreads, kItemsPerThread>
            <<<num_blocks, kBlockThreads>>>(items.data(), result.data(),
                                            items.size());
      });

  if (check_result) {
    value_type min = 0;
    result.copy_to_host(&min, 1);
    EXPECT_EQ(min, expected_min);
  }
}

TYPED_TEST(ReducePerfTest, Max) {
  using value_type = typename TypeParam::item_type::type;

  auto input = this->template GetConfigColumn<value_type>("input");
  ASSERT_NE(input.size(), 0u);

  auto check_result = this->GetConfigValue("check_result", true);
  auto expected_max = this->GetConfigValue("expected_max", 0);

  std::vector<value_type> input_identity(
      1, std::numeric_limits<value_type>::min());

  device_vector<value_type> items(input.size());
  device_vector<value_type> identity(1);
  device_vector<value_type> result(1);

  constexpr int kBlockThreads = TypeParam::launch_params::BLOCK_THREADS;
  constexpr int kItemsPerThread = TypeParam::launch_params::ITEMS_PER_THREAD;
  constexpr int kBlockItems = kBlockThreads * kItemsPerThread;

  int num_blocks = (input.size() + kBlockItems - 1) / kBlockItems;

  // copy input to device memory
  items.copy_from_host(input.data(), input.size());
  identity.copy_from_host(input_identity.data(), input_identity.size());

  // provide throughput information
  this->set_element_count(input.size());
  this->set_element_size(sizeof(value_type));
  this->set_elements_per_thread(kItemsPerThread);
  this->template set_global_memory_loads<value_type>(items.size());

  this->MeasureWithSetup(
      kConfig,
      [&]() {
        hipMemcpyAsync(result.data(), identity.data(), sizeof(value_type),
                        hipMemcpyDeviceToDevice);
      },
      [&]() {
        kernels::Reduce<ReduceOpMax, kBlockThreads, kItemsPerThread>
            <<<num_blocks, kBlockThreads>>>(items.data(), result.data(),
                                            items.size());
      });

  if (check_result) {
    value_type max = 0;
    result.copy_to_host(&max, 1);
    EXPECT_EQ(max, expected_max);
  }
}

}  // namespace test
}  // namespace breeze
