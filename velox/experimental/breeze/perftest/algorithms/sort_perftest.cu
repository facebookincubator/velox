#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * Copyright (c) 2024 by Rivos Inc.
 * Licensed under the Apache License, Version 2.0, see LICENSE for details.
 * SPDX-License-Identifier: Apache-2.0
 */

#include <gtest/gtest.h>

#include <climits>
#include <vector>

#include "breeze/algorithms/sort.h"
#include "breeze/platforms/platform.h"
#include "breeze/utils/block_details.h"
#include "breeze/utils/device_vector.h"
#include "perftest/perftest.h"

namespace breeze {

using namespace algorithms;
using namespace functions;
using namespace utils;

namespace test {
namespace kernels {

enum { CUDA_WARP_THREADS = 32 };

template <int BLOCK_THREADS, int ITEMS_PER_THREAD, int RADIX_BITS, typename T,
          typename U, typename BlockT>
__global__ __launch_bounds__(BLOCK_THREADS) void RadixSort(
    const T* in, const U* in_offsets, int start_bit, int num_pass_bits, T* out,
    int* next_block_idx, BlockT* blocks, int num_items) {
  CudaPlatform<BLOCK_THREADS, CUDA_WARP_THREADS> p;
  using DeviceRadixSortT =
      DeviceRadixSort<decltype(p), ITEMS_PER_THREAD, RADIX_BITS, T>;
  __shared__ typename DeviceRadixSortT::Scratch scratch;
  DeviceRadixSortT::template Sort<BlockT>(
      p, make_slice<GLOBAL>(in), make_slice<GLOBAL>(in_offsets), start_bit,
      num_pass_bits, make_slice<GLOBAL>(out),
      make_slice<GLOBAL>(next_block_idx), make_slice<GLOBAL>(blocks),
      make_slice(&scratch).template reinterpret<SHARED>(), num_items);
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD, int TILE_SIZE,
          int RADIX_BITS, typename T, typename U>
__global__ __launch_bounds__(BLOCK_THREADS) void RadixSortHistogram(
    const T* in, U* histogram, int num_items) {
  CudaPlatform<BLOCK_THREADS, CUDA_WARP_THREADS> p;
  using DeviceRadixSortHistogramT = DeviceRadixSortHistogram<RADIX_BITS, T>;
  __shared__ typename DeviceRadixSortHistogramT::Scratch scratch;
  DeviceRadixSortHistogramT::template Build<ITEMS_PER_THREAD, TILE_SIZE>(
      p, make_slice<GLOBAL>(in), make_slice<GLOBAL>(histogram),
      make_slice(&scratch).template reinterpret<SHARED>(), num_items);
}

}  // namespace kernels

template <typename T>
typename std::enable_if<!std::is_unsigned<T>::value, T>::type to_bit_ordered(
    T value) {
  return value ^ (static_cast<T>(1) << (sizeof(T) * CHAR_BIT - 1));
}

template <typename T>
typename std::enable_if<std::is_unsigned<T>::value, T>::type to_bit_ordered(
    T value) {
  return value;
}

template <typename T>
T extract_bits(T value, int start_bit, int num_pass_bits) {
  return (value >> start_bit) & ((1u << num_pass_bits) - 1);
}

using SortConfig = PerfTestArrayConfig<11>;

const SortConfig kConfig = {{{"num_input_rows", "400000"},
                             {"num_input_rows_short", "6400"},
                             {"num_input_rows_grande", "6400000"},
                             {"num_input_rows_venti", "64000000"},
                             {"input_generate_method", "RANDOM"},
                             {"input_random_engine", "MT19937"},
                             {"input_random_shuffle", "1"},
                             {"input_random_stride", "1000"},
                             {"input_random_stride_short", "10"},
                             {"input_random_stride_grande", "100000"},
                             {"input_random_stride_venti", "100000"}}};

template <typename TypeParam>
class SortPerfTest : public PerfTest<SortConfig>, public testing::Test {
 public:
  template <typename T>
  T GetConfigValue(const char* key, T default_value) const {
    return kConfig.get<T>(key, default_value);
  }
  template <typename T>
  std::vector<T> GetConfigColumn(const char* prefix) const {
    return kConfig.get_column<T>(prefix);
  }
};

template <int N>
struct RadixBits {
  enum {
    VALUE = N,
  };
};

template <typename LaunchParamsAndItemTypeT, typename RadixBitsT>
struct SortTestType {
  using launch_params_and_item_type_type = LaunchParamsAndItemTypeT;
  using item_type = typename LaunchParamsAndItemTypeT::item_type;
  using launch_params = typename LaunchParamsAndItemTypeT::launch_params;
  enum {
    RADIX_BITS = RadixBitsT::VALUE,
  };

  static std::string GetName() {
    return launch_params_and_item_type_type::GetName() + ".RadixBits" +
           std::to_string(RADIX_BITS);
  }
};

using LaunchParamsTypes =
    std::tuple<LaunchParams<256, 8>, LaunchParams<256, 16>,
               LaunchParams<256, 24>>;

using LaunchParamsAndItemTypes =
    CombineLaunchParamsAndTypes<LaunchParamsAndItemType, LaunchParamsTypes, int,
                                unsigned, long long, unsigned long long>;

using TestTypes =
    MakeTestTypes<CombineTestTypes<SortTestType, LaunchParamsAndItemTypes,
                                   RadixBits<8>>>::types;

TYPED_TEST_SUITE(SortPerfTest, TestTypes, TestTypeNames);

TYPED_TEST(SortPerfTest, RadixSort) {
  using value_type = typename TypeParam::item_type::type;
  using size_type = unsigned;
  using block_type = unsigned;

  auto input = this->template GetConfigColumn<value_type>("input");
  ASSERT_NE(input.size(), 0u);

  auto check_result = this->GetConfigValue("check_result", true);

  device_vector<value_type> items(input.size());

  constexpr int kBlockThreads = TypeParam::launch_params::BLOCK_THREADS;
  constexpr int kItemsPerThread = TypeParam::launch_params::ITEMS_PER_THREAD;
  constexpr int kBlockItems = kBlockThreads * kItemsPerThread;
  constexpr int kRadixBits = TypeParam::RADIX_BITS;
  constexpr int kEndBit = sizeof(value_type) * /*BITS_PER_BYTE=*/8;
  constexpr int kNumBins = 1 << kRadixBits;

  auto start_bit = this->GetConfigValue("start_bit", 0);
  ASSERT_LT(start_bit, kEndBit);

  int num_pass_bits = std::min(kRadixBits, kEndBit - start_bit);
  std::vector<size_type> input_histogram(kNumBins);
  for (const auto& value : input) {
    int bin = extract_bits(to_bit_ordered(value), start_bit, num_pass_bits);
    input_histogram[bin] += 1u;
  }
  size_type sum = 0;
  std::vector<size_type> input_prefix_sum(kNumBins);
  for (size_t i = 0; i < kNumBins; ++i) {
    input_prefix_sum[i] = sum;
    sum += input_histogram[i];
  }

  int num_blocks = (input.size() + kBlockItems - 1) / kBlockItems;

  device_vector<int> next_block_idx(1);
  device_vector<block_type> blocks(num_blocks * kNumBins);
  device_vector<size_type> prefix_sum(kNumBins);
  device_vector<size_type> offsets(kNumBins);
  device_vector<value_type> out(input.size());

  // copy input to device memory
  items.copy_from_host(input.data(), input.size());
  prefix_sum.copy_from_host(input_prefix_sum.data(), input_prefix_sum.size());

  // provide throughput information
  this->set_element_count(input.size());
  this->set_element_size(sizeof(value_type));
  this->set_elements_per_thread(kItemsPerThread);
  this->template set_global_memory_loads<value_type>(input.size());
  this->template set_global_memory_stores<value_type>(input.size());

  this->MeasureWithSetup(
      kConfig,
      [&]() {
        hipMemsetAsync(next_block_idx.data(), 0, sizeof(int));
        hipMemsetAsync(blocks.data(), 0,
                        sizeof(block_type) * num_blocks * kNumBins);
        hipMemcpyAsync(offsets.data(), prefix_sum.data(),
                        sizeof(size_type) * kNumBins, hipMemcpyDeviceToDevice);
      },
      [&]() {
        kernels::RadixSort<kBlockThreads, kItemsPerThread, kRadixBits>
            <<<num_blocks, kBlockThreads>>>(
                items.data(), offsets.data(), start_bit, num_pass_bits,
                out.data(), next_block_idx.data(), blocks.data(), items.size());
      });

  if (check_result) {
    std::vector<value_type> actual_result(out.size());
    out.copy_to_host(actual_result.data(), actual_result.size());
    std::vector<value_type> expected_result = input;
    std::stable_sort(expected_result.begin(), expected_result.end(),
                     [start_bit, num_pass_bits](value_type a, value_type b) {
                       return extract_bits(a, start_bit, num_pass_bits) <
                              extract_bits(b, start_bit, num_pass_bits);
                     });
    EXPECT_EQ(expected_result, actual_result);
  }
}

const SortConfig kHistogramConfig = {{{"num_input_rows", "16750000"},
                                      {"num_input_rows_short", "2048000"},
                                      {"num_input_rows_grande", "268000000"},
                                      {"num_input_rows_venti", "2144000000"},
                                      {"input_generate_method", "RANDOM"},
                                      {"input_random_engine", "MT19937"},
                                      {"input_random_shuffle", "1"},
                                      {"input_random_stride", "1000"},
                                      {"input_random_stride_short", "10"},
                                      {"input_random_stride_grande", "100000"},
                                      {"input_random_stride_venti", "100000"}}};

template <typename TypeParam>
class SortHistogramPerfTest : public PerfTest<SortConfig>,
                              public testing::Test {
 public:
  template <typename T>
  T GetConfigValue(const char* key, T default_value) const {
    return kHistogramConfig.get<T>(key, default_value);
  }
  template <typename T>
  std::vector<T> GetConfigColumn(const char* prefix) const {
    return kHistogramConfig.get_column<T>(prefix);
  }
};

template <int N>
struct TileSize {
  enum {
    VALUE = N,
  };
};

template <typename SortTestTypeT, typename TileSizeT>
struct SortHistogramTestType {
  using launch_params_and_item_type_type =
      typename SortTestTypeT::launch_params_and_item_type_type;
  using item_type = typename launch_params_and_item_type_type::item_type;
  using launch_params =
      typename launch_params_and_item_type_type::launch_params;
  enum {
    RADIX_BITS = SortTestTypeT::RADIX_BITS,
    TILE_SIZE = TileSizeT::VALUE,
  };

  static std::string GetName() {
    return SortTestTypeT::GetName() + ".TileSize" + std::to_string(TILE_SIZE);
  }
};

using HistogramLaunchParamsTypes =
    std::tuple<LaunchParams<256, 4>, LaunchParams<256, 8>>;

using HistogramLaunchParamsAndItemTypes =
    CombineLaunchParamsAndTypes<LaunchParamsAndItemType,
                                HistogramLaunchParamsTypes, int, unsigned,
                                long long, unsigned long long>;

using HistogramSortTestTypes =
    CombineTestTypes<SortTestType, HistogramLaunchParamsAndItemTypes,
                     RadixBits<8>>;

using SortHistogramTestTypes = MakeTestTypes<
    CombineTestTypes<SortHistogramTestType, HistogramSortTestTypes,
                     TileSize<16>, TileSize<32>>>::types;

TYPED_TEST_SUITE(SortHistogramPerfTest, SortHistogramTestTypes, TestTypeNames);

TYPED_TEST(SortHistogramPerfTest, RadixSortHistogram) {
  using value_type = typename TypeParam::item_type::type;
  using size_type = unsigned;

  auto input = this->template GetConfigColumn<value_type>("input");
  ASSERT_NE(input.size(), 0u);

  auto check_result = this->GetConfigValue("check_result", true);

  constexpr int kBlockThreads = TypeParam::launch_params::BLOCK_THREADS;
  constexpr int kItemsPerThread = TypeParam::launch_params::ITEMS_PER_THREAD;
  constexpr int kBlockItems = kBlockThreads * kItemsPerThread;
  constexpr int kTileSize = TypeParam::TILE_SIZE;
  constexpr int kRadixBits = TypeParam::RADIX_BITS;
  constexpr int kTileItems = kBlockItems * kTileSize;
  constexpr int kEndBit = sizeof(value_type) * /*BITS_PER_BYTE=*/8;
  constexpr int kNumPasses = DivideAndRoundUp<kEndBit, kRadixBits>::VALUE;
  constexpr int kNumBins = 1 << kRadixBits;
  constexpr int kHistogramSize = kNumBins * kNumPasses;

  device_vector<value_type> items(input.size());
  device_vector<size_type> histogram(kHistogramSize);

  int num_blocks = (input.size() + kTileItems - 1) / kTileItems;

  // copy input to device memory
  items.copy_from_host(input.data(), input.size());

  // provide throughput information
  this->set_element_count(input.size());
  this->set_element_size(sizeof(value_type));
  this->set_elements_per_thread(kItemsPerThread);
  // count each atomic add as 1 load + 1 store
  int num_atomic_adds = kHistogramSize * num_blocks;
  this->set_global_memory_loads(input.size() * sizeof(value_type) +
                                num_atomic_adds * sizeof(size_type));
  this->set_global_memory_stores(num_atomic_adds * sizeof(size_type));

  this->MeasureWithSetup(
      kConfig,
      [&]() {
        hipMemsetAsync(histogram.data(), 0,
                        sizeof(size_type) * kHistogramSize);
      },
      [&]() {
        kernels::RadixSortHistogram<kBlockThreads, kItemsPerThread, kTileSize,
                                    kRadixBits><<<num_blocks, kBlockThreads>>>(
            items.data(), histogram.data(), items.size());
      });

  if (check_result) {
    std::vector<size_type> actual_histogram(histogram.size());
    histogram.copy_to_host(actual_histogram.data(), actual_histogram.size());
    std::vector<size_type> expected_histogram(histogram.size());
    int start_bit = 0;
    for (int j = 0; j < kNumPasses; ++j) {
      int num_pass_bits = std::min(kRadixBits, kEndBit - start_bit);
      for (const auto& value : input) {
        int bin = extract_bits(to_bit_ordered(value), start_bit, num_pass_bits);
        expected_histogram[j * kNumBins + bin] += 1u;
      }
      start_bit += kRadixBits;
    }
    EXPECT_EQ(expected_histogram, actual_histogram);
  }
}

}  // namespace test
}  // namespace breeze
