#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * Copyright (c) 2024 by Rivos Inc.
 * Licensed under the Apache License, Version 2.0, see LICENSE for details.
 * SPDX-License-Identifier: Apache-2.0
 */

#include <gtest/gtest.h>

#include <vector>

#include "breeze/algorithms/scan.h"
#include "breeze/functions/load.h"
#include "breeze/functions/scan.h"
#include "breeze/functions/store.h"
#include "breeze/platforms/platform.h"
#include "breeze/utils/block_details.h"
#include "breeze/utils/device_vector.h"
#include "perftest/perftest.h"

namespace breeze {

using namespace algorithms;
using namespace functions;
using namespace utils;

namespace test {
namespace kernels {

enum { CUDA_WARP_THREADS = 32 };

template <int BLOCK_THREADS, int ITEMS_PER_THREAD, int LOOKBACK_DISTANCE,
          typename T, typename U, typename BlockT>
__global__ __launch_bounds__(BLOCK_THREADS) void Scan(const T* in, U* out,
                                                      int* next_block_idx,
                                                      BlockT* blocks,
                                                      int num_items) {
  CudaPlatform<BLOCK_THREADS, CUDA_WARP_THREADS> p;
  using DeviceScanT =
      DeviceScan<decltype(p), U, ITEMS_PER_THREAD, LOOKBACK_DISTANCE>;
  __shared__ typename DeviceScanT::Scratch scratch;
  DeviceScanT::template Scan<ScanOpAdd>(
      p, make_slice<GLOBAL>(in), make_slice<GLOBAL>(out),
      make_slice<GLOBAL>(next_block_idx), make_slice<GLOBAL>(blocks),
      make_slice(&scratch).template reinterpret<SHARED>(), num_items);
}

}  // namespace kernels

using ScanConfig = PerfTestArrayConfig<12>;

const ScanConfig kConfig = {
    {{"num_input_rows", "525000"},
     {"num_input_rows_short", "10000"},
     {"num_input_rows_grande", "8400000"},
     {"num_input_rows_venti", "100000000"},
     {"input_generate_method", "FILL"},
     {"input_fill_value", "1"},
     {"num_expected_prefix_sums_rows", "525000"},
     {"num_expected_prefix_sums_rows_short", "10000"},
     {"num_expected_prefix_sums_rows_grande", "8400000"},
     {"num_expected_prefix_sums_rows_venti", "100000000"},
     {"expected_prefix_sums_generate_method", "SEQUENCE"},
     {"expected_prefix_sums_sequence_start", "1"}}};

template <typename TypeParam>
class ScanPerfTest : public PerfTest<ScanConfig>, public testing::Test {
 public:
  template <typename T>
  T GetConfigValue(const char* key, T default_value) const {
    return kConfig.get<T>(key, default_value);
  }
  template <typename T>
  std::vector<T> GetConfigColumn(const char* prefix) const {
    return kConfig.get_column<T>(prefix);
  }
};

template <int N>
struct LookbackDistance {
  enum { VALUE = N };
};

template <typename LaunchParamsAndItemTypeT, typename LookbackDistanceT>
struct ScanTestType {
  using launch_params_and_item_type_type = LaunchParamsAndItemTypeT;
  using item_type = typename LaunchParamsAndItemTypeT::item_type;
  using launch_params = typename LaunchParamsAndItemTypeT::launch_params;
  enum {
    LOOKBACK_DISTANCE = LookbackDistanceT::VALUE,
  };

  static std::string GetName() {
    return launch_params_and_item_type_type::GetName() + ".LookbackDistance" +
           std::to_string(LOOKBACK_DISTANCE);
  }
};

using LaunchParamsTypes =
    std::tuple<LaunchParams<256, 8>, LaunchParams<256, 16>>;

using LaunchParamsAndItemTypes =
    CombineLaunchParamsAndTypes<LaunchParamsAndItemType, LaunchParamsTypes, int,
                                long long>;

using ScanTestTypes =
    CombineTestTypes<ScanTestType, LaunchParamsAndItemTypes,
                     LookbackDistance<32>, LookbackDistance<64>,
                     LookbackDistance<96>, LookbackDistance<128>>;

using TestTypes = MakeTestTypes<ScanTestTypes>::types;

TYPED_TEST_SUITE(ScanPerfTest, TestTypes, TestTypeNames);

TYPED_TEST(ScanPerfTest, Scan) {
  using value_type = typename TypeParam::item_type::type;
  using sum_type = value_type;
  using block_type = typename try_make_unsigned<value_type>::type;

  auto input = this->template GetConfigColumn<value_type>("input");
  ASSERT_NE(input.size(), 0u);

  auto check_result = this->GetConfigValue("check_result", true);

  device_vector<value_type> items(input.size());
  device_vector<sum_type> prefix_sums(input.size());

  constexpr int kBlockThreads = TypeParam::launch_params::BLOCK_THREADS;
  constexpr int kItemsPerThread = TypeParam::launch_params::ITEMS_PER_THREAD;
  constexpr int kBlockItems = kBlockThreads * kItemsPerThread;
  constexpr int kLookbackDistance = TypeParam::LOOKBACK_DISTANCE;

  int num_blocks = (input.size() + kBlockItems - 1) / kBlockItems;

  device_vector<int> next_block_idx(1);
  device_vector<block_type> blocks(num_blocks);

  // copy input to device memory
  items.copy_from_host(input.data(), input.size());

  // provide throughput information
  this->set_element_count(input.size());
  this->set_element_size(sizeof(value_type));
  this->set_elements_per_thread(kItemsPerThread);
  this->template set_global_memory_loads<value_type>(input.size());
  this->template set_global_memory_stores<sum_type>(input.size());

  this->MeasureWithSetup(
      kConfig,
      [&]() {
        hipMemsetAsync(next_block_idx.data(), 0, sizeof(int));
        hipMemsetAsync(blocks.data(), 0, sizeof(block_type) * num_blocks);
      },
      [&]() {
        kernels::Scan<kBlockThreads, kItemsPerThread, kLookbackDistance>
            <<<num_blocks, kBlockThreads>>>(items.data(), prefix_sums.data(),
                                            next_block_idx.data(),
                                            blocks.data(), items.size());
      });

  if (check_result) {
    std::vector<sum_type> actual_prefix_sums(input.size());
    prefix_sums.copy_to_host(actual_prefix_sums.data(),
                             actual_prefix_sums.size());
    auto expected_prefix_sums =
        this->template GetConfigColumn<sum_type>("expected_prefix_sums");
    EXPECT_EQ(expected_prefix_sums, actual_prefix_sums);
  }
}

}  // namespace test
}  // namespace breeze
