#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * Copyright (c) 2024 by Rivos Inc.
 * Licensed under the Apache License, Version 2.0, see LICENSE for details.
 * SPDX-License-Identifier: Apache-2.0
 */

#include <gtest/gtest.h>

#include <vector>

#include "breeze/functions/scan.h"
#include "breeze/platforms/platform.h"
#include "breeze/utils/device_vector.h"
#include "perftest/perftest.h"

namespace breeze {

using namespace functions;
using namespace utils;

namespace test {
namespace kernels {

enum { CUDA_WARP_THREADS = 32 };

template <typename Op, int BLOCK_THREADS, int ITEMS_PER_THREAD, typename T>
__global__ __launch_bounds__(BLOCK_THREADS) void Scan(T* out,
                                                      int indirect_zero = 0) {
  CudaPlatform<BLOCK_THREADS, CUDA_WARP_THREADS> p;
  using BlockScanT = BlockScan<decltype(p), T, ITEMS_PER_THREAD>;
  __shared__ typename BlockScanT::Scratch scratch;
  T items[ITEMS_PER_THREAD];
  // generate input values
#pragma unroll
  for (int i = 0; i < ITEMS_PER_THREAD; ++i) {
    items[i] = 1;
  }
  T result[ITEMS_PER_THREAD];
  BlockScanT::template Scan<Op>(
      p, make_slice(items), make_slice(result),
      make_slice(&scratch).template reinterpret<SHARED>());
  if (p.thread_idx() == (BLOCK_THREADS - 1 - indirect_zero)) {
    out[p.block_idx()] = result[ITEMS_PER_THREAD - 1 - indirect_zero];
  }
}

}  // namespace kernels

using BlockScanConfig = PerfTestArrayConfig<4>;

const BlockScanConfig kConfig = {{
    {"num_items", "1050000"},
    {"num_items_short", "100000"},
    {"num_items_grande", "16800000"},
    {"num_items_venti", "500000000"},
}};

template <typename TypeParam>
class BlockScanPerfTest : public PerfTest<BlockScanConfig>,
                          public testing::Test {
 public:
  template <typename T>
  T GetConfigValue(const char* key, T default_value) const {
    return kConfig.get<T>(key, default_value);
  }
  template <typename T>
  T GetSizedConfigValue(const char* key, T default_value) const {
    return kConfig.get_sized<T>(key, default_value);
  }
};

template <typename LaunchParamsAndItemTypeT, typename ScanOpT>
struct ScanTestType {
  using launch_params_and_item_type_type = LaunchParamsAndItemTypeT;
  using item_type = typename LaunchParamsAndItemTypeT::item_type;
  using launch_params = typename LaunchParamsAndItemTypeT::launch_params;
  using scan_op_type = ScanOpT;

  static typename item_type::type GetExpectedResult() {
    if (std::is_same<scan_op_type, ScanOpAdd>())
      return launch_params::BLOCK_THREADS * launch_params::ITEMS_PER_THREAD;
    return 0;
  }
  static std::string GetScanOpName() {
    if (std::is_same<scan_op_type, ScanOpAdd>()) return "Add";
    return "?";
  }
  static std::string GetName() {
    return launch_params_and_item_type_type::GetName() + "." + GetScanOpName();
  }
};

using LaunchParamsTypes =
    std::tuple<LaunchParams<256, 1>, LaunchParams<256, 4>, LaunchParams<256, 8>,
               LaunchParams<256, 16>>;

using LaunchParamsAndItemTypes =
    CombineLaunchParamsAndTypes<LaunchParamsAndItemType, LaunchParamsTypes, int,
                                unsigned, long long, unsigned long long, float>;

using ScanTestTypes =
    CombineTestTypes<ScanTestType, LaunchParamsAndItemTypes, ScanOpAdd>;

using TestTypes = MakeTestTypes<ScanTestTypes>::types;

TYPED_TEST_SUITE(BlockScanPerfTest, TestTypes, TestTypeNames);

TYPED_TEST(BlockScanPerfTest, Scan) {
  using item_type = typename TypeParam::item_type::type;

  auto check_result = this->GetConfigValue("check_result", true);
  auto num_items = this->GetSizedConfigValue("num_items", 1);

  constexpr int kBlockThreads = TypeParam::launch_params::BLOCK_THREADS;
  constexpr int kItemsPerThread = TypeParam::launch_params::ITEMS_PER_THREAD;
  constexpr int kBlockItems = kBlockThreads * kItemsPerThread;

  int num_blocks = (num_items + kBlockItems - 1) / kBlockItems;
  device_vector<item_type> result(num_blocks);

  // provide throughput information
  this->set_element_count(num_blocks * kBlockItems);
  this->set_element_size(sizeof(item_type));
  this->set_elements_per_thread(kItemsPerThread);

  this->Measure(kConfig, [&]() {
    kernels::Scan<ScanOpAdd, kBlockThreads, kItemsPerThread>
        <<<num_blocks, kBlockThreads>>>(result.data());
  });

  if (check_result) {
    std::vector<item_type> actual_result(num_blocks);
    result.copy_to_host(actual_result.data(), actual_result.size());
    std::vector<item_type> expected_result(num_blocks,
                                           TypeParam::GetExpectedResult());
    EXPECT_EQ(actual_result, expected_result);
  }
}

}  // namespace test
}  // namespace breeze
