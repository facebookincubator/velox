#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * Copyright (c) 2024 by Rivos Inc.
 * Licensed under the Apache License, Version 2.0, see LICENSE for details.
 * SPDX-License-Identifier: Apache-2.0
 */

#include <gtest/gtest.h>

#include <limits>
#include <vector>

#include "breeze/functions/sort.h"
#include "breeze/platforms/platform.h"
#include "breeze/utils/device_vector.h"
#include "breeze/utils/types.h"
#include "perftest/perftest.h"

namespace breeze {

using namespace functions;
using namespace utils;

namespace test {
namespace kernels {

enum { CUDA_WARP_THREADS = 32 };

template <int BLOCK_THREADS, int ITEMS_PER_THREAD, int RADIX_BITS, typename T>
__global__ __launch_bounds__(BLOCK_THREADS) void RadixRank(
    T* out, int indirect_zero = 0) {
  CudaPlatform<BLOCK_THREADS, CUDA_WARP_THREADS> p;
  using BlockRankT = BlockRadixRank<decltype(p), ITEMS_PER_THREAD, RADIX_BITS>;
  __shared__ typename BlockRankT::Scratch scratch;
  T items[ITEMS_PER_THREAD];
  // generate input values
#pragma unroll
  for (int i = 0; i < ITEMS_PER_THREAD; ++i) {
    items[i] = (p.thread_idx() * ITEMS_PER_THREAD + i) % (1 << RADIX_BITS);
  }
  int ranks[ITEMS_PER_THREAD];
  BlockRankT::Rank(p, make_slice<THREAD, WARP_STRIPED>(items),
                   make_slice<THREAD, WARP_STRIPED>(ranks),
                   make_slice(&scratch).template reinterpret<SHARED>());
  if (p.thread_idx() == indirect_zero) {
    out[p.block_idx()] = ranks[indirect_zero];
  }
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD, int RADIX_BITS, typename T>
__global__ __launch_bounds__(BLOCK_THREADS) void RadixSort(
    T* out, int indirect_zero = 0) {
  CudaPlatform<BLOCK_THREADS, CUDA_WARP_THREADS> p;
  using BlockSortT =
      BlockRadixSort<decltype(p), ITEMS_PER_THREAD, RADIX_BITS, T>;
  __shared__ typename BlockSortT::Scratch scratch;
  T items[ITEMS_PER_THREAD];
  // generate input values
#pragma unroll
  for (int i = 0; i < ITEMS_PER_THREAD; ++i) {
    T mask = (p.thread_idx() * ITEMS_PER_THREAD + i) % (1 << RADIX_BITS);
    items[i] = mask | (mask << (sizeof(T) * 8 - RADIX_BITS - 1));
  }
  BlockSortT::Sort(p, make_slice<THREAD, WARP_STRIPED>(items),
                   make_slice(&scratch).template reinterpret<SHARED>());
  if (p.thread_idx() == indirect_zero) {
    out[p.block_idx()] = items[indirect_zero];
  }
}

}  // namespace kernels

using BlockRadixSortConfig = PerfTestArrayConfig<4>;

const BlockRadixSortConfig kConfig = {{
    {"num_items", "540000"},
    {"num_items_short", "50000"},
    {"num_items_grande", "8640000"},
    {"num_items_venti", "500000000"},
}};

template <typename TypeParam>
class BlockSortPerfTest : public PerfTest<BlockRadixSortConfig>,
                          public testing::Test {
 public:
  template <typename T>
  T GetConfigValue(const char* key, T default_value) const {
    return kConfig.get<T>(key, default_value);
  }
  template <typename T>
  T GetSizedConfigValue(const char* key, T default_value) const {
    return kConfig.get_sized<T>(key, default_value);
  }
};

template <int N>
struct RadixBits {
  enum {
    VALUE = N,
  };
};

template <typename LaunchParamsAndItemTypeT, typename RadixBitsT>
struct RadixTestType {
  using launch_params_and_item_type_type = LaunchParamsAndItemTypeT;
  using item_type = typename LaunchParamsAndItemTypeT::item_type;
  using launch_params = typename LaunchParamsAndItemTypeT::launch_params;
  enum {
    RADIX_BITS = RadixBitsT::VALUE,
  };

  static std::string GetName() {
    return launch_params_and_item_type_type::GetName() + ".Bits" +
           std::to_string(RADIX_BITS);
  }
};

using LaunchParamsTypes =
    std::tuple<LaunchParams<256, 8>, LaunchParams<256, 16>>;

using LaunchParamsAndItemTypes =
    CombineLaunchParamsAndTypes<LaunchParamsAndItemType, LaunchParamsTypes, int,
                                unsigned, long long, unsigned long long>;

using RadixTestTypes =
    CombineTestTypes<RadixTestType, LaunchParamsAndItemTypes, RadixBits<8>>;

using TestTypes = MakeTestTypes<RadixTestTypes>::types;

TYPED_TEST_SUITE(BlockSortPerfTest, TestTypes, TestTypeNames);

TYPED_TEST(BlockSortPerfTest, RadixRank) {
  using item_type = typename TypeParam::item_type::type;

  auto check_result = this->GetConfigValue("check_result", true);
  auto num_items = this->GetSizedConfigValue("num_items", 1);

  constexpr int kBlockThreads = TypeParam::launch_params::BLOCK_THREADS;
  constexpr int kItemsPerThread = TypeParam::launch_params::ITEMS_PER_THREAD;
  constexpr int kBlockItems = kBlockThreads * kItemsPerThread;

  int num_blocks = (num_items + kBlockItems - 1) / kBlockItems;
  device_vector<item_type> result(num_blocks);

  // provide throughput information
  this->set_element_count(num_blocks * kBlockItems);
  this->set_element_size(sizeof(item_type));
  this->set_elements_per_thread(kItemsPerThread);

  this->Measure(kConfig, [&]() {
    kernels::RadixRank<kBlockThreads, kItemsPerThread, TypeParam::RADIX_BITS>
        <<<num_blocks, kBlockThreads>>>(result.data());
  });

  if (check_result) {
    std::vector<item_type> actual_result(num_blocks);
    result.copy_to_host(actual_result.data(), actual_result.size());
    std::vector<item_type> expected_result(num_blocks, 0);
    EXPECT_EQ(actual_result, expected_result);
  }
}

TYPED_TEST(BlockSortPerfTest, RadixSort) {
  using item_type = typename TypeParam::item_type::type;

  auto check_result = this->GetConfigValue("check_result", true);
  auto num_items = this->GetSizedConfigValue("num_items", 1);

  constexpr int kBlockThreads = TypeParam::launch_params::BLOCK_THREADS;
  constexpr int kItemsPerThread = TypeParam::launch_params::ITEMS_PER_THREAD;
  constexpr int kBlockItems = kBlockThreads * kItemsPerThread;

  int num_blocks = (num_items + kBlockItems - 1) / kBlockItems;
  device_vector<item_type> result(num_blocks);

  // provide throughput information
  this->set_element_count(num_blocks * kBlockItems);
  this->set_element_size(sizeof(item_type));
  this->set_elements_per_thread(kItemsPerThread);

  this->Measure(kConfig, [&]() {
    kernels::RadixSort<kBlockThreads, kItemsPerThread, TypeParam::RADIX_BITS>
        <<<num_blocks, kBlockThreads>>>(result.data());
  });

  if (check_result) {
    std::vector<item_type> actual_result(num_blocks);
    result.copy_to_host(actual_result.data(), actual_result.size());
    std::vector<item_type> expected_result(num_blocks, 0);
    EXPECT_EQ(actual_result, expected_result);
  }
}

}  // namespace test
}  // namespace breeze
