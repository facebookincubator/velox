#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * Copyright (c) 2024 by Rivos Inc.
 * Licensed under the Apache License, Version 2.0, see LICENSE for details.
 * SPDX-License-Identifier: Apache-2.0
 */

#include <gtest/gtest.h>

#include <vector>

#include "breeze/functions/reduce.h"
#include "breeze/platforms/platform.h"
#include "breeze/utils/device_vector.h"
#include "perftest/perftest.h"

namespace breeze {

using namespace functions;
using namespace utils;

namespace test {
namespace kernels {

enum { CUDA_WARP_THREADS = 32 };

template <typename Op, int BLOCK_THREADS, int ITEMS_PER_THREAD, typename T>
__global__ __launch_bounds__(BLOCK_THREADS) void Reduce(T* out) {
  CudaPlatform<BLOCK_THREADS, CUDA_WARP_THREADS> p;
  using BlockReduceT = BlockReduce<decltype(p), T>;
  __shared__ typename BlockReduceT::Scratch scratch;
  T items[ITEMS_PER_THREAD];
  // generate input values
#pragma unroll
  for (int i = 0; i < ITEMS_PER_THREAD; ++i) {
    items[i] = 1;
  }
  T aggregate = BlockReduceT::template Reduce<Op, ITEMS_PER_THREAD>(
      p, make_slice(items),
      make_slice(&scratch).template reinterpret<SHARED>());
  if (p.thread_idx() == 0) {
    out[p.block_idx()] = aggregate;
  }
}

}  // namespace kernels

using BlockReduceConfig = PerfTestArrayConfig<4>;

const BlockReduceConfig kConfig = {{
    {"num_items", "2100000"},
    {"num_items_short", "250000"},
    {"num_items_grande", "33600000"},
    {"num_items_venti", "500000000"},
}};

template <typename TypeParam>
class BlockReducePerfTest : public PerfTest<BlockReduceConfig>,
                            public testing::Test {
 public:
  template <typename T>
  T GetConfigValue(const char* key, T default_value) const {
    return kConfig.get<T>(key, default_value);
  }
  template <typename T>
  T GetSizedConfigValue(const char* key, T default_value) const {
    return kConfig.get_sized<T>(key, default_value);
  }
};

template <typename LaunchParamsAndItemTypeT, typename ReduceOpT>
struct ReduceTestType {
  using launch_params_and_item_type_type = LaunchParamsAndItemTypeT;
  using item_type = typename LaunchParamsAndItemTypeT::item_type;
  using launch_params = typename LaunchParamsAndItemTypeT::launch_params;
  using reduce_op_type = ReduceOpT;

  static typename item_type::type GetExpectedResult() {
    if (std::is_same<reduce_op_type, ReduceOpAdd>())
      return launch_params::BLOCK_THREADS * launch_params::ITEMS_PER_THREAD;
    if (std::is_same<reduce_op_type, ReduceOpMin>()) return 1;
    if (std::is_same<reduce_op_type, ReduceOpMax>()) return 1;
    return 0;
  }
  static std::string GetReduceOpName() {
    if (std::is_same<reduce_op_type, ReduceOpAdd>()) return "Add";
    if (std::is_same<reduce_op_type, ReduceOpMin>()) return "Min";
    if (std::is_same<reduce_op_type, ReduceOpMax>()) return "Max";
    return "?";
  }
  static std::string GetName() {
    return launch_params_and_item_type_type::GetName() + "." +
           GetReduceOpName();
  }
};

using LaunchParamsTypes =
    std::tuple<LaunchParams<256, 8>, LaunchParams<256, 16>>;

using LaunchParamsAndItemTypes =
    CombineLaunchParamsAndTypes<LaunchParamsAndItemType, LaunchParamsTypes, int,
                                unsigned, long long, unsigned long long, float>;

using ReduceTestTypes =
    CombineTestTypes<ReduceTestType, LaunchParamsAndItemTypes, ReduceOpAdd,
                     ReduceOpMin, ReduceOpMax>;

using TestTypes = MakeTestTypes<ReduceTestTypes>::types;

TYPED_TEST_SUITE(BlockReducePerfTest, TestTypes, TestTypeNames);

TYPED_TEST(BlockReducePerfTest, Reduce) {
  using item_type = typename TypeParam::item_type::type;
  using reduce_op_type = typename TypeParam::reduce_op_type;

  auto check_result = this->GetConfigValue("check_result", true);
  auto num_items = this->GetSizedConfigValue("num_items", 1);

  constexpr int kBlockThreads = TypeParam::launch_params::BLOCK_THREADS;
  constexpr int kItemsPerThread = TypeParam::launch_params::ITEMS_PER_THREAD;
  constexpr int kBlockItems = kBlockThreads * kItemsPerThread;

  int num_blocks = (num_items + kBlockItems - 1) / kBlockItems;
  device_vector<item_type> result(num_blocks);

  // provide throughput information
  this->set_element_count(num_blocks * kBlockItems);
  this->set_element_size(sizeof(item_type));
  this->set_elements_per_thread(kItemsPerThread);

  this->Measure(kConfig, [&]() {
    kernels::Reduce<reduce_op_type, kBlockThreads, kItemsPerThread>
        <<<num_blocks, kBlockThreads>>>(result.data());
  });

  if (check_result) {
    std::vector<item_type> actual_result(num_blocks);
    result.copy_to_host(actual_result.data(), actual_result.size());
    std::vector<item_type> expected_result(num_blocks,
                                           TypeParam::GetExpectedResult());
    EXPECT_EQ(actual_result, expected_result);
  }
}

}  // namespace test
}  // namespace breeze
